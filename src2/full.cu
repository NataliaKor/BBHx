#include "hip/hip_runtime.h"
/*  This code was edited by Michael Katz. It is originally from the LAL library.
 *  The original copyright and license is shown below. Michael Katz has edited
 *  the code for his purposes and removed dependencies on the LAL libraries. The code has been confirmed to match the LAL version.
 *  This code is distrbuted under the same GNU license it originally came with.
 *  The comments in the code have been left generally the same. A few comments
 *  have been made for the newer functions added.


 *  Copyright (C) 2017 Sebastian Khan, Francesco Pannarale, Lionel London
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with with program; see the file COPYING. If not, write to the
 *  Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 *  MA  02111-1307  USA
 */
#include <math.h>
#include <complex>
#include <iostream>
#include "stdio.h"
#include <random>

#include "hip/hip_complex.h"
#include "hipblas.h"

#include <stdbool.h>
#include "full.h"

#include "hipsparse.h"

#include "constants.h"
#include "global.h"
// #include "PhenomHM.hh"

#define  NUM_THREADS 256
#define  NUM_THREADS2 64
#define  NUM_THREADS3 256
#define  NUM_THREADS4 256



__device__
double d_dot_product_1d(double* arr1, double* arr2){
    double out = 0.0;
    for (int i=0; i<3; i++){
        out += arr1[i]*arr2[i];
    }
    return out;
}


__device__
cmplx d_vec_H_vec_product(double* arr1, cmplx* H, double* arr2){

    cmplx I(0.0, 1.0);
    cmplx out(0.0, 0.0);
    cmplx trans(0.0, 0.0);
    for (int i=0; i<3; i++){
        trans = cmplx(0.0, 0.0);
        for (int j=0; j<3; j++){
            trans += (H[i*3 + j] * arr2[j]);
        }
        out += arr1[i]*trans;
    }
    return out;
}

__device__
double d_sinc(double x){
    if (x == 0.0) return 1.0;
    else return sin(x)/x;
}


/* # Single-link response
# 'full' does include the orbital-delay term, 'constellation' does not
 */
__device__
d_Gslr_holder d_EvaluateGslr(double t, double f, cmplx *H, double* k, int response, double* p0){
    // response == 1 is full ,, response anything else is constellation
    //# Trajectories, p0 used only for the full response
    cmplx I(0.0, 1.0);
    cmplx m_I(0.0, -1.0);
    double alpha = Omega0*t; double c = cos(alpha); double s = sin(alpha);
    double a = aorbit; double e = eorbit;

    //double p0[3] = {a*c, a*s, 0.*t}; // funcp0(t)
    __shared__ double p1L_all[NUM_THREADS2 * 3];
    double* p1L = &p1L_all[threadIdx.x * 3];
    p1L[0] = - a*e*(1 + s*s);
    p1L[1] = a*e*c*s;
    p1L[2] = -a*e*sqrt3*c;

    __shared__ double p2L_all[NUM_THREADS2 * 3];
    double* p2L = &p2L_all[threadIdx.x * 3];
    p2L[0] = a*e/2*(sqrt3*c*s + (1 + s*s));
    p2L[1] = a*e/2*(-c*s - sqrt3*(1 + c*c));
    p2L[2] = -a*e*sqrt3/2*(sqrt3*s - c);

    __shared__ double p3L_all[NUM_THREADS2 * 3];
    double* p3L = &p3L_all[threadIdx.x * 3];
    p3L[0] = a*e/2*(-sqrt3*c*s + (1 + s*s));
    p3L[1] = a*e/2*(-c*s + sqrt3*(1 + c*c));
    p3L[2] = -a*e*sqrt3/2*(-sqrt3*s - c);

    __shared__ double n_all[NUM_THREADS2 * 3];
    double* n = &n_all[threadIdx.x * 3];

    // n1
    n[0] = -1./2*c*s;
    n[1] = 1./2*(1 + c*c);
    n[2] = sqrt3/2*s;

    double kn1= d_dot_product_1d(k, n);
    cmplx n1Hn1 = d_vec_H_vec_product(n, H, n); //np.dot(n1, np.dot(H, n1))

    // n2
    n[0] = c*s - sqrt3*(1 + s*s);
    n[1] = sqrt3*c*s - (1 + c*c);
    n[2] = -sqrt3*s - 3*c;

    for (int i=0; i<3; i++) n[i] = n[i]*1./4.;

    double kn2= d_dot_product_1d(k, n);
    cmplx n2Hn2 = d_vec_H_vec_product(n, H, n); //np.dot(n1, np.dot(H, n1))

    // n3

    n[0] = c*s + sqrt3*(1 + s*s);
    n[1] = -sqrt3*c*s - (1 + c*c);
    n[2] = -sqrt3*s + 3*c;

    for (int i=0; i<3; i++) n[i] = n[i]*1./4.;

    double kn3= d_dot_product_1d(k, n);
    cmplx n3Hn3 = d_vec_H_vec_product(n, H, n); //np.dot(n1, np.dot(H, n1))


    // # Compute intermediate scalar products
    // t scalar case

    double temp1 = p1L[0]+p2L[0]; double temp2 = p1L[1]+p2L[1]; double temp3 = p1L[2]+p2L[2];
    double temp4 = p2L[0]+p3L[0]; double temp5 = p2L[1]+p3L[1]; double temp6 = p2L[2]+p3L[2];
    double temp7 = p3L[0]+p1L[0]; double temp8 = p3L[1]+p1L[1]; double temp9 = p3L[2]+p1L[2];

    p1L[0] = temp1; p1L[1] = temp2; p1L[2] = temp3;  // now p1L_plus_p2L -> p1L
    p2L[0] = temp4; p2L[1] = temp5; p2L[2] = temp6;  // now p2L_plus_p3L -> p2L
    p3L[0] = temp7; p3L[1] = temp8; p3L[2] = temp9;  // now p3L_plus_p1L -> p3L

    double kp1Lp2L = d_dot_product_1d(k, p1L);
    double kp2Lp3L = d_dot_product_1d(k, p2L);
    double kp3Lp1L = d_dot_product_1d(k, p3L);
    double kp0 = d_dot_product_1d(k, p0);

    // # Prefactors - projections are either scalars or vectors
    cmplx factorcexp0;
    if (response==1) factorcexp0 = gcmplx::exp(I*2.*PI*f/C_SI * kp0); // I*2.*PI*f/C_SI * kp0
    else factorcexp0 = cmplx(1.0, 0.0);
    double prefactor = PI*f*L_SI/C_SI;

    cmplx factorcexp12 = gcmplx::exp(I*prefactor * (1.+kp1Lp2L/L_SI)); //prefactor * (1.+kp1Lp2L/L_SI)
    cmplx factorcexp23 = gcmplx::exp(I*prefactor * (1.+kp2Lp3L/L_SI)); //prefactor * (1.+kp2Lp3L/L_SI)
    cmplx factorcexp31 = gcmplx::exp(I*prefactor * (1.+kp3Lp1L/L_SI)); //prefactor * (1.+kp3Lp1L/L_SI)

    cmplx factorsinc12 = d_sinc( prefactor * (1.-kn3));
    cmplx factorsinc21 = d_sinc( prefactor * (1.+kn3));
    cmplx factorsinc23 = d_sinc( prefactor * (1.-kn1));
    cmplx factorsinc32 = d_sinc( prefactor * (1.+kn1));
    cmplx factorsinc31 = d_sinc( prefactor * (1.-kn2));
    cmplx factorsinc13 = d_sinc( prefactor * (1.+kn2));

    // # Compute the Gslr - either scalars or vectors
    d_Gslr_holder Gslr_out;


    cmplx commonfac = I*prefactor*factorcexp0;
    Gslr_out.G12 = commonfac * n3Hn3 * factorsinc12 * factorcexp12;
    Gslr_out.G21 = commonfac * n3Hn3 * factorsinc21 * factorcexp12;
    Gslr_out.G23 = commonfac * n1Hn1 * factorsinc23 * factorcexp23;
    Gslr_out.G32 = commonfac * n1Hn1 * factorsinc32 * factorcexp23;
    Gslr_out.G31 = commonfac * n2Hn2 * factorsinc31 * factorcexp31;
    Gslr_out.G13 = commonfac * n2Hn2 * factorsinc13 * factorcexp31;

    // ### FIXME
    // # G13 = -1j * prefactor * n2Hn2 * factorsinc31 * np.conjugate(factorcexp31)
    return Gslr_out;
}



__device__
d_transferL_holder d_TDICombinationFD(d_Gslr_holder Gslr, double f, int TDItag, int rescaled){
    // int TDItag == 1 is XYZ int TDItag == 2 is AET
    // int rescaled == 1 is True int rescaled == 0 is False
    d_transferL_holder transferL;
    cmplx factor, factorAE, factorT;
    cmplx I(0.0, 1.0);
    double x = PI*f*L_SI/C_SI;
    cmplx z = gcmplx::exp(I*2.*x);
    cmplx Xraw, Yraw, Zraw, Araw, Eraw, Traw;
    cmplx factor_convention, point5, c_one, c_two;
    if (TDItag==1){
        // # First-generation TDI XYZ
        // # With x=pifL, factor scaled out: 2I*sin2x*e2ix
        if (rescaled == 1) factor = 1.;
        else factor = 2.*I*sin(2.*x)*z;
        Xraw = Gslr.G21 + z*Gslr.G12 - Gslr.G31 - z*Gslr.G13;
        Yraw = Gslr.G32 + z*Gslr.G23 - Gslr.G12 - z*Gslr.G21;
        Zraw = Gslr.G13 + z*Gslr.G31 - Gslr.G23 - z*Gslr.G32;
        transferL.transferL1 = factor * Xraw;
        transferL.transferL2 = factor * Yraw;
        transferL.transferL3 = factor * Zraw;
        return transferL;
    }

    else{
        //# First-generation TDI AET from X,Y,Z
        //# With x=pifL, factors scaled out: A,E:I*sqrt2*sin2x*e2ix T:2*sqrt2*sin2x*sinx*e3ix
        //# Here we include a factor 2, because the code was first written using the definitions (2) of McWilliams&al_0911 where A,E,T are 1/2 of their LDC definitions
        factor_convention = cmplx(2.,0.0);
        if (rescaled == 1){
            factorAE = cmplx(1., 0.0);
            factorT = cmplx(1., 0.0);
        }
        else{
          factorAE = I*sqrt2*sin(2.*x)*z;
          factorT = 2.*sqrt2*sin(2.*x)*sin(x)*gcmplx::exp(I*3.*x);
        }

        Araw = 0.5 * ( (1.+z)*(Gslr.G31 + Gslr.G13) - Gslr.G23 - z*Gslr.G32 - Gslr.G21 - z*Gslr.G12 );
        Eraw = 0.5*invsqrt3 * ( (1.-z)*(Gslr.G13 - Gslr.G31) + (2.+z)*(Gslr.G12 - Gslr.G32) + (1.+2.*z)*(Gslr.G21 - Gslr.G23) );
        Traw = invsqrt6 * ( Gslr.G21 - Gslr.G12 + Gslr.G32 - Gslr.G23 + Gslr.G13 - Gslr.G31);
        transferL.transferL1 = factor_convention * factorAE * Araw;
        transferL.transferL2 = factor_convention * factorAE * Eraw;
        transferL.transferL3 = factor_convention * factorT * Traw;
        return transferL;
    }
}


__device__
d_transferL_holder d_JustLISAFDresponseTDI(cmplx *H, double f, double t, double lam, double beta, double t0, int TDItag, int order_fresnel_stencil){
    t = t + t0*YRSID_SI;

    //funck
    __shared__ double kvec_all[NUM_THREADS2 * 3];
    double* kvec = &kvec_all[threadIdx.x * 3];
    kvec[0] = -cos(beta)*cos(lam);
    kvec[1] = -cos(beta)*sin(lam);
    kvec[2] = -sin(beta);

    // funcp0
    double alpha = Omega0*t; double c = cos(alpha); double s = sin(alpha); double a = aorbit;

    __shared__ double p0_all[NUM_THREADS2 * 3];
    double* p0 = &p0_all[threadIdx.x * 3];
    p0[0] = a*c;
    p0[1] = a*s;
    p0[2] = 0.*t;

    // dot kvec with p0
    double kR = d_dot_product_1d(kvec, p0);

    double phaseRdelay = 2.*PI/clight *f*kR;

    // going to assume order_fresnel_stencil == 0 for now
    d_Gslr_holder Gslr = d_EvaluateGslr(t, f, H, kvec, 1, p0); // assumes full response
    d_Gslr_holder Tslr; // use same struct because its the same setup
    cmplx m_I(0.0, -1.0); // -1.0 -> mu_I

    // fill Tslr
    Tslr.G12 = Gslr.G12*gcmplx::exp(m_I*phaseRdelay); // really -I*
    Tslr.G21 = Gslr.G21*gcmplx::exp(m_I*phaseRdelay);
    Tslr.G23 = Gslr.G23*gcmplx::exp(m_I*phaseRdelay);
    Tslr.G32 = Gslr.G32*gcmplx::exp(m_I*phaseRdelay);
    Tslr.G31 = Gslr.G31*gcmplx::exp(m_I*phaseRdelay);
    Tslr.G13 = Gslr.G13*gcmplx::exp(m_I*phaseRdelay);

    d_transferL_holder transferL = d_TDICombinationFD(Tslr, f, TDItag, 0);
    transferL.phaseRdelay = phaseRdelay;
    return transferL;
}





 /**
  * Michael Katz added this function.
  * internal function that filles amplitude and phase for a specific frequency and mode.
  */
 __device__
 void response_modes(double* phases, double* response_out, int binNum, int mode_i, double* phases_deriv, double* freqs, double phiRef, int ell, int mm, int length, int numBinAll, int numModes,
 cmplx* H, double lam, double beta, double tRef_wave_frame, double tRef_sampling_frame, double tBase, int TDItag, int order_fresnel_stencil)
 {

         double amp_i, phase_i, dphidf, phase_up, phase_down;
         double t_wave_frame, t_sampling_frame;
         int status_in_for;

         int retcode = 0;
         double eps = 1e-9;
         int start_ind = 0;

         for (int i = threadIdx.x; i < length; i += blockDim.x)
         {
             //int mode_index = (i * numModes + mode_i) * numBinAll + binNum;
             //int freq_index = i * numBinAll + binNum;

             int mode_index = (binNum * numModes + mode_i) * length + i;
             int freq_index = binNum * length + i;

             double freq = freqs[freq_index];
             //double freq_geom = freq*M_tot_sec;

             dphidf = phases_deriv[mode_index];

             t_wave_frame = 1./(2.0*PI)*dphidf + tRef_wave_frame;
             t_sampling_frame = 1./(2.0*PI)*dphidf + tRef_sampling_frame;

             d_transferL_holder transferL = d_JustLISAFDresponseTDI(H, freq, t_wave_frame, lam, beta, tBase, TDItag, order_fresnel_stencil);

             // transferL1_re
             start_ind = 0 * numBinAll * numModes * length;
             int start_ind_old = start_ind;
             response_out[start_ind + mode_index] = gcmplx::real(transferL.transferL1);

             // transferL1_im
             start_ind = 1 * numBinAll * numModes * length;
             response_out[start_ind + mode_index] = gcmplx::imag(transferL.transferL1);

             // transferL1_re
             start_ind = 2 * numBinAll * numModes * length;
             response_out[start_ind + mode_index] = gcmplx::real(transferL.transferL2);

             // transferL1_re
             start_ind = 3 * numBinAll * numModes * length;
             response_out[start_ind + mode_index] = gcmplx::imag(transferL.transferL2);

             // transferL1_re
             start_ind = 4 * numBinAll * numModes * length;
             response_out[start_ind + mode_index] = gcmplx::real(transferL.transferL3);

             // transferL1_re
             start_ind = 5 * numBinAll * numModes * length;
             response_out[start_ind + mode_index] = gcmplx::imag(transferL.transferL3);

             // time_freq_corr update
             phases_deriv[mode_index] = t_sampling_frame + tBase * YRSID_SI;
             phases[mode_index] +=  transferL.phaseRdelay; // TODO: check this / I think I just need to remove it if phaseRdelay is exactly equal to (tRef_wave_frame * f) phase shift

         }
}



/*
Calculate spin weighted spherical harmonics
*/
__device__
cmplx SpinWeightedSphericalHarmonic(int s, int l, int m, double theta, double phi){
    // l=2
    double fac;
    if ((l==2) && (m==-2)) fac =  sqrt( 5.0 / ( 64.0 * PI ) ) * ( 1.0 - cos( theta ))*( 1.0 - cos( theta ));
    else if ((l==2) && (m==-1)) fac =  sqrt( 5.0 / ( 16.0 * PI ) ) * sin( theta )*( 1.0 - cos( theta ));
    else if ((l==2) && (m==1)) fac =  sqrt( 5.0 / ( 16.0 * PI ) ) * sin( theta )*( 1.0 + cos( theta ));
    else if ((l==2) && (m==2)) fac =  sqrt( 5.0 / ( 64.0 * PI ) ) * ( 1.0 + cos( theta ))*( 1.0 + cos( theta ));
    // l=3
    else if ((l==3) && (m==-3)) fac =  sqrt(21.0/(2.0*PI))*cos(theta/2.0)*pow(sin(theta/2.0),5.0);
    else if ((l==3) && (m==-2)) fac =  sqrt(7.0/(4.0*PI))*(2.0 + 3.0*cos(theta))*pow(sin(theta/2.0),4.0);
    else if ((l==3) && (m==2)) fac =  sqrt(7.0/PI)*pow(cos(theta/2.0),4.0)*(-2.0 + 3.0*cos(theta))/2.0;
    else if ((l==3) && (m==3)) fac =  -sqrt(21.0/(2.0*PI))*pow(cos(theta/2.0),5.0)*sin(theta/2.0);
    // l=4
    else if ((l==4) && (m==-4)) fac =  3.0*sqrt(7.0/PI)*pow(cos(theta/2.0),2.0)*pow(sin(theta/2.0),6.0);
    else if ((l==4) && (m==-3)) fac =  3.0*sqrt(7.0/(2.0*PI))*cos(theta/2.0)*(1.0 + 2.0*cos(theta))*pow(sin(theta/2.0),5.0);

    else if ((l==4) && (m==3)) fac =  -3.0*sqrt(7.0/(2.0*PI))*pow(cos(theta/2.0),5.0)*(-1.0 + 2.0*cos(theta))*sin(theta/2.0);
    else if ((l==4) && (m==4)) fac =  3.0*sqrt(7.0/PI)*pow(cos(theta/2.0),6.0)*pow(sin(theta/2.0),2.0);

    // Result
    cmplx I(0.0, 1.0);
    if (m==0) return cmplx(fac, 0.0);
    else {
        cmplx phaseTerm(m*phi, 0.0);
        return fac * exp(I*phaseTerm);
    }
}



/*
custom dot product in 2d
*/
__device__
void dot_product_2d(double* out, double* arr1, int m1, int n1, double* arr2, int m2, int n2, int dev, int stride){

    // dev and stride are on output
    for (int i=0; i<m1; i++){
        for (int j=0; j<n2; j++){
            out[stride*(i * 3  + j) + dev] = 0.0;
            for (int k=0; k<n1; k++){
                out[stride*(i * 3  + j) + dev] += arr1[i * 3 + k]*arr2[k * 3 + j];
            }
        }
    }
}

/*
Custom dot product in 1d
*/
__device__
double dot_product_1d(double arr1[3], double arr2[3]){
    double out = 0.0;
    for (int i=0; i<3; i++){
        out += arr1[i]*arr2[i];
    }
    return out;
}



/**
 * Michael Katz added this function.
 * Main function for calculating PhenomHM in the form used by Michael Katz
 * This is setup to allow for pre-allocation of arrays. Therefore, all arrays
 * should be setup outside of this function.
 */
__device__
void responseCore(
    double* phases,
    double* response_out,
    int *ells,
    int *mms,
    double* phases_deriv,
    double* freqs,                      /**< GW frequecny list [Hz] */
    const double phiRef,                        /**< orbital phase at f_ref */
    double f_ref,
    double inc,
    double lam,
    double beta,
    double psi,
    double tRef_wave_frame,
    double tRef_sampling_frame,
    int length,                              /**< reference GW frequency */
    int numModes,
    int binNum,
    int numBinAll,
    double tBase, int TDItag, int order_fresnel_stencil
)
{

    int ell, mm;

    //// setup response
    __shared__ double HSplus[9];
    __shared__ double HScross[9];

    if (threadIdx.x == 0)
    {
        HSplus[0] = 1.;
        HSplus[1] = 0.;
        HSplus[2] = 0.;
        HSplus[3] = 0.;
        HSplus[4] = -1.;
        HSplus[5] = 0.;
        HSplus[6] = 0.;
        HSplus[7] = 0.;
        HSplus[8] = 0.;

        HScross[0] = 0.;
        HScross[1] = 1.;
        HScross[2] = 0.;
        HScross[3] = 1.;
        HScross[4] = 0.;
        HScross[5] = 0.;
        HScross[6] = 0.;
        HScross[7] = 0.;
        HScross[8] = 0.;
    }
    __syncthreads();

    __shared__ cmplx H_mat_all[NUM_THREADS2 * 3 * 3];
    cmplx* H_mat = &H_mat_all[threadIdx.x * 3 * 3];

    //##### Based on the f-n by Sylvain   #####
    //__shared__ double Hplus_all[NUM_THREADS2 * 3 * 3];
    //__shared__ double Hcross_all[NUM_THREADS2 * 3 * 3];
    //double* Hplus = &Hplus_all[threadIdx.x * 3 * 3];
    //double* Hcross = &Hcross_all[threadIdx.x * 3 * 3];

    double* Htemp = (double*) &H_mat[0];  // Htemp alternates with Hplus and Hcross in order to save shared memory: Hp[0], Hc[0], Hp[1], Hc1]
    // Htemp is then transformed into H_mat

    // Wave unit vector
    __shared__ double kvec_all[NUM_THREADS2 * 3];
    double* kvec = &kvec_all[threadIdx.x * 3];
    kvec[0] = -cos(beta)*cos(lam);
    kvec[1] = -cos(beta)*sin(lam);
    kvec[2] = -sin(beta);

    // Compute constant matrices Hplus and Hcross in the SSB frame
    double clambd = cos(lam); double slambd = sin(lam);
    double cbeta = cos(beta); double sbeta = sin(beta);
    double cpsi = cos(psi); double spsi = sin(psi);

    __shared__ double O1_all[NUM_THREADS2 * 3 * 3];
    double* O1 = &O1_all[threadIdx.x * 3 * 3];
    O1[0] = cpsi*slambd-clambd*sbeta*spsi;
    O1[1] = -clambd*cpsi*sbeta-slambd*spsi;
    O1[2] = -cbeta*clambd;
    O1[3] = -clambd*cpsi-sbeta*slambd*spsi;
    O1[4] = -cpsi*sbeta*slambd+clambd*spsi;
    O1[5] = -cbeta*slambd;
    O1[6] = cbeta*spsi;
    O1[7] = cbeta*cpsi;
    O1[8] = -sbeta;

    __shared__ double invO1_all[NUM_THREADS2 * 3 * 3];
    double* invO1 = &invO1_all[threadIdx.x * 3 * 3];;
    invO1[0] = cpsi*slambd-clambd*sbeta*spsi;
    invO1[1] = -clambd*cpsi-sbeta*slambd*spsi;
    invO1[2] = cbeta*spsi;
    invO1[3] = -clambd*cpsi*sbeta-slambd*spsi;
    invO1[4] = -cpsi*sbeta*slambd+clambd*spsi;
    invO1[5] = cbeta*cpsi;
    invO1[6] = -cbeta*clambd;
    invO1[7] = -cbeta*slambd;
    invO1[8] = -sbeta;

    __shared__ double out1_all[NUM_THREADS2 * 3 * 3];

    double* out1 = &out1_all[threadIdx.x * 3 * 3];


    // get Hplus
    //if ((threadIdx.x + blockDim.x * blockIdx.x <= 1)) printf("INNER %d %e %e %e\n", threadIdx.x + blockDim.x * blockIdx.x, invO1[0], invO1[1], invO1[6]);

    dot_product_2d(out1, HSplus, 3, 3, invO1, 3, 3, 0, 1);

    dot_product_2d(Htemp, O1, 3, 3, out1, 3, 3, 0, 2);

    // get Hcross
    dot_product_2d(out1, HScross, 3, 3, invO1, 3, 3, 0, 1);
    dot_product_2d(Htemp, O1, 3, 3, out1, 3, 3, 1, 2);

    cmplx I = cmplx(0.0, 1.0);
    cmplx Ylm, Yl_m, Yfactorplus, Yfactorcross;

    cmplx trans1, trans2;

    for (int mode_i=0; mode_i<numModes; mode_i++){
        ell = ells[mode_i];
        mm = mms[mode_i];

        Ylm = SpinWeightedSphericalHarmonic(-2, ell, mm, inc, phiRef);
        Yl_m = pow(-1.0, ell)*gcmplx::conj(SpinWeightedSphericalHarmonic(-2, ell, -1*mm, inc, phiRef));
        Yfactorplus = 1./2 * (Ylm + Yl_m);
        //# Yfactorcross = 1j/2 * (Y22 - Y2m2)  ### SB, should be for correct phase conventions
        Yfactorcross = 1./2. * I * (Ylm - Yl_m); //  ### SB, minus because the phase convention is opposite, we'll tace c.c. at the end
        //# Yfactorcross = -1j/2 * (Y22 - Y2m2)  ### SB, minus because the phase convention is opposite, we'll tace c.c. at the end
        //# Yfactorcross = 1j/2 * (Y22 - Y2m2)  ### SB, minus because the phase convention is opposite, we'll tace c.c. at the end
        //# The matrix H_mat is now complex

        //# H_mat = np.conjugate((Yfactorplus*Hplus + Yfactorcross*Hcross))  ### SB: H_ij = H_mat A_22 exp(i\Psi(f))
        for (int i=0; i<3; i++){
            for (int j=0; j<3; j++){
                trans1 = Htemp[2*(i * 3 + j) + 0];
                trans2 = Htemp[2*(i * 3 + j) + 1];
                H_mat[(i * 3 + j)] = (Yfactorplus*trans1+ Yfactorcross*trans2);
                //printf("(%d, %d): %e, %e\n", i, j, Hplus[i][j], Hcross[i][j]);
            }
        }

        response_modes(phases, response_out, binNum, mode_i, phases_deriv, freqs, phiRef, ell, mm, length, numBinAll, numModes,
        H_mat, lam, beta, tRef_wave_frame, tRef_sampling_frame, tBase, TDItag, order_fresnel_stencil);

    }
}



////////////
// response
////////////

#define MAX_MODES 6

 CUDA_KERNEL
 void response(
     double* phases,
     double* response_out,
     double* phases_deriv,
     int* ells_in,
     int* mms_in,
     double* freqs,               /**< Frequency points at which to evaluate the waveform (Hz) */
     double* phiRef,                 /**< reference orbital phase (rad) */
     double* f_ref,                        /**< Reference frequency */
     double* inc,
     double* lam,
     double* beta,
     double* psi,
     double* tRef_wave_frame,
     double* tRef_sampling_frame,
     double tBase, int TDItag, int order_fresnel_stencil,
     int numModes,
     int length,
     int numBinAll
)
{

    __shared__ int ells[MAX_MODES];
    __shared__ int mms[MAX_MODES];

    for (int i = threadIdx.x; i < numModes; i += blockDim.x)
    {
        ells[i] = ells_in[i];
        mms[i] = mms_in[i];
    }

    __syncthreads();

    int binNum = blockIdx.x; // threadIdx.x + blockDim.x * blockIdx.x;

    if (binNum < numBinAll)
    {
        responseCore(phases, response_out, ells, mms, phases_deriv, freqs, phiRef[binNum], f_ref[binNum], inc[binNum], lam[binNum], beta[binNum], psi[binNum], tRef_wave_frame[binNum], tRef_sampling_frame[binNum], length, numModes, binNum, numBinAll,
        tBase, TDItag, order_fresnel_stencil);
    }
}


__device__
void prep_splines(int i, int length, int interp_i, int ninterps, int num_intermediates, double *b, double *ud, double *diag, double *ld, double *x, double *y, int numBinAll, int param, int nsub, int sub_i){
  double dx1, dx2, d, slope1, slope2;
  int ind0x, ind1x, ind2x, ind0y, ind1y, ind2y, ind_out;

  double xval0, xval1, xval2, yval1;

  int numFreqarrs = int(ninterps / num_intermediates);
  int freqArr_i = int(interp_i / num_intermediates);

  //if ((threadIdx.x == 10) && (blockIdx.x == 1)) printf("numFreqarrs %d %d %d %d %d\n", ninterps, interp_i, num_intermediates, numFreqarrs, freqArr_i);
  if (i == length - 1){
    ind0y = (param * nsub + sub_i) * length + (length - 3);
    ind1y = (param * nsub + sub_i) * length + (length - 2);
    ind2y = (param * nsub + sub_i) * length + (length - 1);

    ind0x = freqArr_i * length + (length - 3);
    ind1x = freqArr_i * length + (length - 2);
    ind2x = freqArr_i * length + (length - 1);

    ind_out = (param * nsub + sub_i) * length + (length - 1);

    xval0 = x[ind0x];
    xval1 = x[ind1x];
    xval2 = x[ind2x];

    dx1 = xval1 - xval0;
    dx2 = xval2 - xval1;
    d = xval2 - xval0;

    yval1 = y[ind1y];

    slope1 = (yval1 - y[ind0y])/dx1;
    slope2 = (y[ind2y] - yval1)/dx2;

    b[ind_out] = ((dx2*dx2*slope1 +
                             (2*d + dx2)*dx1*slope2) / d);
    diag[ind_out] = dx1;
    ld[ind_out] = d;
    ud[ind_out] = 0.0;

  } else if (i == 0){

      ind0y = (param * nsub + sub_i) * length + 0;
      ind1y = (param * nsub + sub_i) * length + 1;
      ind2y = (param * nsub + sub_i) * length + 2;

      ind0x = freqArr_i * length + 0;
      ind1x = freqArr_i * length + 1;
      ind2x = freqArr_i * length + 2;

      ind_out = (param * nsub + sub_i) * length + 0;

      xval0 = x[ind0x];
      xval1 = x[ind1x];
      xval2 = x[ind2x];


      dx1 = xval1 - xval0;
      dx2 = xval2 - xval1;
      d = xval2 - xval0;

      yval1 = y[ind1y];

      //amp
      slope1 = (yval1 - y[ind0y])/dx1;
      slope2 = (y[ind2y] - yval1)/dx2;

      b[ind_out] = ((dx1 + 2*d) * dx2 * slope1 +
                          dx1*dx1 * slope2) / d;
    ud[ind_out] = d;
    ld[ind_out] = 0.0;
      diag[ind_out] = dx2;

  } else{

      ind0y = (param * nsub + sub_i) * length + (i - 1);
      ind1y = (param * nsub + sub_i) * length + (i + 0);
      ind2y = (param * nsub + sub_i) * length + (i + 1);

      ind0x = freqArr_i * length + (i - 1);
      ind1x = freqArr_i * length + (i - 0);
      ind2x = freqArr_i * length + (i + 1);

      ind_out = (param * nsub + sub_i) * length + i;

      xval0 = x[ind0x];
      xval1 = x[ind1x];
      xval2 = x[ind2x];

    dx1 = xval1 - xval0;
    dx2 = xval2 - xval1;

    yval1 = y[ind1y];

    //amp
    slope1 = (yval1 - y[ind0y])/dx1;
    slope2 = (y[ind2y] - yval1)/dx2;

    b[ind_out] = 3.0* (dx2*slope1 + dx1*slope2);
    diag[ind_out] = 2*(dx1 + dx2);
    ud[ind_out] = dx1;
    ld[ind_out] = dx2;
  }

}



CUDA_KERNEL
void fill_B(double *freqs_arr, double *y_all, double *B, double *upper_diag, double *diag, double *lower_diag,
                      int ninterps, int length, int num_intermediates, int numModes, int numBinAll){

    int param = 0;
    int nsub = 0;
    int sub_i = 0;
    #ifdef __HIPCC__

    int start1 = blockIdx.x;
    int end1 = ninterps;
    int diff1 = gridDim.x;

    #else

    int start1 = 0;
    int end1 = ninterps;
    int diff1 = 1;

    #endif
    for (int interp_i = start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1){

         #ifdef __HIPCC__

         int start2 = threadIdx.x;
         int end2 = length;
         int diff2 = blockDim.x;

         #else

         int start2 = 0;
         int end2 = length;
         int diff2 = 1;

         #endif

        param = int((double) interp_i/(numModes * numBinAll));
        nsub = numModes * numBinAll;
        sub_i = interp_i % (numModes * numBinAll);

       for (int i = start2;
            i < end2;
            i += diff2){

            int lead_ind = interp_i*length;
            prep_splines(i, length, interp_i, ninterps, num_intermediates, B, upper_diag, diag, lower_diag, freqs_arr, y_all, numBinAll, param, nsub, sub_i);

}
}
}

/*
CuSparse error checking
*/
#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
                             fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
                             exit(-1);}} while(0)

#define CUSPARSE_CALL(X) ERR_NE((X),HIPSPARSE_STATUS_SUCCESS)

void interpolate_kern(int m, int n, double *a, double *b, double *c, double *d_in)
{
        #ifdef __HIPCC__
        size_t bufferSizeInBytes;

        hipsparseHandle_t handle;
        void *pBuffer;

        CUSPARSE_CALL(hipsparseCreate(&handle));
        CUSPARSE_CALL( hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m, a, b, c, d_in, n, m, &bufferSizeInBytes));
        gpuErrchk(hipMalloc(&pBuffer, bufferSizeInBytes));

        CUSPARSE_CALL(hipsparseDgtsv2StridedBatch(handle,
                                                  m,
                                                  a, // dl
                                                  b, //diag
                                                  c, // du
                                                  d_in,
                                                  n,
                                                  m,
                                                  pBuffer));

      CUSPARSE_CALL(hipsparseDestroy(handle));
      gpuErrchk(hipFree(pBuffer));

      #else

    #ifdef __USE_OMP__
    #pragma omp parallel for
    #endif
    for (int j = 0;
         j < n;
         j += 1){
           //fit_constants_serial(m, n, w, a, b, c, d_in, x_in, j);
           int info = LAPACKE_dgtsv(LAPACK_COL_MAJOR, m, 1, &a[j*m + 1], &b[j*m], &c[j*m], &d_in[j*m], m);
           //if (info != m) printf("lapack info check: %d\n", info);

       }

      #endif


    /*
    int interp_i = threadIdx.x + blockDim.x * blockIdx.x;

    int param = (int) (interp_i / (numModes * numBinAll));
    int nsub = numBinAll * numModes;
    int sub_i = interp_i % (numModes * numBinAll);

    int ind_i, ind_im1, ind_ip1;
    if (interp_i < ninterps)
    {

        double w = 0.0;
        for (int i = 1; i < n; i += 1)
        {
            ind_i = (param * n + i) * nsub + sub_i;
            ind_im1 = (param * n + (i-1)) * nsub + sub_i;


            w = a[ind_i]/b[ind_im1];
            b[ind_i] = b[ind_i] - w * c[ind_im1];
            d[ind_i] = d[ind_i] - w * d[ind_im1];
        }

        ind_i = (param * n + (n-1)) * nsub + sub_i;

        d[ind_i] = d[ind_i]/b[ind_i];
        for (int i = n - 2; i >= 0; i -= 1)
        {
            ind_i = (param * n + i) * nsub + sub_i;
            ind_ip1 = (param * n + (i+1)) * nsub + sub_i;

            d[ind_i] = (d[ind_i] - c[ind_i] * d[ind_ip1])/b[ind_i];

        }
    }
    */
}


CUDA_CALLABLE_MEMBER
void fill_coefficients(int i, int length, int sub_i, int nsub, double *dydx, double dx, double *y, double *coeff1, double *coeff2, double *coeff3, int param){
  double slope, t, dydx_i;

  int ind_i = (param * nsub + sub_i) * length + i;
  int ind_ip1 = (param * nsub + sub_i) * length + (i + 1);

  slope = (y[ind_ip1] - y[ind_i])/dx;

  dydx_i = dydx[ind_i];

  t = (dydx_i + dydx[ind_ip1] - 2*slope)/dx;

  coeff1[ind_i] = dydx_i;
  coeff2[ind_i] = (slope - dydx_i) / dx - t;
  coeff3[ind_i] = t/dx;

  //if ((param == 1) && (i == length - 3) && (sub_i == 0)) printf("freq check: %d %d %d %d %d\n", i, dydx[ind_i], dydx[ind_ip1]);


}

CUDA_KERNEL
void set_spline_constants(double *f_arr, double* y, double *c1, double* c2, double* c3, double *B,
                      int ninterps, int length, int num_intermediates, int numBinAll, int numModes){

    double df;
    #ifdef __HIPCC__
    int start1 = blockIdx.x;
    int end1 = ninterps;
    int diff1 = gridDim.x;
    #else

    int start1 = 0;
    int end1 = ninterps;
    int diff1 = 1;

    #endif

    for (int interp_i= start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1){

     int numFreqarrs = int(ninterps / num_intermediates);
     int freqArr_i = int(interp_i / num_intermediates);

     int param = (int) (interp_i / (numModes * numBinAll));
     int nsub = numBinAll * numModes;
     int sub_i = interp_i % (numModes * numBinAll);

     #ifdef __HIPCC__
     int start2 = threadIdx.x;
     int end2 = length - 1;
     int diff2 = blockDim.x;
     #else

     int start2 = 0;
     int end2 = length - 1;
     int diff2 = 1;

     #endif
     for (int i = start2;
            i < end2;
            i += diff2){

                // TODO: check if there is faster way to do this
              df = f_arr[freqArr_i * length + (i + 1)] - f_arr[freqArr_i * length + i];

              int lead_ind = interp_i*length;
              fill_coefficients(i, length, sub_i, nsub, B, df,
                                y,
                                c1,
                                c2,
                                c3, param);

}
}
}


__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long* address_as_ull =
                              (unsigned long long*)address;
    unsigned long long old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ void atomicAddComplex(cmplx* a, cmplx b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAddDouble(x, b.real());
  atomicAddDouble(y, b.imag());
}



#define  DATA_BLOCK 128
#define  NUM_INTERPS 9

__device__
cmplx get_ampphasefactor(double amp, double phase, double phaseShift){
    return amp*gcmplx::exp(cmplx(0.0, phase + phaseShift));
}

__device__
cmplx combine_information(cmplx* channel1, cmplx* channel2, cmplx* channel3, double amp, double phase, double tf, cmplx transferL1, cmplx transferL2, cmplx transferL3, double t_start, double t_end)
{
    // TODO: make sure the end of the ringdown is included
    if ((tf >= t_start) && ((tf <= t_end) || (t_end <= 0.0)) && (amp > 1e-40))
    {
        cmplx amp_phase_term = amp*gcmplx::exp(cmplx(0.0, -phase));  // add phase shift

        *channel1 = gcmplx::conj(transferL1 * amp_phase_term);
        *channel2 = gcmplx::conj(transferL2 * amp_phase_term);
        *channel3 = gcmplx::conj(transferL3 * amp_phase_term);

    }
}

#define  NUM_TERMS 4

#define  MAX_NUM_COEFF_TERMS 1200

CUDA_KERNEL
void TDI(cmplx* templateChannels, double* dataFreqsIn, double dlog10f, double* freqsOld, double* propArrays, double* c1In, double* c2In, double* c3In, double t_mrg, int old_length, int data_length, int numBinAll, int numModes, double t_obs_start, double t_obs_end, int* inds, int ind_start, int ind_length, int bin_i)
{

    int num_params = 9;
    //int mode_i = blockIdx.y;

    int numAll = numBinAll * numModes * old_length;

    double tempLike, addLike, time_check, phaseShift;
    cmplx trans_complex1, trans_complex2, trans_complex3, ampphasefactor;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= ind_length) return;

    double f = dataFreqsIn[i + ind_start];

    int ind_here = inds[i];

    double f_old = freqsOld[ind_here];

    double x = f - f_old;
    double x2 = x * x;
    double x3 = x * x2;

    trans_complex1 = 0.0; trans_complex2 = 0.0; trans_complex3 = 0.0;

    for (int mode_i = 0; mode_i < numModes; mode_i += 1)
    {
        int int_shared = ((0 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double amp = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        //if ((i == 100) || (i == 101)) printf("%d %d %d %e %e %e %e %e %e\n", window_i, mode_i, i, amp, f, f_old, y[int_shared], c1[int_shared], c2[int_shared]);

        int_shared = ((1 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double phase = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((2 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double tf = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((3 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL1_re = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((4 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL1_im = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((5 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL2_re = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((6 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL2_im = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((7 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL3_re = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((8 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL3_im = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        cmplx channel1(0.0, 0.0);
        cmplx channel2(0.0, 0.0);
        cmplx channel3(0.0, 0.0);

        combine_information(&channel1, &channel2, &channel3, amp, phase, tf, cmplx(transferL1_re, transferL1_im), cmplx(transferL2_re, transferL2_im), cmplx(transferL3_re, transferL3_im), t_obs_start, t_obs_end);

        trans_complex1 += channel1;
        trans_complex2 += channel2;
        trans_complex3 += channel3;
    }

    atomicAddComplex(&templateChannels[0 * ind_length + i], trans_complex1);
    atomicAddComplex(&templateChannels[1 * ind_length + i], trans_complex2);
    atomicAddComplex(&templateChannels[2 * ind_length + i], trans_complex3);


    /*
    __shared__ double y[MAX_NUM_COEFF_TERMS];
    __shared__ double c1[MAX_NUM_COEFF_TERMS];
    __shared__ double c2[MAX_NUM_COEFF_TERMS];
    __shared__ double c3[MAX_NUM_COEFF_TERMS];
    __shared__ double freqs_shared[MAX_NUM_COEFF_TERMS];

    int num_params = 9;
    int mode_i = blockIdx.y;

    int numAll = numBinAll * numModes * old_length;

    double amp, phase, tfCorr, transferL1_re, transferL1_im, transferL2_re, transferL2_im, transferL3_re, transferL3_im;
    double x, x2, x3, tempLike, addLike, time_check, phaseShift;
    cmplx trans_complex1, trans_complex2, trans_complex3, ampphasefactor;

    __shared__ int start_ind, end_ind;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    bool run = true;
    if (i >= ind_length) run = false;

    __syncthreads();

    if (threadIdx.x == 0)
    {
        start_ind = inds[i];
    }
    int max_thread_num = (ind_length - blockDim.x*blockIdx.x > NUM_THREADS3) ? NUM_THREADS3 : ind_length - blockDim.x*blockIdx.x;

    if (threadIdx.x == max_thread_num - 1)
    {
        end_ind = inds[i];
    }

    __syncthreads();

    int num_windows = end_ind - start_ind + 1;

    int nsub = numModes * numBinAll;

    //if (run) printf("%d %d %d %d %d %d %d %d\n", max_thread_num, threadIdx.x, blockDim.x, NUM_THREADS3, i, ind_length, start_ind, end_ind);

    for (int j = threadIdx.x; j < num_windows; j += blockDim.x)
    {
        int window_i = j;

        int old_ind = start_ind + window_i;

        if ((old_ind < 0) || (old_ind >= old_length))
        {
            continue;
        }

        freqs_shared[window_i] = freqsOld[old_ind];

        //if ((blockIdx.x == 0) && (blockIdx.y == 0)) printf("%d %d %e %e\n", old_ind, window_i, freqs_shared[window_i], freqsOld[old_ind]);
    }

    __syncthreads();

    for (int j = threadIdx.x; j < num_params * num_windows; j += blockDim.x)
    {
        int window_i = j % num_windows;
        int param_i = (int) (j / num_windows);

        int old_ind = start_ind + window_i;

        if ((old_ind < 0) || (old_ind >= old_length))
        {
            continue;
        }

        int ind = ((param_i * numBinAll + bin_i) * numModes + mode_i) * old_length + old_ind;
        int ind_shared = window_i * num_params + param_i;

        y[ind_shared] = propArrays[ind];
        c1[ind_shared] = c1In[ind];
        c2[ind_shared] = c2In[ind];
        c3[ind_shared] = c3In[ind];

        if (ind_shared > MAX_NUM_COEFF_TERMS) printf("BAD %d %d\n", ind_shared, window_i);
    }

    __syncthreads();

    if (run)
    {
        double f = dataFreqsIn[i + ind_start];

        int ind_here = inds[i];

        int window_i = ind_here - start_ind;

        double f_old = freqs_shared[window_i];

        double x = f - f_old;
        double x2 = x * x;
        double x3 = x * x2;

        int int_shared = window_i * num_params + 0;
        double amp = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        //if ((i == 100) || (i == 101)) printf("%d %d %d %e %e %e %e %e %e\n", window_i, mode_i, i, amp, f, f_old, y[int_shared], c1[int_shared], c2[int_shared]);

        int_shared = window_i * num_params + 1;
        double phase = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 2;
        double tf = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 3;
        double transferL1_re = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 4;
        double transferL1_im = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 5;
        double transferL2_re = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 6;
        double transferL2_im = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 7;
        double transferL3_re = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 8;
        double transferL3_im = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        cmplx channel1(0.0, 0.0);
        cmplx channel2(0.0, 0.0);
        cmplx channel3(0.0, 0.0);

        combine_information(&channel1, &channel2, &channel3, amp, phase, tf, cmplx(transferL1_re, transferL1_im), cmplx(transferL2_re, transferL2_im), cmplx(transferL3_re, transferL3_im), t_obs_start, t_obs_end);

        atomicAddComplex(&templateChannels[0 * ind_length + i], channel1);
        atomicAddComplex(&templateChannels[1 * ind_length + i], channel2);
        atomicAddComplex(&templateChannels[2 * ind_length + i], channel3);

    }
    */
}

CUDA_KERNEL
void fill_waveform(cmplx* templateChannels,
                double* bbh_buffer,
                int numBinAll, int data_length, int nChannels, int numModes, double* t_start, double* t_end)
{

    cmplx I(0.0, 1.0);

    cmplx temp_channel1 = 0.0, temp_channel2 = 0.0, temp_channel3 = 0.0;
    for (int bin_i = blockIdx.x; bin_i < numBinAll; bin_i += gridDim.x)
    {

        double t_start_bin = t_start[bin_i];
        double t_end_bin = t_end[bin_i];

        for (int i = threadIdx.x; i < data_length; i += blockDim.x)
        {
            cmplx temp_channel1 = 0.0, temp_channel2 = 0.0, temp_channel3 = 0.0;
            for (int mode_i = 0; mode_i < numModes; mode_i += 1)
            {

                int ind = ((0 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double amp = bbh_buffer[ind];

                ind = ((1 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double phase = bbh_buffer[ind];

                ind = ((2 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double tf = bbh_buffer[ind];

                ind = ((3 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL1_re = bbh_buffer[ind];

                ind = ((4 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL1_im = bbh_buffer[ind];

                ind = ((5 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL2_re = bbh_buffer[ind];

                ind = ((6 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL2_im = bbh_buffer[ind];

                ind = ((7 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL3_re = bbh_buffer[ind];

                ind = ((8 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL3_im = bbh_buffer[ind];

                cmplx channel1 = 0.0 + 0.0 * I;
                cmplx channel2 = 0.0 + 0.0 * I;
                cmplx channel3 = 0.0 + 0.0 * I;

                combine_information(&channel1, &channel2, &channel3, amp, phase, tf, cmplx(transferL1_re, transferL1_im), cmplx(transferL2_re, transferL2_im), cmplx(transferL3_re, transferL3_im), t_start_bin, t_end_bin);

                temp_channel1 += channel1;
                temp_channel2 += channel2;
                temp_channel3 += channel3;
            }

            templateChannels[(bin_i * 3 + 0) * data_length + i] = temp_channel1;
            templateChannels[(bin_i * 3 + 1) * data_length + i] = temp_channel2;
            templateChannels[(bin_i * 3 + 2) * data_length + i] = temp_channel3;

        }
    }
}

void direct_sum(cmplx* templateChannels,
                double* bbh_buffer,
                int numBinAll, int data_length, int nChannels, int numModes, double* t_start, double* t_end)
{

    int nblocks5 = numBinAll; // std::ceil((numBinAll + NUM_THREADS4 -1)/NUM_THREADS4);

    fill_waveform<<<nblocks5, NUM_THREADS4>>>(templateChannels, bbh_buffer, numBinAll, data_length, nChannels, numModes, t_start, t_end);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
}


void InterpTDI(long* templateChannels_ptrs, double* dataFreqs, double dlog10f, double* freqs, double* propArrays, double* c1, double* c2, double* c3, double* t_mrg_in, double* t_start_in, double* t_end_in, int length, int data_length, int numBinAll, int numModes, double t_obs_start, double t_obs_end, long* inds_ptrs, int* inds_start, int* ind_lengths)
{

    hipStream_t streams[numBinAll];

    #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        int length_bin_i = ind_lengths[bin_i];
        int ind_start = inds_start[bin_i];
        int* inds = (int*) inds_ptrs[bin_i];

        double t_mrg = t_mrg_in[bin_i];
        double t_start = t_start_in[bin_i];
        double t_end = t_end_in[bin_i];

        cmplx* templateChannels = (cmplx*) templateChannels_ptrs[bin_i];

        int nblocks3 = std::ceil((length_bin_i + NUM_THREADS3 -1)/NUM_THREADS3);
        hipStreamCreate(&streams[bin_i]);

        dim3 gridDim(nblocks3, 1);
        TDI<<<gridDim, NUM_THREADS3, 0, streams[bin_i]>>>(templateChannels, dataFreqs, dlog10f, freqs, propArrays, c1, c2, c3, t_mrg, length, data_length, numBinAll, numModes, t_start, t_end, inds, ind_start, length_bin_i, bin_i);

    }

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        //destroy the streams
        hipStreamDestroy(streams[bin_i]);
    }
}

#define  DATA_BLOCK2 512
CUDA_KERNEL
void hdynLikelihood(cmplx* likeOut1, cmplx* likeOut2,
                    cmplx* templateChannels, cmplx* dataConstants,
                    double* dataFreqsIn,
                    int numBinAll, int data_length, int nChannels)
{
    __shared__ cmplx A0temp[DATA_BLOCK2];
    __shared__ cmplx A1temp[DATA_BLOCK2];
    __shared__ cmplx B0temp[DATA_BLOCK2];
    __shared__ cmplx B1temp[DATA_BLOCK2];
    __shared__ double dataFreqs[DATA_BLOCK2];

    cmplx A0, A1, B0, B1;

    cmplx trans_complex(0.0, 0.0);
    cmplx prev_trans_complex(0.0, 0.0);
    double prevFreq = 0.0;
    double freq = 0.0;

    int currentStart = 0;

    cmplx r0, r1, r1Conj, tempLike1, tempLike2;
    double mag_r0, midFreq;

    int binNum = threadIdx.x + blockDim.x * blockIdx.x;

    if (true) // for (int binNum = threadIdx.x + blockDim.x * blockIdx.x; binNum < numBinAll; binNum += blockDim.x * gridDim.x)
    {
        tempLike1 = 0.0;
        tempLike2 = 0.0;
        for (int channel = 0; channel < nChannels; channel += 1)
        {
            prevFreq = 0.0;
            currentStart = 0;
            while (currentStart < data_length)
            {
                __syncthreads();
                for (int jj = threadIdx.x; jj < DATA_BLOCK2; jj += blockDim.x)
                {
                    if ((jj + currentStart) >= data_length) continue;
                    A0temp[jj] = dataConstants[(0 * nChannels + channel) * data_length + currentStart + jj];
                    A1temp[jj] = dataConstants[(1 * nChannels + channel) * data_length + currentStart + jj];
                    B0temp[jj] = dataConstants[(2 * nChannels + channel) * data_length + currentStart + jj];
                    B1temp[jj] = dataConstants[(3 * nChannels + channel) * data_length + currentStart + jj];

                    dataFreqs[jj] = dataFreqsIn[currentStart + jj];

                    //if ((jj + currentStart < 3) && (binNum == 0) & (channel == 0))
                    //    printf("check %e %e, %e %e, %e %e, %e %e, %e \n", A0temp[jj], A1temp[jj], B0temp[jj], B1temp[jj], dataFreqs[jj]);

                }
                __syncthreads();
                if (binNum < numBinAll)
                {
                    for (int jj = 0; jj < DATA_BLOCK2; jj += 1)
                    {
                        if ((jj + currentStart) >= data_length) continue;
                        freq = dataFreqs[jj];
                        trans_complex = templateChannels[((jj + currentStart) * nChannels + channel) * numBinAll + binNum];

                        if ((prevFreq != 0.0) && (jj + currentStart > 0))
                        {
                            A0 = A0temp[jj]; // constants will need to be aligned with 1..n-1 because there are data_length - 1 bins
                            A1 = A1temp[jj];
                            B0 = B0temp[jj];
                            B1 = B1temp[jj];

                            r1 = (trans_complex - prev_trans_complex)/(freq - prevFreq);
                            midFreq = (freq + prevFreq)/2.0;

                            r0 = trans_complex - r1 * (freq - midFreq);

                            //if (((binNum == 767) || (binNum == 768)) & (channel == 0))
                            //    printf("CHECK2: %d %d %d %e %e\n", jj + currentStart, binNum, jj, A0); // , %e %e, %e %e, %e %e, %e %e,  %e %e,  %e %e , %e\n", ind, binNum, jj + currentStart, A0, A1, B0, B1, freq, prevFreq, trans_complex, prev_trans_complex, midFreq);

                            r1Conj = gcmplx::conj(r1);

                            tempLike1 += A0 * gcmplx::conj(r0) + A1 * r1Conj;

                            mag_r0 = gcmplx::abs(r0);
                            tempLike2 += B0 * (mag_r0 * mag_r0) + 2. * B1 * gcmplx::real(r0 * r1Conj);
                        }

                        prev_trans_complex = trans_complex;
                        prevFreq = freq;
                    }
                }
                currentStart += DATA_BLOCK2;
            }
        }
        if (binNum < numBinAll)
        {
            likeOut1[binNum] = tempLike1;
            likeOut2[binNum] = tempLike2;
        }
    }
}





void LISA_response(
    double* response_out,
    int* ells_in,
    int* mms_in,
    double* freqs,               /**< Frequency points at which to evaluate the waveform (Hz) */
    double* phiRef,                 /**< reference orbital phase (rad) */
    double* f_ref,                        /**< Reference frequency */
    double* inc,
    double* lam,
    double* beta,
    double* psi,
    double* tRef_wave_frame,
    double* tRef_sampling_frame,
    double tBase, int TDItag, int order_fresnel_stencil,
    int numModes,
    int length,
    int numBinAll,
    int includesAmps
)
{

    int start_param = includesAmps;  // if it has amps, start_param is 1, else 0

    double* phases = &response_out[start_param * numBinAll * numModes * length];
    double* phases_deriv = &response_out[(start_param + 1) * numBinAll * numModes * length];
    double* response_vals = &response_out[(start_param + 2) * numBinAll * numModes * length];

    int nblocks2 = numBinAll; //std::ceil((numBinAll + NUM_THREADS2 -1)/NUM_THREADS2);

    response<<<nblocks2, NUM_THREADS2>>>(
        phases,
        response_vals,
        phases_deriv,
        ells_in,
        mms_in,
        freqs,               /**< Frequency points at which to evaluate the waveform (Hz) */
        phiRef,                 /**< reference orbital phase (rad) */
        f_ref,                        /**< Reference frequency */
        inc,
        lam,
        beta,
        psi,
        tRef_wave_frame,
        tRef_sampling_frame,
        tBase, TDItag, order_fresnel_stencil,
        numModes,
        length,
        numBinAll
   );
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
}


void interpolate(double* freqs, double* propArrays,
                 double* B, double* upper_diag, double* diag, double* lower_diag,
                 int length, int numInterpParams, int numModes, int numBinAll)
{

    int num_intermediates = numModes * numInterpParams;
    int ninterps = numModes * numInterpParams * numBinAll;

    int nblocks = std::ceil((ninterps + NUM_THREADS -1)/NUM_THREADS);

    double* c1 = upper_diag; //&interp_array[0 * numInterpParams * amp_phase_size];
    double* c2 = diag; //&interp_array[1 * numInterpParams * amp_phase_size];
    double* c3 = lower_diag; //&interp_array[2 * numInterpParams * amp_phase_size];

    //printf("%d after response, %d\n", jj, nblocks2);

     fill_B<<<nblocks, NUM_THREADS>>>(freqs, propArrays, B, upper_diag, diag, lower_diag, ninterps, length, num_intermediates, numModes, numBinAll);
     hipDeviceSynchronize();
     gpuErrchk(hipGetLastError());

     //printf("%d after fill b\n", jj);
     interpolate_kern(length, ninterps, lower_diag, diag, upper_diag, B);


  set_spline_constants<<<nblocks, NUM_THREADS>>>(freqs, propArrays, c1, c2, c3, B,
                    ninterps, length, num_intermediates, numBinAll, numModes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    //printf("%d after set spline\n", jj);
}

void hdyn(cmplx* likeOut1, cmplx* likeOut2,
                    cmplx* templateChannels, cmplx* dataConstants,
                    double* dataFreqs,
                    int numBinAll, int data_length, int nChannels)
{

    int nblocks4 = std::ceil((numBinAll + NUM_THREADS4 -1)/NUM_THREADS4);

    hdynLikelihood<<<nblocks4, NUM_THREADS4>>>(likeOut1, likeOut2, templateChannels, dataConstants, dataFreqs, numBinAll, data_length, nChannels);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
}

CUDA_KERNEL
void noiseweight_template(cmplx* templateChannels, double* noise_weight_times_df, int ind_start, int length, int data_stream_length)
{
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < length; i += gridDim.x * blockDim.x)
    {
        for (int j = 0; j < 3; j+= 1)
        {
            templateChannels[j * length + i] = templateChannels[j * length + i] * noise_weight_times_df[j * data_stream_length + ind_start + i];
        }
    }
}

#define NUM_THREADS_LIKE 256

void direct_like(double* d_h, double* h_h, cmplx* dataChannels, double* noise_weight_times_df, long* templateChannels_ptrs, int* inds_start, int* ind_lengths, int data_stream_length, int numBinAll)
{

    hipStream_t streams[numBinAll];
    hipblasHandle_t handle;

    hipDoubleComplex result_d_h[numBinAll];
    hipDoubleComplex result_h_h[numBinAll];

    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
      printf ("CUBLAS initialization failed\n");
      exit(0);
    }

    #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        int length_bin_i = ind_lengths[bin_i];
        int ind_start = inds_start[bin_i];

        cmplx* templateChannels = (cmplx*) templateChannels_ptrs[bin_i];

        int nblocks = std::ceil((length_bin_i + NUM_THREADS_LIKE -1)/NUM_THREADS_LIKE);
        hipStreamCreate(&streams[bin_i]);

        noiseweight_template<<<nblocks, NUM_THREADS_LIKE, 0, streams[bin_i]>>>(templateChannels, noise_weight_times_df, ind_start, length_bin_i, data_stream_length);
        hipStreamSynchronize(streams[bin_i]);

        for (int j = 0; j < 3; j += 1)
        {

            hipblasSetStream(handle, streams[bin_i]);
            stat = hipblasZdotc(handle, length_bin_i,
                              (hipDoubleComplex*)&dataChannels[j * data_stream_length + ind_start], 1,
                              (hipDoubleComplex*)&templateChannels[j * length_bin_i], 1,
                              &result_d_h[bin_i]);
            hipStreamSynchronize(streams[bin_i]);
            if (stat != HIPBLAS_STATUS_SUCCESS)
            {
                exit(0);
            }

            d_h[bin_i] += 4.0 * hipCreal(result_d_h[bin_i]);

            hipblasSetStream(handle, streams[bin_i]);
            stat = hipblasZdotc(handle, length_bin_i,
                              (hipDoubleComplex*)&templateChannels[j * length_bin_i], 1,
                              (hipDoubleComplex*)&templateChannels[j * length_bin_i], 1,
                              &result_h_h[bin_i]);
            hipStreamSynchronize(streams[bin_i]);
            if (stat != HIPBLAS_STATUS_SUCCESS)
            {
                exit(0);
            }
            h_h[bin_i] += 4.0 * hipCreal(result_h_h[bin_i]);

        }
    }

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        //destroy the streams
        hipStreamDestroy(streams[bin_i]);
    }
    hipblasDestroy(handle);
}
/*
int main()
{

    int TDItag = 1;
    int order_fresnel_stencil = 0;
    double tBase = 1.0;

    int numBinAll = 5000;
    int numModes = 6;
    int length = 1024;
    int data_length = 4096;

    int *ells_in, *mms_in;

    gpuErrchk(hipMallocManaged(&ells_in, numModes * sizeof(int)));
    gpuErrchk(hipMallocManaged(&mms_in, numModes * sizeof(int)));

    ells_in[0] = 2;
    ells_in[1] = 3;
    ells_in[2] = 4;

    ells_in[3] = 2;
    ells_in[4] = 3;
    ells_in[5] = 4;

    mms_in[0] = 2;
    mms_in[1] = 3;
    mms_in[2] = 4;

    mms_in[3] = 1;
    mms_in[4] = 2;
    mms_in[5] = 3;

    double *amps, *phases, *phases_deriv, *freqs, *m1_SI, *m2_SI, *chi1z, *chi2z, *distance, *phiRef, *fRef;
    double *inc, *lam, *beta, *psi, *tRef_wave_frame, *tRef_sampling_frame;
    double *response_out;
    double *B, *interp_array; // plays roll of upper lower diag, and then coefficients 1, 2, 3

    size_t amp_phase_size = numBinAll * numModes * length *sizeof(double);
    size_t freqs_size = numBinAll * length * sizeof(double);
    size_t bin_size = numBinAll * sizeof(double);

    int numInterpParams = 9;

    gpuErrchk(hipMallocManaged(&amps, numInterpParams * amp_phase_size));

    response_out = &amps[1 * numBinAll * numModes * length];

    double *upper_diag, *diag, *lower_diag;
    gpuErrchk(hipMallocManaged(&B, numInterpParams * amp_phase_size));
    gpuErrchk(hipMallocManaged(&upper_diag, numInterpParams * amp_phase_size));
    gpuErrchk(hipMallocManaged(&diag, numInterpParams * amp_phase_size));
    gpuErrchk(hipMallocManaged(&lower_diag, numInterpParams * amp_phase_size));

    //double* upper_diag = &interp_array[0 * numInterpParams * amp_phase_size];
    //double* diag = &interp_array[1 * numInterpParams * amp_phase_size];
    //double* lower_diag = &interp_array[2 * numInterpParams * amp_phase_size];

    double* propArrays = amps;

    gpuErrchk(hipMallocManaged(&freqs, freqs_size));

    gpuErrchk(hipMallocManaged(&m1_SI, bin_size));
    gpuErrchk(hipMallocManaged(&m2_SI, bin_size));
    gpuErrchk(hipMallocManaged(&chi1z, bin_size));
    gpuErrchk(hipMallocManaged(&chi2z, bin_size));
    gpuErrchk(hipMallocManaged(&distance, bin_size));
    gpuErrchk(hipMallocManaged(&phiRef, bin_size));
    gpuErrchk(hipMallocManaged(&fRef, bin_size));

    gpuErrchk(hipMallocManaged(&inc, bin_size));
    gpuErrchk(hipMallocManaged(&lam, bin_size));
    gpuErrchk(hipMallocManaged(&beta, bin_size));
    gpuErrchk(hipMallocManaged(&psi, bin_size));
    gpuErrchk(hipMallocManaged(&tRef_wave_frame, bin_size));
    gpuErrchk(hipMallocManaged(&tRef_sampling_frame, bin_size));

    double m1 = 2e6; // solar
    double m2 = 1e6;
    double a1 = 0.8;
    double a2 = 0.8;
    double dist = 30.0; // Gpc
    double phi_ref = 0.0;
    double f_ref = 0.0;
    double inc_in = PI/3.;
    double lam_in = 0.4;
    double beta_in = 0.24;
    double psi_in = 1.0;
    double tRef_wave_frame_in = 10.0;
    double tRef_sampling_frame_in = 50.0;

    double Msec = (m1 + m2) * MTSUN_SI;

    double log10f_start = log10(1e-4/Msec);
    double log10f_end = log10(0.6/Msec);

    double dlog10f = (log10f_end - log10f_start)/(length - 1);

    std::default_random_engine generator;
    std::uniform_real_distribution<double> distribution(0.0,1.0);

    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        m1_SI[bin_i] = (1e6 * MSUN_SI) * (1 + distribution(generator));
        m2_SI[bin_i] = (4e5 * MSUN_SI) * (1 + distribution(generator));

        chi1z[bin_i] = (distribution(generator))* 0.9;
        chi2z[bin_i] = (distribution(generator))* 0.9;

        distance[bin_i] = (35) * (1 + distribution(generator)) * 1e9 * PC_SI;
        phiRef[bin_i] = (1 + distribution(generator));
        fRef[bin_i] = f_ref;

        inc[bin_i] = (distribution(generator)) * 0.25 + 0.25;
        lam[bin_i] = (distribution(generator)) * 0.25 + 0.25;
        beta[bin_i] = (distribution(generator)) * 0.25 + 0.25;
        psi[bin_i] = (distribution(generator)) * 0.25 + 0.25;
        tRef_wave_frame[bin_i] = (1 + distribution(generator)) * 20.0;
        tRef_sampling_frame[bin_i] = (1 + distribution(generator)) * 20.0;

        for (int i = 0; i < length; i += 1)
        {
            freqs[i * numBinAll + bin_i] = pow(10.0, log10f_start + i * dlog10f);
        }
    }

    cmplx *dataChannels, *templateChannels, *dataConstants;
    double *dataFreqs;
    int nChannels = 3;

    double t_obs_start = 1.0;
    double t_obs_end = 0.0;

    gpuErrchk(hipMallocManaged(&dataChannels, nChannels * data_length * sizeof(cmplx)));
    gpuErrchk(hipMallocManaged(&dataConstants, NUM_TERMS * nChannels * data_length * sizeof(cmplx)));
    gpuErrchk(hipMallocManaged(&templateChannels, numBinAll * nChannels * data_length * sizeof(cmplx)));
    gpuErrchk(hipMallocManaged(&dataFreqs, data_length * sizeof(double)));

    double dlog10fData = (log10f_end - log10f_start)/(data_length - 1);

    for (int i = 0; i < data_length; i += 1)
    {
        dataFreqs[i] = pow(10.0, log10f_start + i * dlog10fData);

        for (int channel = 0; channel < nChannels; channel += 1)
        {
            dataChannels[channel * data_length + i] = cmplx(1.0, 1.0);

            for (int constant = 0; constant < NUM_TERMS; constant += 1)
            {
                dataConstants[(constant * nChannels + channel) * data_length + i] = cmplx(1.0, 1.0);
            }
        }
    }

    cmplx *likeOut1;
    gpuErrchk(hipMallocManaged(&likeOut1, numBinAll * sizeof(cmplx)));

    cmplx *likeOut2;
    gpuErrchk(hipMallocManaged(&likeOut2, numBinAll * sizeof(cmplx)));

    double *c1, *c2, *c3;
    int numIter = 10;

    for (int jj = 0; jj < numIter; jj += 1)
    {

        //printf("%d begin\n", jj);
        waveform_amp_phase(
        amps, ///**< [out] Frequency-domain waveform hx
        ells_in,
        mms_in,
        freqs,               ///**< Frequency points at which to evaluate the waveform (Hz)
        m1_SI,                       // /**< mass of companion 1 (kg)
        m2_SI,                        ///**< mass of companion 2 (kg)
        chi1z,                        ///**< z-component of the dimensionless spin of object 1 w.r.t. Lhat = (0,0,1)
        chi2z,                        ///**< z-component of the dimensionless spin of object 2 w.r.t. Lhat = (0,0,1)
        distance,               ///**< distance of source (m)
        phiRef,                 ///**< reference orbital phase (rad)
        fRef,                      //  /**< Reference frequency
        numModes,
        length,
        numBinAll
   );

   int includesAmps = 0;
   LISA_response(
       response_out,
       ells_in,
       mms_in,
       freqs,               ///**< Frequency points at which to evaluate the waveform (Hz)
       phiRef,                // /**< reference orbital phase (rad)
       fRef,                    //    /**< Reference frequency
       inc,
       lam,
       beta,
       psi,
       tRef_wave_frame,
       tRef_sampling_frame,
       tBase, TDItag, order_fresnel_stencil,
       numModes,
       length,
       numBinAll,
       includesAmps
  );

  interpolate(freqs, propArrays,
                   B, upper_diag, diag, lower_diag,
                 length, numInterpParams, numModes, numBinAll);

    //printf("%d middle\n", jj);

    c1 = upper_diag; //&interp_array[0 * numInterpParams * amp_phase_size];
    c2 = diag; //&interp_array[1 * numInterpParams * amp_phase_size];
    c3 = lower_diag; //&interp_array[2 * numInterpParams * amp_phase_size];


    InterpTDI(templateChannels, dataChannels, dataFreqs, freqs, propArrays, c1, c2, c3, tBase, tRef_sampling_frame, tRef_wave_frame, length, data_length,   numBinAll, numModes, t_obs_start, t_obs_end);

    hdyn(likeOut1, likeOut2, templateChannels, dataConstants, dataFreqs, numBinAll, data_length, nChannels);
    }

    int binNum = 1000;
    int mode_i = 0;
    for (int i = 0; i < 5; i += 1) printf("%d %e %e\n", i, c1[(i * numModes + 0) * numBinAll + 0], c2[(i * numModes + 0) * numBinAll + 0]);

    return 0;
}

*/

/*
__device__
void fill_coefficients(int i, int length, int mode_i, int numModes, int interp_i, int ninterps, double *dydx, double dx, double *y, double *coeff1, double *coeff2, double *coeff3){
  double slope, t, dydx_i;

  int indip1 = ((i + 1) * numModes + mode_i) * ninterps + interp_i;
  int indi = ((i) * numModes + mode_i) * ninterps + interp_i;

  slope = (y[indip1] - y[indi])/dx;

  dydx_i = dydx[indi];

  t = (dydx_i + dydx[indip1] - 2*slope)/dx;

  coeff1[indi] = dydx_i;
  coeff2[indi] = (slope - dydx_i) / dx - t;
  coeff3[indi] = t/dx;
}




__device__
void prep_splines(int i, int length, int mode_i, int numModes, int interp_i, int ninterps,  double *b, double *ud, double *diag, double *ld, double *x, double *y){
  double dx1, dx2, d, slope1, slope2;
  int ind1x, ind2x, ind3x, ind1y, ind2y, ind3y;
  if (i == length - 1){

     ind1x = (length - 2) * ninterps + interp_i;
     ind2x = (length - 3) * ninterps + interp_i;
     ind3x = (length - 1) * ninterps + interp_i;

     ind1y = ((length - 2) * numModes + mode_i) * ninterps + interp_i;
     ind2y = ((length - 3) * numModes + mode_i) * ninterps + interp_i;
     ind3y = ((length - 1) * numModes + mode_i) * ninterps + interp_i;


  } else if (i == 0){

      ind1x = 1 * ninterps + interp_i;
      ind2x = 0 * ninterps + interp_i;
      ind3x = 2 * ninterps + interp_i;

      ind1y = (1 * numModes + mode_i) * ninterps + interp_i;
      ind2y = (0 * numModes + mode_i) * ninterps + interp_i;
      ind3y = (2 * numModes + mode_i) * ninterps + interp_i;


  } else{

      ind1x = (i) * ninterps + interp_i;
      ind2x = (i-1) * ninterps + interp_i;
      ind3x = (i+1) * ninterps + interp_i;

      ind1y = ((i) * numModes + mode_i) * ninterps + interp_i;
      ind2y = ((i-1) * numModes + mode_i) * ninterps + interp_i;
      ind3y = ((i+1) * numModes + mode_i) * ninterps + interp_i;
  }

    dx1 = x[ind1x] - x[ind2x];
    dx2 = x[ind3x] - x[ind1x];

    //amp
    slope1 = (y[ind1y] - y[ind2y])/dx1;
    slope2 = (y[ind3y] - y[ind1y])/dx2;

    b[ind1y] = 3.0* (dx2*slope1 + dx1*slope2);
    diag[ind1y] = 2*(dx1 + dx2);
    ud[ind1y] = dx1;
    ld[ind1y] = dx2;
}



CUDA_KERNEL
void fill_B(double *x_arr, double *y_all, double *B, double *upper_diag, double *diag, double *lower_diag,
                      int ninterps, int length, int numModes){


    int start1 = blockIdx.x*blockDim.x + threadIdx.x;
    int end1 = ninterps;
    int diff1 = blockDim.x*gridDim.x;

    for (int interp_i= start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1)
        {

       for (int mode_i = 0; mode_i < numModes; mode_i += 1)
       {
           for (int i = start2;
                i < end2;
                i += diff2)
                {
                    prep_splines(i, length, mode_i, numModes, interp_i, ninterps,  B, upper_diag, diag, lower_diag, x_arr, y_all);

                }
       }

    }
}



CUDA_KERNEL
void set_spline_constants(double *x_arr, double *interp_array, double *B,
                      int ninterps, int length, int numModes){

    double dx;
    InterpContainer mode_vals;

    int start1 = blockIdx.x*blockDim.x + threadIdx.x;
    int end1 = ninterps;
    int diff1 = blockDim.x*gridDim.x;

    int npts = ninterps * length * numModes;

    for (int interp_i= start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1){

             for (int mode_i = 0; mode_i < numModes; mode_i += 1)
             {
                 for (int i = start2;
                      i < end2;
                      i += diff2)
                      {
                          dx = x_arr[i + 1] - x_arr[i];

                          int lead_ind = interp_i*length;
                          fill_coefficients(i, length, mode_i, numModes, interp_i, ninterps, B, dx,
                                            &interp_array[0 * npts],
                                            &interp_array[1 * npts],
                                            &interp_array[2 * npts],
                                            &interp_array[3 * npts]);

                      }
             }
}



void fit_wrap(int m, int n, double *a, double *b, double *c, double *d_in){

    #ifdef __HIPCC__
    size_t bufferSizeInBytes;

    hipsparseHandle_t handle;
    void *pBuffer;

    CUSPARSE_CALL(hipsparseCreate(&handle));
    CUSPARSE_CALL( hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m, a, b, c, d_in, n, m, &bufferSizeInBytes));
    gpuErrchk(hipMalloc(&pBuffer, bufferSizeInBytes));

    CUSPARSE_CALL(hipsparseDgtsv2StridedBatch(handle,
                                              m,
                                              a, // dl
                                              b, //diag
                                              c, // du
                                              d_in,
                                              n,
                                              m,
                                              pBuffer));

  CUSPARSE_CALL(hipsparseDestroy(handle));
  gpuErrchk(hipFree(pBuffer));

  #else

#ifdef __USE_OMP__
#pragma omp parallel for
#endif
for (int j = 0;
     j < n;
     j += 1){
       //fit_constants_serial(m, n, w, a, b, c, d_in, x_in, j);
       int info = LAPACKE_dgtsv(LAPACK_COL_MAJOR, m, 1, &a[j*m + 1], &b[j*m], &c[j*m], &d_in[j*m], m);
       //if (info != m) printf("lapack info check: %d\n", info);

   }

  #endif

}
*/
