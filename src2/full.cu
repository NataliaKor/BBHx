#include "hip/hip_runtime.h"
/*  This code was edited by Michael Katz. It is originally from the LAL library.
 *  The original copyright and license is shown below. Michael Katz has edited
 *  the code for his purposes and removed dependencies on the LAL libraries. The code has been confirmed to match the LAL version.
 *  This code is distrbuted under the same GNU license it originally came with.
 *  The comments in the code have been left generally the same. A few comments
 *  have been made for the newer functions added.


 *  Copyright (C) 2017 Sebastian Khan, Francesco Pannarale, Lionel London
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with with program; see the file COPYING. If not, write to the
 *  Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 *  MA  02111-1307  USA
 */
#include <math.h>
#include <complex>
#include <iostream>
#include "stdio.h"
#include <random>

#include "hip/hip_complex.h"
#include "hipblas.h"

#include <stdbool.h>
#include "full.h"

#include "hipsparse.h"

#include "constants.h"
#include "global.h"
// #include "PhenomHM.hh"

#define  NUM_THREADS 256
#define  NUM_THREADS2 64
#define  NUM_THREADS3 256
#define  NUM_THREADS4 256




__device__
void prep_splines(int i, int length, int interp_i, int ninterps, int num_intermediates, double *b, double *ud, double *diag, double *ld, double *x, double *y, int numBinAll, int param, int nsub, int sub_i){
  double dx1, dx2, d, slope1, slope2;
  int ind0x, ind1x, ind2x, ind0y, ind1y, ind2y, ind_out;

  double xval0, xval1, xval2, yval1;

  int numFreqarrs = int(ninterps / num_intermediates);
  int freqArr_i = int(interp_i / num_intermediates);

  //if ((threadIdx.x == 10) && (blockIdx.x == 1)) printf("numFreqarrs %d %d %d %d %d\n", ninterps, interp_i, num_intermediates, numFreqarrs, freqArr_i);
  if (i == length - 1){
    ind0y = (param * nsub + sub_i) * length + (length - 3);
    ind1y = (param * nsub + sub_i) * length + (length - 2);
    ind2y = (param * nsub + sub_i) * length + (length - 1);

    ind0x = freqArr_i * length + (length - 3);
    ind1x = freqArr_i * length + (length - 2);
    ind2x = freqArr_i * length + (length - 1);

    ind_out = (param * nsub + sub_i) * length + (length - 1);

    xval0 = x[ind0x];
    xval1 = x[ind1x];
    xval2 = x[ind2x];

    dx1 = xval1 - xval0;
    dx2 = xval2 - xval1;
    d = xval2 - xval0;

    yval1 = y[ind1y];

    slope1 = (yval1 - y[ind0y])/dx1;
    slope2 = (y[ind2y] - yval1)/dx2;

    b[ind_out] = ((dx2*dx2*slope1 +
                             (2*d + dx2)*dx1*slope2) / d);
    diag[ind_out] = dx1;
    ld[ind_out] = d;
    ud[ind_out] = 0.0;

  } else if (i == 0){

      ind0y = (param * nsub + sub_i) * length + 0;
      ind1y = (param * nsub + sub_i) * length + 1;
      ind2y = (param * nsub + sub_i) * length + 2;

      ind0x = freqArr_i * length + 0;
      ind1x = freqArr_i * length + 1;
      ind2x = freqArr_i * length + 2;

      ind_out = (param * nsub + sub_i) * length + 0;

      xval0 = x[ind0x];
      xval1 = x[ind1x];
      xval2 = x[ind2x];


      dx1 = xval1 - xval0;
      dx2 = xval2 - xval1;
      d = xval2 - xval0;

      yval1 = y[ind1y];

      //amp
      slope1 = (yval1 - y[ind0y])/dx1;
      slope2 = (y[ind2y] - yval1)/dx2;

      b[ind_out] = ((dx1 + 2*d) * dx2 * slope1 +
                          dx1*dx1 * slope2) / d;
    ud[ind_out] = d;
    ld[ind_out] = 0.0;
      diag[ind_out] = dx2;

  } else{

      ind0y = (param * nsub + sub_i) * length + (i - 1);
      ind1y = (param * nsub + sub_i) * length + (i + 0);
      ind2y = (param * nsub + sub_i) * length + (i + 1);

      ind0x = freqArr_i * length + (i - 1);
      ind1x = freqArr_i * length + (i - 0);
      ind2x = freqArr_i * length + (i + 1);

      ind_out = (param * nsub + sub_i) * length + i;

      xval0 = x[ind0x];
      xval1 = x[ind1x];
      xval2 = x[ind2x];

    dx1 = xval1 - xval0;
    dx2 = xval2 - xval1;

    yval1 = y[ind1y];

    //amp
    slope1 = (yval1 - y[ind0y])/dx1;
    slope2 = (y[ind2y] - yval1)/dx2;

    b[ind_out] = 3.0* (dx2*slope1 + dx1*slope2);
    diag[ind_out] = 2*(dx1 + dx2);
    ud[ind_out] = dx1;
    ld[ind_out] = dx2;
  }

}



CUDA_KERNEL
void fill_B(double *freqs_arr, double *y_all, double *B, double *upper_diag, double *diag, double *lower_diag,
                      int ninterps, int length, int num_intermediates, int numModes, int numBinAll){

    int param = 0;
    int nsub = 0;
    int sub_i = 0;
    #ifdef __HIPCC__

    int start1 = blockIdx.x;
    int end1 = ninterps;
    int diff1 = gridDim.x;

    #else

    int start1 = 0;
    int end1 = ninterps;
    int diff1 = 1;

    #endif
    for (int interp_i = start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1){

         #ifdef __HIPCC__

         int start2 = threadIdx.x;
         int end2 = length;
         int diff2 = blockDim.x;

         #else

         int start2 = 0;
         int end2 = length;
         int diff2 = 1;

         #endif

        param = int((double) interp_i/(numModes * numBinAll));
        nsub = numModes * numBinAll;
        sub_i = interp_i % (numModes * numBinAll);

       for (int i = start2;
            i < end2;
            i += diff2){

            int lead_ind = interp_i*length;
            prep_splines(i, length, interp_i, ninterps, num_intermediates, B, upper_diag, diag, lower_diag, freqs_arr, y_all, numBinAll, param, nsub, sub_i);

}
}
}

/*
CuSparse error checking
*/
#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
                             fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
                             exit(-1);}} while(0)

#define CUSPARSE_CALL(X) ERR_NE((X),HIPSPARSE_STATUS_SUCCESS)

void interpolate_kern(int m, int n, double *a, double *b, double *c, double *d_in)
{
        #ifdef __HIPCC__
        size_t bufferSizeInBytes;

        hipsparseHandle_t handle;
        void *pBuffer;

        CUSPARSE_CALL(hipsparseCreate(&handle));
        CUSPARSE_CALL( hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m, a, b, c, d_in, n, m, &bufferSizeInBytes));
        gpuErrchk(hipMalloc(&pBuffer, bufferSizeInBytes));

        CUSPARSE_CALL(hipsparseDgtsv2StridedBatch(handle,
                                                  m,
                                                  a, // dl
                                                  b, //diag
                                                  c, // du
                                                  d_in,
                                                  n,
                                                  m,
                                                  pBuffer));

      CUSPARSE_CALL(hipsparseDestroy(handle));
      gpuErrchk(hipFree(pBuffer));

      #else

    #ifdef __USE_OMP__
    #pragma omp parallel for
    #endif
    for (int j = 0;
         j < n;
         j += 1){
           //fit_constants_serial(m, n, w, a, b, c, d_in, x_in, j);
           int info = LAPACKE_dgtsv(LAPACK_COL_MAJOR, m, 1, &a[j*m + 1], &b[j*m], &c[j*m], &d_in[j*m], m);
           //if (info != m) printf("lapack info check: %d\n", info);

       }

      #endif


    /*
    int interp_i = threadIdx.x + blockDim.x * blockIdx.x;

    int param = (int) (interp_i / (numModes * numBinAll));
    int nsub = numBinAll * numModes;
    int sub_i = interp_i % (numModes * numBinAll);

    int ind_i, ind_im1, ind_ip1;
    if (interp_i < ninterps)
    {

        double w = 0.0;
        for (int i = 1; i < n; i += 1)
        {
            ind_i = (param * n + i) * nsub + sub_i;
            ind_im1 = (param * n + (i-1)) * nsub + sub_i;


            w = a[ind_i]/b[ind_im1];
            b[ind_i] = b[ind_i] - w * c[ind_im1];
            d[ind_i] = d[ind_i] - w * d[ind_im1];
        }

        ind_i = (param * n + (n-1)) * nsub + sub_i;

        d[ind_i] = d[ind_i]/b[ind_i];
        for (int i = n - 2; i >= 0; i -= 1)
        {
            ind_i = (param * n + i) * nsub + sub_i;
            ind_ip1 = (param * n + (i+1)) * nsub + sub_i;

            d[ind_i] = (d[ind_i] - c[ind_i] * d[ind_ip1])/b[ind_i];

        }
    }
    */
}


CUDA_CALLABLE_MEMBER
void fill_coefficients(int i, int length, int sub_i, int nsub, double *dydx, double dx, double *y, double *coeff1, double *coeff2, double *coeff3, int param){
  double slope, t, dydx_i;

  int ind_i = (param * nsub + sub_i) * length + i;
  int ind_ip1 = (param * nsub + sub_i) * length + (i + 1);

  slope = (y[ind_ip1] - y[ind_i])/dx;

  dydx_i = dydx[ind_i];

  t = (dydx_i + dydx[ind_ip1] - 2*slope)/dx;

  coeff1[ind_i] = dydx_i;
  coeff2[ind_i] = (slope - dydx_i) / dx - t;
  coeff3[ind_i] = t/dx;

  //if ((param == 1) && (i == length - 3) && (sub_i == 0)) printf("freq check: %d %d %d %d %d\n", i, dydx[ind_i], dydx[ind_ip1]);


}

CUDA_KERNEL
void set_spline_constants(double *f_arr, double* y, double *c1, double* c2, double* c3, double *B,
                      int ninterps, int length, int num_intermediates, int numBinAll, int numModes){

    double df;
    #ifdef __HIPCC__
    int start1 = blockIdx.x;
    int end1 = ninterps;
    int diff1 = gridDim.x;
    #else

    int start1 = 0;
    int end1 = ninterps;
    int diff1 = 1;

    #endif

    for (int interp_i= start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1){

     int numFreqarrs = int(ninterps / num_intermediates);
     int freqArr_i = int(interp_i / num_intermediates);

     int param = (int) (interp_i / (numModes * numBinAll));
     int nsub = numBinAll * numModes;
     int sub_i = interp_i % (numModes * numBinAll);

     #ifdef __HIPCC__
     int start2 = threadIdx.x;
     int end2 = length - 1;
     int diff2 = blockDim.x;
     #else

     int start2 = 0;
     int end2 = length - 1;
     int diff2 = 1;

     #endif
     for (int i = start2;
            i < end2;
            i += diff2){

                // TODO: check if there is faster way to do this
              df = f_arr[freqArr_i * length + (i + 1)] - f_arr[freqArr_i * length + i];

              int lead_ind = interp_i*length;
              fill_coefficients(i, length, sub_i, nsub, B, df,
                                y,
                                c1,
                                c2,
                                c3, param);

}
}
}


__device__ double atomicAddDouble(double* address, double val)
{
    unsigned long long* address_as_ull =
                              (unsigned long long*)address;
    unsigned long long old = *address_as_ull, assumed;

    do {
        assumed = old;
        old = atomicCAS(address_as_ull, assumed,
                        __double_as_longlong(val +
                               __longlong_as_double(assumed)));

    // Note: uses integer comparison to avoid hang in case of NaN (since NaN != NaN)
    } while (assumed != old);

    return __longlong_as_double(old);
}

__device__ void atomicAddComplex(cmplx* a, cmplx b){
  //transform the addresses of real and imag. parts to double pointers
  double *x = (double*)a;
  double *y = x+1;
  //use atomicAdd for double variables
  atomicAddDouble(x, b.real());
  atomicAddDouble(y, b.imag());
}



#define  DATA_BLOCK 128
#define  NUM_INTERPS 9

__device__
cmplx get_ampphasefactor(double amp, double phase, double phaseShift){
    return amp*gcmplx::exp(cmplx(0.0, phase + phaseShift));
}

__device__
cmplx combine_information(cmplx* channel1, cmplx* channel2, cmplx* channel3, double amp, double phase, double tf, cmplx transferL1, cmplx transferL2, cmplx transferL3, double t_start, double t_end)
{
    // TODO: make sure the end of the ringdown is included
    if ((tf >= t_start) && ((tf <= t_end) || (t_end <= 0.0)) && (amp > 1e-40))
    {
        cmplx amp_phase_term = amp*gcmplx::exp(cmplx(0.0, -phase));  // add phase shift

        *channel1 = gcmplx::conj(transferL1 * amp_phase_term);
        *channel2 = gcmplx::conj(transferL2 * amp_phase_term);
        *channel3 = gcmplx::conj(transferL3 * amp_phase_term);

    }
}

#define  NUM_TERMS 4

#define  MAX_NUM_COEFF_TERMS 1200

CUDA_KERNEL
void TDI(cmplx* templateChannels, double* dataFreqsIn, double dlog10f, double* freqsOld, double* propArrays, double* c1In, double* c2In, double* c3In, double t_mrg, int old_length, int data_length, int numBinAll, int numModes, double t_obs_start, double t_obs_end, int* inds, int ind_start, int ind_length, int bin_i)
{

    int num_params = 9;
    //int mode_i = blockIdx.y;

    int numAll = numBinAll * numModes * old_length;

    double tempLike, addLike, time_check, phaseShift;
    cmplx trans_complex1, trans_complex2, trans_complex3, ampphasefactor;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (i >= ind_length) return;

    double f = dataFreqsIn[i + ind_start];

    int ind_here = inds[i];

    double f_old = freqsOld[ind_here];

    double x = f - f_old;
    double x2 = x * x;
    double x3 = x * x2;

    trans_complex1 = 0.0; trans_complex2 = 0.0; trans_complex3 = 0.0;

    for (int mode_i = 0; mode_i < numModes; mode_i += 1)
    {
        int int_shared = ((0 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double amp = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        //if ((i == 100) || (i == 101)) printf("%d %d %d %e %e %e %e %e %e\n", window_i, mode_i, i, amp, f, f_old, y[int_shared], c1[int_shared], c2[int_shared]);

        int_shared = ((1 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double phase = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((2 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double tf = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((3 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL1_re = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((4 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL1_im = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((5 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL2_re = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((6 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL2_im = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((7 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL3_re = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        int_shared = ((8 * numBinAll + bin_i) * numModes + mode_i) * old_length + ind_here;
        double transferL3_im = propArrays[int_shared] + c1In[int_shared] * x + c2In[int_shared] * x2 + c3In[int_shared] * x3;

        cmplx channel1(0.0, 0.0);
        cmplx channel2(0.0, 0.0);
        cmplx channel3(0.0, 0.0);

        combine_information(&channel1, &channel2, &channel3, amp, phase, tf, cmplx(transferL1_re, transferL1_im), cmplx(transferL2_re, transferL2_im), cmplx(transferL3_re, transferL3_im), t_obs_start, t_obs_end);

        trans_complex1 += channel1;
        trans_complex2 += channel2;
        trans_complex3 += channel3;
    }

    atomicAddComplex(&templateChannels[0 * ind_length + i], trans_complex1);
    atomicAddComplex(&templateChannels[1 * ind_length + i], trans_complex2);
    atomicAddComplex(&templateChannels[2 * ind_length + i], trans_complex3);


    /*
    __shared__ double y[MAX_NUM_COEFF_TERMS];
    __shared__ double c1[MAX_NUM_COEFF_TERMS];
    __shared__ double c2[MAX_NUM_COEFF_TERMS];
    __shared__ double c3[MAX_NUM_COEFF_TERMS];
    __shared__ double freqs_shared[MAX_NUM_COEFF_TERMS];

    int num_params = 9;
    int mode_i = blockIdx.y;

    int numAll = numBinAll * numModes * old_length;

    double amp, phase, tfCorr, transferL1_re, transferL1_im, transferL2_re, transferL2_im, transferL3_re, transferL3_im;
    double x, x2, x3, tempLike, addLike, time_check, phaseShift;
    cmplx trans_complex1, trans_complex2, trans_complex3, ampphasefactor;

    __shared__ int start_ind, end_ind;

    int i = blockIdx.x * blockDim.x + threadIdx.x;

    bool run = true;
    if (i >= ind_length) run = false;

    __syncthreads();

    if (threadIdx.x == 0)
    {
        start_ind = inds[i];
    }
    int max_thread_num = (ind_length - blockDim.x*blockIdx.x > NUM_THREADS3) ? NUM_THREADS3 : ind_length - blockDim.x*blockIdx.x;

    if (threadIdx.x == max_thread_num - 1)
    {
        end_ind = inds[i];
    }

    __syncthreads();

    int num_windows = end_ind - start_ind + 1;

    int nsub = numModes * numBinAll;

    //if (run) printf("%d %d %d %d %d %d %d %d\n", max_thread_num, threadIdx.x, blockDim.x, NUM_THREADS3, i, ind_length, start_ind, end_ind);

    for (int j = threadIdx.x; j < num_windows; j += blockDim.x)
    {
        int window_i = j;

        int old_ind = start_ind + window_i;

        if ((old_ind < 0) || (old_ind >= old_length))
        {
            continue;
        }

        freqs_shared[window_i] = freqsOld[old_ind];

        //if ((blockIdx.x == 0) && (blockIdx.y == 0)) printf("%d %d %e %e\n", old_ind, window_i, freqs_shared[window_i], freqsOld[old_ind]);
    }

    __syncthreads();

    for (int j = threadIdx.x; j < num_params * num_windows; j += blockDim.x)
    {
        int window_i = j % num_windows;
        int param_i = (int) (j / num_windows);

        int old_ind = start_ind + window_i;

        if ((old_ind < 0) || (old_ind >= old_length))
        {
            continue;
        }

        int ind = ((param_i * numBinAll + bin_i) * numModes + mode_i) * old_length + old_ind;
        int ind_shared = window_i * num_params + param_i;

        y[ind_shared] = propArrays[ind];
        c1[ind_shared] = c1In[ind];
        c2[ind_shared] = c2In[ind];
        c3[ind_shared] = c3In[ind];

        if (ind_shared > MAX_NUM_COEFF_TERMS) printf("BAD %d %d\n", ind_shared, window_i);
    }

    __syncthreads();

    if (run)
    {
        double f = dataFreqsIn[i + ind_start];

        int ind_here = inds[i];

        int window_i = ind_here - start_ind;

        double f_old = freqs_shared[window_i];

        double x = f - f_old;
        double x2 = x * x;
        double x3 = x * x2;

        int int_shared = window_i * num_params + 0;
        double amp = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        //if ((i == 100) || (i == 101)) printf("%d %d %d %e %e %e %e %e %e\n", window_i, mode_i, i, amp, f, f_old, y[int_shared], c1[int_shared], c2[int_shared]);

        int_shared = window_i * num_params + 1;
        double phase = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 2;
        double tf = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 3;
        double transferL1_re = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 4;
        double transferL1_im = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 5;
        double transferL2_re = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 6;
        double transferL2_im = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 7;
        double transferL3_re = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        int_shared = window_i * num_params + 8;
        double transferL3_im = y[int_shared] + c1[int_shared] * x + c2[int_shared] * x2 + c3[int_shared] * x3;

        cmplx channel1(0.0, 0.0);
        cmplx channel2(0.0, 0.0);
        cmplx channel3(0.0, 0.0);

        combine_information(&channel1, &channel2, &channel3, amp, phase, tf, cmplx(transferL1_re, transferL1_im), cmplx(transferL2_re, transferL2_im), cmplx(transferL3_re, transferL3_im), t_obs_start, t_obs_end);

        atomicAddComplex(&templateChannels[0 * ind_length + i], channel1);
        atomicAddComplex(&templateChannels[1 * ind_length + i], channel2);
        atomicAddComplex(&templateChannels[2 * ind_length + i], channel3);

    }
    */
}

CUDA_KERNEL
void fill_waveform(cmplx* templateChannels,
                double* bbh_buffer,
                int numBinAll, int data_length, int nChannels, int numModes, double* t_start, double* t_end)
{

    cmplx I(0.0, 1.0);

    cmplx temp_channel1 = 0.0, temp_channel2 = 0.0, temp_channel3 = 0.0;
    for (int bin_i = blockIdx.x; bin_i < numBinAll; bin_i += gridDim.x)
    {

        double t_start_bin = t_start[bin_i];
        double t_end_bin = t_end[bin_i];

        for (int i = threadIdx.x; i < data_length; i += blockDim.x)
        {
            cmplx temp_channel1 = 0.0, temp_channel2 = 0.0, temp_channel3 = 0.0;
            for (int mode_i = 0; mode_i < numModes; mode_i += 1)
            {

                int ind = ((0 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double amp = bbh_buffer[ind];

                ind = ((1 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double phase = bbh_buffer[ind];

                ind = ((2 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double tf = bbh_buffer[ind];

                ind = ((3 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL1_re = bbh_buffer[ind];

                ind = ((4 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL1_im = bbh_buffer[ind];

                ind = ((5 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL2_re = bbh_buffer[ind];

                ind = ((6 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL2_im = bbh_buffer[ind];

                ind = ((7 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL3_re = bbh_buffer[ind];

                ind = ((8 * numBinAll + bin_i) * numModes + mode_i) * data_length + i;
                double transferL3_im = bbh_buffer[ind];

                cmplx channel1 = 0.0 + 0.0 * I;
                cmplx channel2 = 0.0 + 0.0 * I;
                cmplx channel3 = 0.0 + 0.0 * I;

                combine_information(&channel1, &channel2, &channel3, amp, phase, tf, cmplx(transferL1_re, transferL1_im), cmplx(transferL2_re, transferL2_im), cmplx(transferL3_re, transferL3_im), t_start_bin, t_end_bin);

                temp_channel1 += channel1;
                temp_channel2 += channel2;
                temp_channel3 += channel3;
            }

            templateChannels[(bin_i * 3 + 0) * data_length + i] = temp_channel1;
            templateChannels[(bin_i * 3 + 1) * data_length + i] = temp_channel2;
            templateChannels[(bin_i * 3 + 2) * data_length + i] = temp_channel3;

        }
    }
}

void direct_sum(cmplx* templateChannels,
                double* bbh_buffer,
                int numBinAll, int data_length, int nChannels, int numModes, double* t_start, double* t_end)
{

    int nblocks5 = numBinAll; // std::ceil((numBinAll + NUM_THREADS4 -1)/NUM_THREADS4);

    fill_waveform<<<nblocks5, NUM_THREADS4>>>(templateChannels, bbh_buffer, numBinAll, data_length, nChannels, numModes, t_start, t_end);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
}


void InterpTDI(long* templateChannels_ptrs, double* dataFreqs, double dlog10f, double* freqs, double* propArrays, double* c1, double* c2, double* c3, double* t_mrg_in, double* t_start_in, double* t_end_in, int length, int data_length, int numBinAll, int numModes, double t_obs_start, double t_obs_end, long* inds_ptrs, int* inds_start, int* ind_lengths)
{

    hipStream_t streams[numBinAll];

    #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        int length_bin_i = ind_lengths[bin_i];
        int ind_start = inds_start[bin_i];
        int* inds = (int*) inds_ptrs[bin_i];

        double t_mrg = t_mrg_in[bin_i];
        double t_start = t_start_in[bin_i];
        double t_end = t_end_in[bin_i];

        cmplx* templateChannels = (cmplx*) templateChannels_ptrs[bin_i];

        int nblocks3 = std::ceil((length_bin_i + NUM_THREADS3 -1)/NUM_THREADS3);
        hipStreamCreate(&streams[bin_i]);

        dim3 gridDim(nblocks3, 1);
        TDI<<<gridDim, NUM_THREADS3, 0, streams[bin_i]>>>(templateChannels, dataFreqs, dlog10f, freqs, propArrays, c1, c2, c3, t_mrg, length, data_length, numBinAll, numModes, t_start, t_end, inds, ind_start, length_bin_i, bin_i);

    }

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        //destroy the streams
        hipStreamDestroy(streams[bin_i]);
    }
}

#define  DATA_BLOCK2 512
CUDA_KERNEL
void hdynLikelihood(cmplx* likeOut1, cmplx* likeOut2,
                    cmplx* templateChannels, cmplx* dataConstants,
                    double* dataFreqsIn,
                    int numBinAll, int data_length, int nChannels)
{
    __shared__ cmplx A0temp[DATA_BLOCK2];
    __shared__ cmplx A1temp[DATA_BLOCK2];
    __shared__ cmplx B0temp[DATA_BLOCK2];
    __shared__ cmplx B1temp[DATA_BLOCK2];
    __shared__ double dataFreqs[DATA_BLOCK2];

    cmplx A0, A1, B0, B1;

    cmplx trans_complex(0.0, 0.0);
    cmplx prev_trans_complex(0.0, 0.0);
    double prevFreq = 0.0;
    double freq = 0.0;

    int currentStart = 0;

    cmplx r0, r1, r1Conj, tempLike1, tempLike2;
    double mag_r0, midFreq;

    int binNum = threadIdx.x + blockDim.x * blockIdx.x;

    if (true) // for (int binNum = threadIdx.x + blockDim.x * blockIdx.x; binNum < numBinAll; binNum += blockDim.x * gridDim.x)
    {
        tempLike1 = 0.0;
        tempLike2 = 0.0;
        for (int channel = 0; channel < nChannels; channel += 1)
        {
            prevFreq = 0.0;
            currentStart = 0;
            while (currentStart < data_length)
            {
                __syncthreads();
                for (int jj = threadIdx.x; jj < DATA_BLOCK2; jj += blockDim.x)
                {
                    if ((jj + currentStart) >= data_length) continue;
                    A0temp[jj] = dataConstants[(0 * nChannels + channel) * data_length + currentStart + jj];
                    A1temp[jj] = dataConstants[(1 * nChannels + channel) * data_length + currentStart + jj];
                    B0temp[jj] = dataConstants[(2 * nChannels + channel) * data_length + currentStart + jj];
                    B1temp[jj] = dataConstants[(3 * nChannels + channel) * data_length + currentStart + jj];

                    dataFreqs[jj] = dataFreqsIn[currentStart + jj];

                    //if ((jj + currentStart < 3) && (binNum == 0) & (channel == 0))
                    //    printf("check %e %e, %e %e, %e %e, %e %e, %e \n", A0temp[jj], A1temp[jj], B0temp[jj], B1temp[jj], dataFreqs[jj]);

                }
                __syncthreads();
                if (binNum < numBinAll)
                {
                    for (int jj = 0; jj < DATA_BLOCK2; jj += 1)
                    {
                        if ((jj + currentStart) >= data_length) continue;
                        freq = dataFreqs[jj];
                        trans_complex = templateChannels[((jj + currentStart) * nChannels + channel) * numBinAll + binNum];

                        if ((prevFreq != 0.0) && (jj + currentStart > 0))
                        {
                            A0 = A0temp[jj]; // constants will need to be aligned with 1..n-1 because there are data_length - 1 bins
                            A1 = A1temp[jj];
                            B0 = B0temp[jj];
                            B1 = B1temp[jj];

                            r1 = (trans_complex - prev_trans_complex)/(freq - prevFreq);
                            midFreq = (freq + prevFreq)/2.0;

                            r0 = trans_complex - r1 * (freq - midFreq);

                            //if (((binNum == 767) || (binNum == 768)) & (channel == 0))
                            //    printf("CHECK2: %d %d %d %e %e\n", jj + currentStart, binNum, jj, A0); // , %e %e, %e %e, %e %e, %e %e,  %e %e,  %e %e , %e\n", ind, binNum, jj + currentStart, A0, A1, B0, B1, freq, prevFreq, trans_complex, prev_trans_complex, midFreq);

                            r1Conj = gcmplx::conj(r1);

                            tempLike1 += A0 * gcmplx::conj(r0) + A1 * r1Conj;

                            mag_r0 = gcmplx::abs(r0);
                            tempLike2 += B0 * (mag_r0 * mag_r0) + 2. * B1 * gcmplx::real(r0 * r1Conj);
                        }

                        prev_trans_complex = trans_complex;
                        prevFreq = freq;
                    }
                }
                currentStart += DATA_BLOCK2;
            }
        }
        if (binNum < numBinAll)
        {
            likeOut1[binNum] = tempLike1;
            likeOut2[binNum] = tempLike2;
        }
    }
}




void interpolate(double* freqs, double* propArrays,
                 double* B, double* upper_diag, double* diag, double* lower_diag,
                 int length, int numInterpParams, int numModes, int numBinAll)
{

    int num_intermediates = numModes * numInterpParams;
    int ninterps = numModes * numInterpParams * numBinAll;

    int nblocks = std::ceil((ninterps + NUM_THREADS -1)/NUM_THREADS);

    double* c1 = upper_diag; //&interp_array[0 * numInterpParams * amp_phase_size];
    double* c2 = diag; //&interp_array[1 * numInterpParams * amp_phase_size];
    double* c3 = lower_diag; //&interp_array[2 * numInterpParams * amp_phase_size];

    //printf("%d after response, %d\n", jj, nblocks2);

     fill_B<<<nblocks, NUM_THREADS>>>(freqs, propArrays, B, upper_diag, diag, lower_diag, ninterps, length, num_intermediates, numModes, numBinAll);
     hipDeviceSynchronize();
     gpuErrchk(hipGetLastError());

     //printf("%d after fill b\n", jj);
     interpolate_kern(length, ninterps, lower_diag, diag, upper_diag, B);


  set_spline_constants<<<nblocks, NUM_THREADS>>>(freqs, propArrays, c1, c2, c3, B,
                    ninterps, length, num_intermediates, numBinAll, numModes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    //printf("%d after set spline\n", jj);
}

void hdyn(cmplx* likeOut1, cmplx* likeOut2,
                    cmplx* templateChannels, cmplx* dataConstants,
                    double* dataFreqs,
                    int numBinAll, int data_length, int nChannels)
{

    int nblocks4 = std::ceil((numBinAll + NUM_THREADS4 -1)/NUM_THREADS4);

    hdynLikelihood<<<nblocks4, NUM_THREADS4>>>(likeOut1, likeOut2, templateChannels, dataConstants, dataFreqs, numBinAll, data_length, nChannels);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
}

CUDA_KERNEL
void noiseweight_template(cmplx* templateChannels, double* noise_weight_times_df, int ind_start, int length, int data_stream_length)
{
    for (int i = threadIdx.x + blockDim.x * blockIdx.x; i < length; i += gridDim.x * blockDim.x)
    {
        for (int j = 0; j < 3; j+= 1)
        {
            templateChannels[j * length + i] = templateChannels[j * length + i] * noise_weight_times_df[j * data_stream_length + ind_start + i];
        }
    }
}

#define NUM_THREADS_LIKE 256

void direct_like(double* d_h, double* h_h, cmplx* dataChannels, double* noise_weight_times_df, long* templateChannels_ptrs, int* inds_start, int* ind_lengths, int data_stream_length, int numBinAll)
{

    hipStream_t streams[numBinAll];
    hipblasHandle_t handle;

    hipDoubleComplex result_d_h[numBinAll];
    hipDoubleComplex result_h_h[numBinAll];

    hipblasStatus_t stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS)
    {
      printf ("CUBLAS initialization failed\n");
      exit(0);
    }

    #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        int length_bin_i = ind_lengths[bin_i];
        int ind_start = inds_start[bin_i];

        cmplx* templateChannels = (cmplx*) templateChannels_ptrs[bin_i];

        int nblocks = std::ceil((length_bin_i + NUM_THREADS_LIKE -1)/NUM_THREADS_LIKE);
        hipStreamCreate(&streams[bin_i]);

        noiseweight_template<<<nblocks, NUM_THREADS_LIKE, 0, streams[bin_i]>>>(templateChannels, noise_weight_times_df, ind_start, length_bin_i, data_stream_length);
        hipStreamSynchronize(streams[bin_i]);

        for (int j = 0; j < 3; j += 1)
        {

            hipblasSetStream(handle, streams[bin_i]);
            stat = hipblasZdotc(handle, length_bin_i,
                              (hipDoubleComplex*)&dataChannels[j * data_stream_length + ind_start], 1,
                              (hipDoubleComplex*)&templateChannels[j * length_bin_i], 1,
                              &result_d_h[bin_i]);
            hipStreamSynchronize(streams[bin_i]);
            if (stat != HIPBLAS_STATUS_SUCCESS)
            {
                exit(0);
            }

            d_h[bin_i] += 4.0 * hipCreal(result_d_h[bin_i]);

            hipblasSetStream(handle, streams[bin_i]);
            stat = hipblasZdotc(handle, length_bin_i,
                              (hipDoubleComplex*)&templateChannels[j * length_bin_i], 1,
                              (hipDoubleComplex*)&templateChannels[j * length_bin_i], 1,
                              &result_h_h[bin_i]);
            hipStreamSynchronize(streams[bin_i]);
            if (stat != HIPBLAS_STATUS_SUCCESS)
            {
                exit(0);
            }
            h_h[bin_i] += 4.0 * hipCreal(result_h_h[bin_i]);

        }
    }

    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    #pragma omp parallel for
    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        //destroy the streams
        hipStreamDestroy(streams[bin_i]);
    }
    hipblasDestroy(handle);
}
/*
int main()
{

    int TDItag = 1;
    int order_fresnel_stencil = 0;
    double tBase = 1.0;

    int numBinAll = 5000;
    int numModes = 6;
    int length = 1024;
    int data_length = 4096;

    int *ells_in, *mms_in;

    gpuErrchk(hipMallocManaged(&ells_in, numModes * sizeof(int)));
    gpuErrchk(hipMallocManaged(&mms_in, numModes * sizeof(int)));

    ells_in[0] = 2;
    ells_in[1] = 3;
    ells_in[2] = 4;

    ells_in[3] = 2;
    ells_in[4] = 3;
    ells_in[5] = 4;

    mms_in[0] = 2;
    mms_in[1] = 3;
    mms_in[2] = 4;

    mms_in[3] = 1;
    mms_in[4] = 2;
    mms_in[5] = 3;

    double *amps, *phases, *phases_deriv, *freqs, *m1_SI, *m2_SI, *chi1z, *chi2z, *distance, *phiRef, *fRef;
    double *inc, *lam, *beta, *psi, *tRef_wave_frame, *tRef_sampling_frame;
    double *response_out;
    double *B, *interp_array; // plays roll of upper lower diag, and then coefficients 1, 2, 3

    size_t amp_phase_size = numBinAll * numModes * length *sizeof(double);
    size_t freqs_size = numBinAll * length * sizeof(double);
    size_t bin_size = numBinAll * sizeof(double);

    int numInterpParams = 9;

    gpuErrchk(hipMallocManaged(&amps, numInterpParams * amp_phase_size));

    response_out = &amps[1 * numBinAll * numModes * length];

    double *upper_diag, *diag, *lower_diag;
    gpuErrchk(hipMallocManaged(&B, numInterpParams * amp_phase_size));
    gpuErrchk(hipMallocManaged(&upper_diag, numInterpParams * amp_phase_size));
    gpuErrchk(hipMallocManaged(&diag, numInterpParams * amp_phase_size));
    gpuErrchk(hipMallocManaged(&lower_diag, numInterpParams * amp_phase_size));

    //double* upper_diag = &interp_array[0 * numInterpParams * amp_phase_size];
    //double* diag = &interp_array[1 * numInterpParams * amp_phase_size];
    //double* lower_diag = &interp_array[2 * numInterpParams * amp_phase_size];

    double* propArrays = amps;

    gpuErrchk(hipMallocManaged(&freqs, freqs_size));

    gpuErrchk(hipMallocManaged(&m1_SI, bin_size));
    gpuErrchk(hipMallocManaged(&m2_SI, bin_size));
    gpuErrchk(hipMallocManaged(&chi1z, bin_size));
    gpuErrchk(hipMallocManaged(&chi2z, bin_size));
    gpuErrchk(hipMallocManaged(&distance, bin_size));
    gpuErrchk(hipMallocManaged(&phiRef, bin_size));
    gpuErrchk(hipMallocManaged(&fRef, bin_size));

    gpuErrchk(hipMallocManaged(&inc, bin_size));
    gpuErrchk(hipMallocManaged(&lam, bin_size));
    gpuErrchk(hipMallocManaged(&beta, bin_size));
    gpuErrchk(hipMallocManaged(&psi, bin_size));
    gpuErrchk(hipMallocManaged(&tRef_wave_frame, bin_size));
    gpuErrchk(hipMallocManaged(&tRef_sampling_frame, bin_size));

    double m1 = 2e6; // solar
    double m2 = 1e6;
    double a1 = 0.8;
    double a2 = 0.8;
    double dist = 30.0; // Gpc
    double phi_ref = 0.0;
    double f_ref = 0.0;
    double inc_in = PI/3.;
    double lam_in = 0.4;
    double beta_in = 0.24;
    double psi_in = 1.0;
    double tRef_wave_frame_in = 10.0;
    double tRef_sampling_frame_in = 50.0;

    double Msec = (m1 + m2) * MTSUN_SI;

    double log10f_start = log10(1e-4/Msec);
    double log10f_end = log10(0.6/Msec);

    double dlog10f = (log10f_end - log10f_start)/(length - 1);

    std::default_random_engine generator;
    std::uniform_real_distribution<double> distribution(0.0,1.0);

    for (int bin_i = 0; bin_i < numBinAll; bin_i += 1)
    {
        m1_SI[bin_i] = (1e6 * MSUN_SI) * (1 + distribution(generator));
        m2_SI[bin_i] = (4e5 * MSUN_SI) * (1 + distribution(generator));

        chi1z[bin_i] = (distribution(generator))* 0.9;
        chi2z[bin_i] = (distribution(generator))* 0.9;

        distance[bin_i] = (35) * (1 + distribution(generator)) * 1e9 * PC_SI;
        phiRef[bin_i] = (1 + distribution(generator));
        fRef[bin_i] = f_ref;

        inc[bin_i] = (distribution(generator)) * 0.25 + 0.25;
        lam[bin_i] = (distribution(generator)) * 0.25 + 0.25;
        beta[bin_i] = (distribution(generator)) * 0.25 + 0.25;
        psi[bin_i] = (distribution(generator)) * 0.25 + 0.25;
        tRef_wave_frame[bin_i] = (1 + distribution(generator)) * 20.0;
        tRef_sampling_frame[bin_i] = (1 + distribution(generator)) * 20.0;

        for (int i = 0; i < length; i += 1)
        {
            freqs[i * numBinAll + bin_i] = pow(10.0, log10f_start + i * dlog10f);
        }
    }

    cmplx *dataChannels, *templateChannels, *dataConstants;
    double *dataFreqs;
    int nChannels = 3;

    double t_obs_start = 1.0;
    double t_obs_end = 0.0;

    gpuErrchk(hipMallocManaged(&dataChannels, nChannels * data_length * sizeof(cmplx)));
    gpuErrchk(hipMallocManaged(&dataConstants, NUM_TERMS * nChannels * data_length * sizeof(cmplx)));
    gpuErrchk(hipMallocManaged(&templateChannels, numBinAll * nChannels * data_length * sizeof(cmplx)));
    gpuErrchk(hipMallocManaged(&dataFreqs, data_length * sizeof(double)));

    double dlog10fData = (log10f_end - log10f_start)/(data_length - 1);

    for (int i = 0; i < data_length; i += 1)
    {
        dataFreqs[i] = pow(10.0, log10f_start + i * dlog10fData);

        for (int channel = 0; channel < nChannels; channel += 1)
        {
            dataChannels[channel * data_length + i] = cmplx(1.0, 1.0);

            for (int constant = 0; constant < NUM_TERMS; constant += 1)
            {
                dataConstants[(constant * nChannels + channel) * data_length + i] = cmplx(1.0, 1.0);
            }
        }
    }

    cmplx *likeOut1;
    gpuErrchk(hipMallocManaged(&likeOut1, numBinAll * sizeof(cmplx)));

    cmplx *likeOut2;
    gpuErrchk(hipMallocManaged(&likeOut2, numBinAll * sizeof(cmplx)));

    double *c1, *c2, *c3;
    int numIter = 10;

    for (int jj = 0; jj < numIter; jj += 1)
    {

        //printf("%d begin\n", jj);
        waveform_amp_phase(
        amps, ///**< [out] Frequency-domain waveform hx
        ells_in,
        mms_in,
        freqs,               ///**< Frequency points at which to evaluate the waveform (Hz)
        m1_SI,                       // /**< mass of companion 1 (kg)
        m2_SI,                        ///**< mass of companion 2 (kg)
        chi1z,                        ///**< z-component of the dimensionless spin of object 1 w.r.t. Lhat = (0,0,1)
        chi2z,                        ///**< z-component of the dimensionless spin of object 2 w.r.t. Lhat = (0,0,1)
        distance,               ///**< distance of source (m)
        phiRef,                 ///**< reference orbital phase (rad)
        fRef,                      //  /**< Reference frequency
        numModes,
        length,
        numBinAll
   );

   int includesAmps = 0;
   LISA_response(
       response_out,
       ells_in,
       mms_in,
       freqs,               ///**< Frequency points at which to evaluate the waveform (Hz)
       phiRef,                // /**< reference orbital phase (rad)
       fRef,                    //    /**< Reference frequency
       inc,
       lam,
       beta,
       psi,
       tRef_wave_frame,
       tRef_sampling_frame,
       tBase, TDItag, order_fresnel_stencil,
       numModes,
       length,
       numBinAll,
       includesAmps
  );

  interpolate(freqs, propArrays,
                   B, upper_diag, diag, lower_diag,
                 length, numInterpParams, numModes, numBinAll);

    //printf("%d middle\n", jj);

    c1 = upper_diag; //&interp_array[0 * numInterpParams * amp_phase_size];
    c2 = diag; //&interp_array[1 * numInterpParams * amp_phase_size];
    c3 = lower_diag; //&interp_array[2 * numInterpParams * amp_phase_size];


    InterpTDI(templateChannels, dataChannels, dataFreqs, freqs, propArrays, c1, c2, c3, tBase, tRef_sampling_frame, tRef_wave_frame, length, data_length,   numBinAll, numModes, t_obs_start, t_obs_end);

    hdyn(likeOut1, likeOut2, templateChannels, dataConstants, dataFreqs, numBinAll, data_length, nChannels);
    }

    int binNum = 1000;
    int mode_i = 0;
    for (int i = 0; i < 5; i += 1) printf("%d %e %e\n", i, c1[(i * numModes + 0) * numBinAll + 0], c2[(i * numModes + 0) * numBinAll + 0]);

    return 0;
}

*/

/*
__device__
void fill_coefficients(int i, int length, int mode_i, int numModes, int interp_i, int ninterps, double *dydx, double dx, double *y, double *coeff1, double *coeff2, double *coeff3){
  double slope, t, dydx_i;

  int indip1 = ((i + 1) * numModes + mode_i) * ninterps + interp_i;
  int indi = ((i) * numModes + mode_i) * ninterps + interp_i;

  slope = (y[indip1] - y[indi])/dx;

  dydx_i = dydx[indi];

  t = (dydx_i + dydx[indip1] - 2*slope)/dx;

  coeff1[indi] = dydx_i;
  coeff2[indi] = (slope - dydx_i) / dx - t;
  coeff3[indi] = t/dx;
}




__device__
void prep_splines(int i, int length, int mode_i, int numModes, int interp_i, int ninterps,  double *b, double *ud, double *diag, double *ld, double *x, double *y){
  double dx1, dx2, d, slope1, slope2;
  int ind1x, ind2x, ind3x, ind1y, ind2y, ind3y;
  if (i == length - 1){

     ind1x = (length - 2) * ninterps + interp_i;
     ind2x = (length - 3) * ninterps + interp_i;
     ind3x = (length - 1) * ninterps + interp_i;

     ind1y = ((length - 2) * numModes + mode_i) * ninterps + interp_i;
     ind2y = ((length - 3) * numModes + mode_i) * ninterps + interp_i;
     ind3y = ((length - 1) * numModes + mode_i) * ninterps + interp_i;


  } else if (i == 0){

      ind1x = 1 * ninterps + interp_i;
      ind2x = 0 * ninterps + interp_i;
      ind3x = 2 * ninterps + interp_i;

      ind1y = (1 * numModes + mode_i) * ninterps + interp_i;
      ind2y = (0 * numModes + mode_i) * ninterps + interp_i;
      ind3y = (2 * numModes + mode_i) * ninterps + interp_i;


  } else{

      ind1x = (i) * ninterps + interp_i;
      ind2x = (i-1) * ninterps + interp_i;
      ind3x = (i+1) * ninterps + interp_i;

      ind1y = ((i) * numModes + mode_i) * ninterps + interp_i;
      ind2y = ((i-1) * numModes + mode_i) * ninterps + interp_i;
      ind3y = ((i+1) * numModes + mode_i) * ninterps + interp_i;
  }

    dx1 = x[ind1x] - x[ind2x];
    dx2 = x[ind3x] - x[ind1x];

    //amp
    slope1 = (y[ind1y] - y[ind2y])/dx1;
    slope2 = (y[ind3y] - y[ind1y])/dx2;

    b[ind1y] = 3.0* (dx2*slope1 + dx1*slope2);
    diag[ind1y] = 2*(dx1 + dx2);
    ud[ind1y] = dx1;
    ld[ind1y] = dx2;
}



CUDA_KERNEL
void fill_B(double *x_arr, double *y_all, double *B, double *upper_diag, double *diag, double *lower_diag,
                      int ninterps, int length, int numModes){


    int start1 = blockIdx.x*blockDim.x + threadIdx.x;
    int end1 = ninterps;
    int diff1 = blockDim.x*gridDim.x;

    for (int interp_i= start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1)
        {

       for (int mode_i = 0; mode_i < numModes; mode_i += 1)
       {
           for (int i = start2;
                i < end2;
                i += diff2)
                {
                    prep_splines(i, length, mode_i, numModes, interp_i, ninterps,  B, upper_diag, diag, lower_diag, x_arr, y_all);

                }
       }

    }
}



CUDA_KERNEL
void set_spline_constants(double *x_arr, double *interp_array, double *B,
                      int ninterps, int length, int numModes){

    double dx;
    InterpContainer mode_vals;

    int start1 = blockIdx.x*blockDim.x + threadIdx.x;
    int end1 = ninterps;
    int diff1 = blockDim.x*gridDim.x;

    int npts = ninterps * length * numModes;

    for (int interp_i= start1;
         interp_i<end1; // 2 for re and im
         interp_i+= diff1){

             for (int mode_i = 0; mode_i < numModes; mode_i += 1)
             {
                 for (int i = start2;
                      i < end2;
                      i += diff2)
                      {
                          dx = x_arr[i + 1] - x_arr[i];

                          int lead_ind = interp_i*length;
                          fill_coefficients(i, length, mode_i, numModes, interp_i, ninterps, B, dx,
                                            &interp_array[0 * npts],
                                            &interp_array[1 * npts],
                                            &interp_array[2 * npts],
                                            &interp_array[3 * npts]);

                      }
             }
}



void fit_wrap(int m, int n, double *a, double *b, double *c, double *d_in){

    #ifdef __HIPCC__
    size_t bufferSizeInBytes;

    hipsparseHandle_t handle;
    void *pBuffer;

    CUSPARSE_CALL(hipsparseCreate(&handle));
    CUSPARSE_CALL( hipsparseDgtsv2StridedBatch_bufferSizeExt(handle, m, a, b, c, d_in, n, m, &bufferSizeInBytes));
    gpuErrchk(hipMalloc(&pBuffer, bufferSizeInBytes));

    CUSPARSE_CALL(hipsparseDgtsv2StridedBatch(handle,
                                              m,
                                              a, // dl
                                              b, //diag
                                              c, // du
                                              d_in,
                                              n,
                                              m,
                                              pBuffer));

  CUSPARSE_CALL(hipsparseDestroy(handle));
  gpuErrchk(hipFree(pBuffer));

  #else

#ifdef __USE_OMP__
#pragma omp parallel for
#endif
for (int j = 0;
     j < n;
     j += 1){
       //fit_constants_serial(m, n, w, a, b, c, d_in, x_in, j);
       int info = LAPACKE_dgtsv(LAPACK_COL_MAJOR, m, 1, &a[j*m + 1], &b[j*m], &c[j*m], &d_in[j*m], m);
       //if (info != m) printf("lapack info check: %d\n", info);

   }

  #endif

}
*/
