#include "hip/hip_runtime.h"
/*  This code was created by Michael Katz.
 *  It is shared under the GNU license (see below).
 *  This is the central piece of code. This file implements a class
 *  that takes data in on the cpu side, copies
 *  it to the gpu, and exposes functions that let
 *  you perform actions with the GPU.
 *
 *  This class will get translated into python via cython.
 *
 *
 *
 *  Copyright (C) 2019 Michael Katz
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with with program; see the file COPYING. If not, write to the
 *  Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 *  MA  02111-1307  USA
 */

#include <kernel.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include "globalPhenomHM.h"
#include <complex>
#include "hip/hip_complex.h"
#include "hipblas.h"
#include "interpolate.cu"
#include "fdresponse.h"
#include "createGPUHolders.cu"
#include "kernel_response.cu"
#include "omp.h"
#include "cuda_complex.hpp"
// TODO: CUTOFF PHASE WHEN IT STARTS TO GO BACK UP!!!

using namespace std;

void print_mem_info(){
        // show memory usage of GPU

        hipError_t cuda_status;

        size_t free_byte ;

        size_t total_byte ;

        cuda_status = hipMemGetInfo( &free_byte, &total_byte ) ;

        if ( hipSuccess != cuda_status ){

            printf("Error: hipMemGetInfo fails, %s \n", hipGetErrorString(cuda_status) );

            exit(1);

        }



        double free_db = (double)free_byte ;

        double total_db = (double)total_byte ;

        double used_db = total_db - free_db ;

        printf("GPU memory usage: used = %f, free = %f MB, total = %f MB\n",

            used_db/1024.0/1024.0, free_db/1024.0/1024.0, total_db/1024.0/1024.0);
}

PhenomHM::PhenomHM (int max_length_init_,
    unsigned int *l_vals_,
    unsigned int *m_vals_,
    int num_modes_,
    double *data_freqs_,
    cmplx *data_channel1_,
    cmplx *data_channel2_,
    cmplx *data_channel3_, int data_stream_length_,
    double *channel1_ASDinv_, double *channel2_ASDinv_, double *channel3_ASDinv_,
    int TDItag_,
    double t_obs_start_,
    double t_obs_end_,
    int nwalkers_,
    int ndevices_){


    max_length_init = max_length_init_;
    l_vals = l_vals_;
    m_vals = m_vals_;
    num_modes = num_modes_;
    data_freqs = data_freqs_;
    data_stream_length = data_stream_length_;
    channel1_ASDinv = channel1_ASDinv_;
    channel2_ASDinv = channel2_ASDinv_;
    channel3_ASDinv = channel3_ASDinv_;
    data_channel1 = data_channel1_;
    data_channel2 = data_channel2_;
    data_channel3 = data_channel3_;
    nwalkers = nwalkers_;

    TDItag = TDItag_;
    t_obs_start = t_obs_start_;
    t_obs_end = t_obs_end_;
    to_gpu = 1;

    ndevices = ndevices_;


    hipError_t err;

    // DECLARE ALL THE  NECESSARY STRUCTS
    pHM_trans = new PhenomHMStorage[nwalkers*ndevices];

    pAmp_trans = new IMRPhenomDAmplitudeCoefficients[nwalkers*ndevices];

    amp_prefactors_trans = new AmpInsPrefactors[nwalkers*ndevices];

    pDPreComp_all_trans = new PhenDAmpAndPhasePreComp[num_modes*nwalkers*ndevices];

    q_all_trans = new HMPhasePreComp[num_modes*nwalkers*ndevices];

    t0 = new double[nwalkers*ndevices];

    phi0 = new double[nwalkers*ndevices];

    amp0 = new double[nwalkers*ndevices];

    // malloc and setup for the GPU

  mode_vals = cpu_create_modes(num_modes, nwalkers*ndevices, l_vals, m_vals, max_length_init, to_gpu, 1);

  // phase shifts for each m mode
  double cShift[7] = {0.0,
                       PI_2 /* i shift */,
                       0.0,
                       -PI_2 /* -i shift */,
                       PI /* 1 shift */,
                       PI_2 /* -1 shift */,
                       0.0};

   H = new cmplx[9*nwalkers*num_modes*ndevices];

   M_tot_sec = new double[nwalkers*ndevices];

  d_mode_vals = new ModeContainer*[ndevices];
  d_freqs = new double*[ndevices];
  d_H = new agcmplx*[ndevices];
  d_B = new double*[ndevices];

  d_template_channel1 = new agcmplx*[ndevices];
  d_template_channel2 = new agcmplx*[ndevices];
  d_template_channel3 = new agcmplx*[ndevices];

  d_data_freqs = new double*[ndevices];

  d_data_channel1 = new agcmplx*[ndevices];
  d_data_channel2 = new agcmplx*[ndevices];
  d_data_channel3 = new agcmplx*[ndevices];

  d_channel1_ASDinv = new double*[ndevices];
  d_channel2_ASDinv = new double*[ndevices];
  d_channel3_ASDinv = new double*[ndevices];

  d_pHM_trans = new PhenomHMStorage*[ndevices];

  d_pAmp_trans = new IMRPhenomDAmplitudeCoefficients*[ndevices];

  d_amp_prefactors_trans = new AmpInsPrefactors*[ndevices];

  d_pDPreComp_all_trans = new PhenDAmpAndPhasePreComp*[ndevices];

  d_q_all_trans = new HMPhasePreComp*[ndevices];

  d_t0 = new double*[ndevices];

  d_phi0 = new double*[ndevices];

  d_amp0 = new double*[ndevices];

  d_M_tot_sec = new double*[ndevices];

  d_cShift = new double*[ndevices];

  d_inc = new double*[ndevices];
  d_lam = new double*[ndevices];
  d_beta = new double*[ndevices];
  d_psi = new double*[ndevices];
  d_t0_epoch = new double*[ndevices];
  d_tRef_wave_frame = new double*[ndevices];
  d_tRef_sampling_frame = new double*[ndevices];
  d_merger_freq = new double*[ndevices];
  d_phiRef = new double*[ndevices];

  handle = new hipblasHandle_t[ndevices];
  interp = new Interpolate[ndevices];

  for (int i=0; i<ndevices; i++){
      hipSetDevice(i);
      d_mode_vals[i] = gpu_create_modes(num_modes, nwalkers, l_vals, m_vals, max_length_init, to_gpu, 1);
      gpuErrchk(hipMalloc(&d_freqs[i], nwalkers*max_length_init*sizeof(double)));

      gpuErrchk(hipMalloc(&d_H[i], 9*num_modes*nwalkers*sizeof(agcmplx)));

      gpuErrchk(hipMalloc(&d_B[i], 8*max_length_init*num_modes*nwalkers*sizeof(double)));

      gpuErrchk(hipMalloc(&d_template_channel1[i], data_stream_length*nwalkers*sizeof(agcmplx)));
      gpuErrchk(hipMalloc(&d_template_channel2[i], data_stream_length*nwalkers*sizeof(agcmplx)));
      gpuErrchk(hipMalloc(&d_template_channel3[i], data_stream_length*nwalkers*sizeof(agcmplx)));

      gpuErrchk(hipMalloc(&d_data_freqs[i], data_stream_length*sizeof(double)));

      gpuErrchk(hipMalloc(&d_data_channel1[i], data_stream_length*sizeof(agcmplx)));

      gpuErrchk(hipMalloc(&d_data_channel2[i], data_stream_length*sizeof(agcmplx)));

      gpuErrchk(hipMalloc(&d_data_channel3[i], data_stream_length*sizeof(agcmplx)));

      gpuErrchk(hipMalloc(&d_channel1_ASDinv[i], data_stream_length*sizeof(double)));

      gpuErrchk(hipMalloc(&d_channel2_ASDinv[i], data_stream_length*sizeof(double)));

      gpuErrchk(hipMalloc(&d_channel3_ASDinv[i], data_stream_length*sizeof(double)));

      gpuErrchk(hipMalloc(&d_pHM_trans[i], nwalkers*sizeof(PhenomHMStorage)));

      gpuErrchk(hipMalloc(&d_pAmp_trans[i], nwalkers*sizeof(IMRPhenomDAmplitudeCoefficients)));

      gpuErrchk(hipMalloc(&d_amp_prefactors_trans[i], nwalkers*sizeof(AmpInsPrefactors)));

      gpuErrchk(hipMalloc(&d_pDPreComp_all_trans[i], num_modes*nwalkers*sizeof(PhenDAmpAndPhasePreComp)));

      gpuErrchk(hipMalloc(&d_q_all_trans[i], num_modes*nwalkers*sizeof(HMPhasePreComp)));

      gpuErrchk(hipMalloc(&d_t0[i], nwalkers*sizeof(double)));

      //double phi0_;
      gpuErrchk(hipMalloc(&d_phi0[i], nwalkers*sizeof(double)));

      //double amp0_;
      gpuErrchk(hipMalloc(&d_amp0[i], nwalkers*sizeof(double)));

      gpuErrchk(hipMalloc(&d_M_tot_sec[i], nwalkers*sizeof(double)));

      gpuErrchk(hipMalloc(&d_cShift[i], 7*sizeof(double)));
      gpuErrchk(hipMemcpy(d_cShift[i], &cShift, 7*sizeof(double), hipMemcpyHostToDevice));

      gpuErrchk(hipMalloc(&d_inc[i], nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&d_lam[i], nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&d_beta[i], nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&d_psi[i], nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&d_t0_epoch[i], nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&d_tRef_wave_frame[i], nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&d_tRef_sampling_frame[i], nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&d_merger_freq[i], nwalkers*sizeof(double)));
      gpuErrchk(hipMalloc(&d_phiRef[i], nwalkers*sizeof(double)));

      // for likelihood
      // --------------
      stat = hipblasCreate(&handle[i]);
      if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("CUBLAS initialization failed\n");
              exit(0);
          }
          // ----------------

    // initialize values needed for GPU waveform creation
      //double t0_;

      // alocate GPU arrays for interpolation
      interp[i].alloc_arrays(max_length_init, 8*num_modes*nwalkers, d_B[i]);
  }

  PhenomHM::input_data(data_freqs, data_channel1,
                        data_channel2, data_channel3,
                        channel1_ASDinv, channel2_ASDinv,
                        channel3_ASDinv, data_stream_length);
}


void PhenomHM::input_data(double *data_freqs, cmplx *data_channel1,
                          cmplx *data_channel2, cmplx *data_channel3,
                          double *channel1_ASDinv, double *channel2_ASDinv,
                          double *channel3_ASDinv, int data_stream_length_){

    assert(data_stream_length_ == data_stream_length);

    for (int i=0; i<ndevices; i++){
        hipSetDevice(i);
        gpuErrchk(hipMemcpy(d_data_freqs[i], data_freqs, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_data_channel1[i], data_channel1, data_stream_length*sizeof(agcmplx), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_data_channel2[i], data_channel2, data_stream_length*sizeof(agcmplx), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_data_channel3[i], data_channel3, data_stream_length*sizeof(agcmplx), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_channel1_ASDinv[i], channel1_ASDinv, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_channel2_ASDinv[i], channel2_ASDinv, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

        gpuErrchk(hipMemcpy(d_channel3_ASDinv[i], channel3_ASDinv, data_stream_length*sizeof(double), hipMemcpyHostToDevice));
    }
}

/*
generate gpu amp and phase
*/
void PhenomHM::gen_amp_phase(double *freqs_, int current_length_,
    double* m1_, //solar masses
    double* m2_, //solar masses
    double* chi1z_,
    double* chi2z_,
    double* distance_,
    double* phiRef_,
    double* f_ref_){

    assert(to_gpu == 1);
    assert(current_length_ <= nwalkers*max_length_init);

    freqs = freqs_;
    //printf("fsss: %e, %e\n", freqs[1], freqs[0]);
    d_log10f = log10(freqs[1]) - log10(freqs[0]);
    current_length = current_length_;
    m1 = m1_; //solar masses
    m2 = m2_; //solar masses
    chi1z = chi1z_;
    chi2z = chi2z_;
    distance = distance_;
    phiRef = phiRef_;
    f_ref = f_ref_;

    int i, th_id, nthreads;
    #pragma omp parallel private(th_id, i)
    {
    //for (int i=0; i<ndevices*nwalkers; i++){
        nthreads = omp_get_num_threads();
        th_id = omp_get_thread_num();
        for (int i=th_id; i<ndevices*nwalkers; i+=nthreads){
            PhenomHM::gen_amp_phase_prep(i, &freqs[i*current_length], current_length_,
                m1_[i], //solar masses
                m2_[i], //solar masses
                chi1z_[i],
                chi2z_[i],
                distance_[i],
                phiRef_[i],
                f_ref_[i]);

            M_tot_sec[i] = (m1[i]+m2[i])*MTSUN_SI;
      }
    }

    /* main: evaluate model at given frequencies on GPU */
    NUM_THREADS = 256;

    num_blocks = std::ceil((current_length + NUM_THREADS -1)/NUM_THREADS);
    dim3 gridDim(num_blocks, num_modes, nwalkers);
    //printf("%d walkers \n", nwalkers);

    #pragma omp parallel private(th_id, i)
    {
    //for (int i=0; i<nwalkers; i++){
        nthreads = omp_get_num_threads();
        th_id = omp_get_thread_num();
        for (int i=th_id; i<ndevices; i+=nthreads){
            hipSetDevice(i);

            // copy everything to GPU
            gpuErrchk(hipMemcpy(d_freqs[i], &freqs[i*nwalkers*current_length], nwalkers*current_length*sizeof(double), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(d_pHM_trans[i], &pHM_trans[i*nwalkers], nwalkers*sizeof(PhenomHMStorage), hipMemcpyHostToDevice));

            //printf("%.12e, %.12e, %.12e, %.12e\n\n", pHM_trans[2].m1, pHM_trans[2].m2, m1[2], m2[2]);

            gpuErrchk(hipMemcpy(d_pAmp_trans[i], &pAmp_trans[i*nwalkers], nwalkers*sizeof(IMRPhenomDAmplitudeCoefficients), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(d_amp_prefactors_trans[i], &amp_prefactors_trans[i*nwalkers], nwalkers*sizeof(AmpInsPrefactors), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(d_pDPreComp_all_trans[i], &pDPreComp_all_trans[i*nwalkers*num_modes], nwalkers*num_modes*sizeof(PhenDAmpAndPhasePreComp), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(d_q_all_trans[i], &q_all_trans[i*nwalkers*num_modes], nwalkers*num_modes*sizeof(HMPhasePreComp), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(d_t0[i], &t0[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(d_phi0[i], &phi0[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(d_amp0[i], &amp0[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));

            gpuErrchk(hipMemcpy(d_M_tot_sec[i], &M_tot_sec[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));

            kernel_calculate_all_modes<<<gridDim, NUM_THREADS>>>(d_mode_vals[i],
                  d_pHM_trans[i],
                  d_freqs[i],
                  d_M_tot_sec[i],
                  d_pAmp_trans[i],
                  d_amp_prefactors_trans[i],
                  d_pDPreComp_all_trans[i],
                  d_q_all_trans[i],
                  d_amp0[i],
                  num_modes,
                  d_t0[i],
                  d_phi0[i],
                  d_cShift[i],
                  nwalkers,
                  current_length
              );
              hipDeviceSynchronize();
              gpuErrchk(hipGetLastError());
        }
    }
    //printf("intrinsic: %e, %e, %e, %e, %e, %e, %e\n", m1, m2, chi1z, chi2z, distance, phiRef, f_ref);



     // ensure calls are run in correct order
     current_status = 1;
}

/*
generate structures for GPU creation of amp and phase
*/
void PhenomHM::gen_amp_phase_prep(int ind_walker, double *freqs_gen, int current_length,
    double m1_gen, //solar masses
    double m2_gen, //solar masses
    double chi1z_gen,
    double chi2z_gen,
    double distance_gen,
    double phiRef_gen,
    double f_ref_gen){

    double m1_SI, m2_SI, deltaF;

    // for phenomHM internal calls
    deltaF = -1.0;
    for (int i=0; i<num_modes; i++){
        mode_vals[ind_walker*num_modes + i].length = current_length;
    }

    m1_SI = m1_gen*MSUN_SI;
    m2_SI = m2_gen*MSUN_SI;

    /* fill all the structures necessary for waveform creation */
    retcode = 0;
    retcode = IMRPhenomHMCore(
        mode_vals,
        freqs_gen,
        current_length,
        m1_SI,
        m2_SI,
        chi1z_gen,
        chi2z_gen,
        distance_gen,
        phiRef_gen,
        deltaF,
        f_ref_gen,
        num_modes,
        to_gpu,
        &pHM_trans[ind_walker],
        &pAmp_trans[ind_walker],
        &amp_prefactors_trans[ind_walker],
        &pDPreComp_all_trans[ind_walker*num_modes],
        &q_all_trans[ind_walker*num_modes],
        &t0[ind_walker],
        &phi0[ind_walker],
        &amp0[ind_walker]);
    //assert (retcode == 1); //,PD_EFUNC, "IMRPhenomHMCore failed in
      //printf("%d, %.12e, %.12e, %.12e, %.12e, %.12e, %.12e, %.12e\n\n", ind_walker, pHM_trans[ind_walker].m1, pHM_trans[ind_walker].m2, m1[ind_walker], m2[ind_walker], t0[ind_walker], phi0[ind_walker], amp0[ind_walker]);

}


/*
Setup interpolation of amp and phase
*/
void PhenomHM::setup_interp_wave(){

    assert(current_status >= 2);
    dim3 waveInterpDim(num_blocks, num_modes*nwalkers);

    int i, th_id, nthreads;
    #pragma omp parallel private(th_id, i)
    {
    //for (int i=0; i<nwalkers; i++){
        nthreads = omp_get_num_threads();
        th_id = omp_get_thread_num();
        for (int i=th_id; i<ndevices; i+=nthreads){
            hipSetDevice(i);
            // fill B array
            fill_B_wave<<<waveInterpDim, NUM_THREADS>>>(d_mode_vals[i], d_B[i], current_length, num_modes*nwalkers);
            hipDeviceSynchronize();
            gpuErrchk(hipGetLastError());

            // perform interpolation
            interp[i].prep(d_B[i], current_length, 2*num_modes*nwalkers, 1);
            hipDeviceSynchronize();
            gpuErrchk(hipGetLastError());

            set_spline_constants_wave<<<waveInterpDim, NUM_THREADS>>>(d_mode_vals[i], d_B[i], current_length, num_modes*nwalkers);
            hipDeviceSynchronize();
            gpuErrchk(hipGetLastError());
        }
    }

    if (current_status == 2) current_status = 3;
}

/*
Get LISA fast Fourier domain response on GPU
*/
__global__
void check_response(ModeContainer *mode_vals, int num_modes, int nwalkers, int length){
    double orig_val, val;
    int mode_ind;
    for (int i=0; i< length; i++){

    for (int mode_i=0; mode_i<num_modes; mode_i++){
        for (int j=0; j<nwalkers; j++){
            mode_ind = j*num_modes + mode_i;
            if (j==0) orig_val = mode_vals[mode_ind].transferL1_re[i];
            else {
                val = mode_vals[mode_ind].transferL1_re[i];
                if (val != orig_val){
                    # if __CUDA_ARCH__>=200
                        printf("%d, %d, %d, %.12e, %.12e\n", j, mode_i, i, val, orig_val);
                    #endif //*/
                }
            }
        }
    }
    }



}

void PhenomHM::LISAresponseFD(double* inc_, double* lam_, double* beta_, double* psi_, double* t0_epoch_, double* tRef_wave_frame_, double* tRef_sampling_frame_, double* merger_freq_){
    inc = inc_;
    lam = lam_;
    beta = beta_;
    psi = psi_;
    t0_epoch = t0_epoch_;
    tRef_wave_frame = tRef_wave_frame_;
    tRef_sampling_frame = tRef_sampling_frame_;
    merger_freq = merger_freq_;

    //printf("extrinsic: %e, %e, %e, %e, %e, %e, %e\n", inc, lam, beta, psi, t0_epoch, tRef_wave_frame, tRef_sampling_frame, merger_freq);

    assert(current_status >= 1);
    // get H on the CPU
    int i, th_id, nthreads;
    #pragma omp parallel private(th_id, i)
    {
    //for (int i=0; i<nwalkers; i++){
        nthreads = omp_get_num_threads();
        th_id = omp_get_thread_num();
        for (int i=th_id; i<ndevices*nwalkers; i+=nthreads){
            prep_H_info(&H[i*num_modes*9], l_vals, m_vals, num_modes, inc[i], lam[i], beta[i], psi[i], phiRef[i]);
        }
    }

    double d_log10f = log10(freqs[1]) - log10(freqs[0]);
    //printf("fs2: %e, %e, %e\n", freqs[1], freqs[0], d_log10f);
    int num_blocks = std::ceil((current_length + NUM_THREADS - 1)/NUM_THREADS);
    dim3 gridDim(num_blocks, num_modes, nwalkers);

    #pragma omp parallel private(th_id, i)
    {
    //for (int i=0; i<nwalkers; i++){
        nthreads = omp_get_num_threads();
        th_id = omp_get_thread_num();
        for (int i=th_id; i<ndevices; i+=nthreads){
            hipSetDevice(i);
            gpuErrchk(hipMemcpy(d_H[i], &H[i*9*num_modes*nwalkers], 9*num_modes*nwalkers*sizeof(agcmplx), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_inc[i], &inc[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_lam[i], &lam[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_beta[i], &beta[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_psi[i], &psi[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_phiRef[i], &phiRef[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_t0_epoch[i], &t0_epoch[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_tRef_wave_frame[i], &tRef_wave_frame[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_tRef_sampling_frame[i], &tRef_sampling_frame[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_merger_freq[i], &merger_freq[i*nwalkers], nwalkers*sizeof(double), hipMemcpyHostToDevice));

            int num_blocks = std::ceil((current_length + NUM_THREADS - 1)/NUM_THREADS);

            // Perform response

            kernel_JustLISAFDresponseTDI_wrap<<<gridDim, NUM_THREADS>>>(d_mode_vals[i], (hipDoubleComplex*)d_H[i], d_freqs[i], d_freqs[i], d_log10f, d_l_vals, d_m_vals,
                        num_modes, current_length, d_inc[i], d_lam[i], d_beta[i], d_psi[i], d_phiRef[i], d_t0_epoch[i],
                        d_tRef_wave_frame[i], d_tRef_sampling_frame[i], d_merger_freq[i], TDItag, 0, nwalkers);
            hipDeviceSynchronize();
            gpuErrchk(hipGetLastError());

            kernel_add_tRef_phase_shift<<<gridDim, NUM_THREADS>>>(d_mode_vals[i], d_freqs[i],
                        num_modes, current_length, d_tRef_wave_frame[i], nwalkers);
            hipDeviceSynchronize();
            gpuErrchk(hipGetLastError());
        }
    }

    /*hipSetDevice(0);
    check_response<<<1,1>>>(d_mode_vals[0], num_modes, nwalkers, current_length);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());*/

    if (current_status == 1) current_status = 2;
}

/*
setup interpolation for the response transfer functions
*/
void PhenomHM::setup_interp_response(){

    assert(current_status >= 3);

    dim3 responseInterpDim(num_blocks, num_modes*nwalkers);

    int i, th_id, nthreads;
    #pragma omp parallel private(th_id, i)
    {
    //for (int i=0; i<nwalkers; i++){
        nthreads = omp_get_num_threads();
        th_id = omp_get_thread_num();
        for (int i=th_id; i<ndevices; i+=nthreads){
            hipSetDevice(i);
            fill_B_response<<<responseInterpDim, NUM_THREADS>>>(d_mode_vals[i], d_B[i], current_length, num_modes*nwalkers);
            hipDeviceSynchronize();
            gpuErrchk(hipGetLastError());

            interp[i].prep(d_B[i], current_length, 8*num_modes*nwalkers, 1);  // TODO check the 8?
            hipDeviceSynchronize();
            gpuErrchk(hipGetLastError());

            set_spline_constants_response<<<responseInterpDim, NUM_THREADS>>>(d_mode_vals[i], d_B[i], current_length, num_modes*nwalkers);
            hipDeviceSynchronize();
            gpuErrchk(hipGetLastError());
        }
    }
    if (current_status == 3) current_status = 4;
}

/*
interpolate amp and phase up to frequencies of the data stream.
*/
void PhenomHM::perform_interp(){
    assert(current_status >= 4);
    int num_block_interp = std::ceil((data_stream_length + NUM_THREADS - 1)/NUM_THREADS);
    //dim3 mainInterpDim(num_block_interp, 1, nwalkers);//, num_modes);
    dim3 mainInterpDim(num_block_interp, 1, nwalkers);//, num_modes);

    int i, th_id, nthreads;
    #pragma omp parallel private(th_id, i)
    {
    //for (int i=0; i<nwalkers; i++){
        nthreads = omp_get_num_threads();
        th_id = omp_get_thread_num();
        for (int i=th_id; i<ndevices; i+=nthreads){
            hipSetDevice(i);
            interpolate<<<mainInterpDim, NUM_THREADS>>>(d_template_channel1[i], d_template_channel2[i], d_template_channel3[i], d_mode_vals[i], num_modes,
                d_log10f, d_freqs[i], current_length, d_data_freqs[i], data_stream_length, d_t0_epoch[i],
                d_tRef_sampling_frame[i], d_channel1_ASDinv[i], d_channel2_ASDinv[i], d_channel3_ASDinv[i], t_obs_start, t_obs_end, nwalkers);
            hipDeviceSynchronize();
            gpuErrchk(hipGetLastError());
        }
    }
    if (current_status == 4) current_status = 5;
}

/*
Compute likelihood on the GPU
*/
void PhenomHM::Likelihood (double *d_h_arr, double *h_h_arr){

    //printf("like mem\n");
    //print_mem_info();
     assert(current_status == 5);

     //#pragma omp parallel
     //{
     //for (int i=0; i<nwalkers; i++){
        int j, i, th_id, nthreads;
         double d_h = 0.0;
         double h_h = 0.0;
         char * status;
         double res;
         hipDoubleComplex result;
         //nthreads = omp_get_num_threads();
         //th_id = omp_get_thread_num();
         for (int j=0; j<ndevices; j+=1){
             hipSetDevice(j);
             for (int i=0; i<nwalkers; i++){
                 d_h = 0.0;
                 h_h = 0.0;
                 // get data - template terms
                  stat = hipblasZdotc(handle[j], data_stream_length,
                          (hipDoubleComplex*)&d_template_channel1[j][data_stream_length*i], 1,
                          (hipDoubleComplex*)d_data_channel1[j], 1,
                          &result);
                  status = _cudaGetErrorEnum(stat);
                   hipDeviceSynchronize();

                   if (stat != HIPBLAS_STATUS_SUCCESS) {
                           exit(0);
                       }
                  d_h += hipCreal(result);
                  //printf("channel1 d_h: %e\n", hipCreal(result));

                  stat = hipblasZdotc(handle[j], data_stream_length,
                          (hipDoubleComplex*)&d_template_channel2[j][data_stream_length*i], 1,
                          (hipDoubleComplex*)d_data_channel2[j], 1,
                          &result);
                  status = _cudaGetErrorEnum(stat);
                   hipDeviceSynchronize();

                   if (stat != HIPBLAS_STATUS_SUCCESS) {
                           exit(0);
                       }
                  d_h += hipCreal(result);
                  //printf("channel2 d_h: %e\n", hipCreal(result));

                  stat = hipblasZdotc(handle[j], data_stream_length,
                          (hipDoubleComplex*)&d_template_channel3[j][data_stream_length*i], 1,
                          (hipDoubleComplex*)d_data_channel3[j], 1,
                          &result);
                  status = _cudaGetErrorEnum(stat);
                   hipDeviceSynchronize();

                   if (stat != HIPBLAS_STATUS_SUCCESS) {
                           exit(0);
                       }
                  d_h += hipCreal(result);
                  //printf("channel3 d_h: %e\n", hipCreal(result));


                  // get template template terms
                 stat = hipblasZdotc(handle[j], data_stream_length,
                              (hipDoubleComplex*)&d_template_channel1[j][data_stream_length*i], 1,
                              (hipDoubleComplex*)&d_template_channel1[j][data_stream_length*i], 1,
                              &result);
                      status = _cudaGetErrorEnum(stat);
                       hipDeviceSynchronize();

                       if (stat != HIPBLAS_STATUS_SUCCESS) {
                               exit(0);
                           }
                      h_h += hipCreal(result);
                      //printf("channel1 h_h: %e\n", hipCreal(result));

                      stat = hipblasZdotc(handle[j], data_stream_length,
                              (hipDoubleComplex*)&d_template_channel2[j][data_stream_length*i], 1,
                              (hipDoubleComplex*)&d_template_channel2[j][data_stream_length*i], 1,
                              &result);
                      status = _cudaGetErrorEnum(stat);
                       hipDeviceSynchronize();

                       if (stat != HIPBLAS_STATUS_SUCCESS) {
                               exit(0);
                           }
                      h_h += hipCreal(result);
                      //printf("channel2 h_h: %e\n", hipCreal(result));

                      stat = hipblasZdotc(handle[j], data_stream_length,
                              (hipDoubleComplex*)&d_template_channel3[j][data_stream_length*i], 1,
                              (hipDoubleComplex*)&d_template_channel3[j][data_stream_length*i], 1,
                              &result);
                      status = _cudaGetErrorEnum(stat);
                       hipDeviceSynchronize();

                       if (stat != HIPBLAS_STATUS_SUCCESS) {
                               exit(0);
                           }
                      h_h += hipCreal(result);
                      //printf("channel3 h_h: %e\n", hipCreal(result));
                  d_h_arr[j*nwalkers + i] = 4*d_h;
                  h_h_arr[j*nwalkers + i] = 4*h_h;
             }
        }
    //}
}

/*
Copy TDI channels to CPU and return to python.
*/
void PhenomHM::GetTDI (cmplx* channel1_, cmplx* channel2_, cmplx* channel3_) {

  assert(current_status > 4);
  for (int i=0; i<ndevices; i++){
      gpuErrchk(hipSetDevice(i));
      gpuErrchk(hipMemcpy(&channel1_[i*nwalkers*data_stream_length], d_template_channel1[i], data_stream_length*nwalkers*sizeof(cmplx), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(&channel2_[i*nwalkers*data_stream_length], d_template_channel2[i], data_stream_length*nwalkers*sizeof(cmplx), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(&channel3_[i*nwalkers*data_stream_length], d_template_channel3[i], data_stream_length*nwalkers*sizeof(cmplx), hipMemcpyDeviceToHost));
  }
}

/*
auxillary function for getting amplitude and phase to the CPU
*/
__global__ void read_out_amp_phase(ModeContainer *mode_vals, double *amp, double *phase, int num_modes, int length){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int mode_i = blockIdx.y;
    if (i >= length) return;
    if (mode_i >= num_modes) return;
    amp[mode_i*length + i] = mode_vals[mode_i].amp[i];
    phase[mode_i*length + i] = mode_vals[mode_i].phase[i];
}

/*
Return amplitude and phase in python on CPU
*/
void PhenomHM::GetAmpPhase(double* amp_, double* phase_) {
  assert(current_status >= 1);
  double *amp, *phase;


  dim3 readOutDim(num_blocks, num_modes*nwalkers);
  for (int i=0; i<ndevices; i++){
      hipSetDevice(i);
      gpuErrchk(hipMalloc(&amp, nwalkers*num_modes*current_length*sizeof(double)));
      gpuErrchk(hipMalloc(&phase, nwalkers*num_modes*current_length*sizeof(double)));
      read_out_amp_phase<<<readOutDim, NUM_THREADS>>>(d_mode_vals[i], amp, phase, nwalkers*num_modes, current_length);
      hipDeviceSynchronize();
      gpuErrchk(hipGetLastError());
      gpuErrchk(hipMemcpy(&amp_[i*nwalkers*num_modes*current_length], amp, nwalkers*num_modes*current_length*sizeof(double), hipMemcpyDeviceToHost));
      gpuErrchk(hipMemcpy(&phase_[i*nwalkers*num_modes*current_length], phase, nwalkers*num_modes*current_length*sizeof(double), hipMemcpyDeviceToHost));
      gpuErrchk( hipFree(amp));
      gpuErrchk(hipFree(phase));
  }
}

/*
Destructor
*/
PhenomHM::~PhenomHM() {
  delete[] pHM_trans;
  delete[] pAmp_trans;
  delete[] amp_prefactors_trans;
  delete[] pDPreComp_all_trans;
  delete[] q_all_trans;
  delete[] t0;
  delete[] amp0;
  delete[] phi0;
  delete[] M_tot_sec;
  cpu_destroy_modes(mode_vals);
  delete[] H;

  gpuErrchk(hipFree(d_data_freqs));
  for (int i=0; i<ndevices; i++){
      gpuErrchk(hipFree(d_freqs[i]));
      gpu_destroy_modes(d_mode_vals[i]);

      gpuErrchk(hipFree(d_pHM_trans[i]));
      gpuErrchk(hipFree(d_pAmp_trans[i]));
      gpuErrchk(hipFree(d_amp_prefactors_trans[i]));
      gpuErrchk(hipFree(d_pDPreComp_all_trans[i]));
      gpuErrchk(hipFree(d_q_all_trans[i]));
      gpuErrchk(hipFree(d_cShift[i]));

      gpuErrchk(hipFree(d_data_channel1[i]));
      gpuErrchk(hipFree(d_data_channel2[i]));
      gpuErrchk(hipFree(d_data_channel3[i]));

      gpuErrchk(hipFree(d_template_channel1[i]));
      gpuErrchk(hipFree(d_template_channel2[i]));
      gpuErrchk(hipFree(d_template_channel3));

      gpuErrchk(hipFree(d_channel1_ASDinv[i]));
      gpuErrchk(hipFree(d_channel2_ASDinv[i]));
      gpuErrchk(hipFree(d_channel3_ASDinv[i]));
      hipblasDestroy(handle[i]);
      gpuErrchk(hipFree(d_B[i]));
      gpuErrchk(hipFree(d_t0[i]));
      gpuErrchk(hipFree(d_phi0[i]));
      gpuErrchk(hipFree(d_amp0[i]));
      gpuErrchk(hipFree(d_M_tot_sec[i]));

      gpuErrchk(hipFree(d_inc[i]));
      gpuErrchk(hipFree(d_lam[i]));
      gpuErrchk(hipFree(d_beta[i]));
      gpuErrchk(hipFree(d_psi[i]));
      gpuErrchk(hipFree(d_t0_epoch[i]));
      gpuErrchk(hipFree(d_tRef_wave_frame[i]));
      gpuErrchk(hipFree(d_tRef_sampling_frame[i]));
      gpuErrchk(hipFree(d_merger_freq[i]));
      gpuErrchk(hipFree(d_phiRef[i]));
  }

  delete[] d_freqs;
  delete[] d_mode_vals;

  delete[] d_pHM_trans;
  delete[] d_pAmp_trans;
  delete[] d_amp_prefactors_trans;
  delete[] d_pDPreComp_all_trans;
  delete[] d_q_all_trans;
  delete[] d_cShift;

  delete[] d_data_channel1;
  delete[] d_data_channel2;
  delete[] d_data_channel3;

  delete[] d_template_channel1;
  delete[] d_template_channel2;
  delete[] d_template_channel3;

  delete[] d_channel1_ASDinv;
  delete[] d_channel2_ASDinv;
  delete[] d_channel3_ASDinv;
  delete[] handle;
  delete[] d_B;
  delete[] d_t0;
  delete[] d_phi0;
  delete[] d_amp0;
  delete[] d_M_tot_sec;

  delete[] d_inc;
  delete[] d_lam;
  delete[] d_beta;
  delete[] d_psi;
  delete[] d_t0_epoch;
  delete[] d_tRef_wave_frame;
  delete[] d_tRef_sampling_frame;
  delete[] d_merger_freq;
  delete[] d_phiRef;

    delete[] handle;
    delete[] interp;
}
