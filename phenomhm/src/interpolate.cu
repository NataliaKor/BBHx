#include "hip/hip_runtime.h"
/*  This code was created by Michael Katz.
 *  It is shared under the GNU license (see below).
 *  This code computes the interpolations for the GPU PhenomHM waveform.
 *
 *
 *  Copyright (C) 2019 Michael Katz
 *
 *  This program is free software; you can redistribute it and/or modify
 *  it under the terms of the GNU General Public License as published by
 *  the Free Software Foundation; either version 2 of the License, or
 *  (at your option) any later version.
 *
 *  This program is distributed in the hope that it will be useful,
 *  but WITHOUT ANY WARRANTY; without even the implied warranty of
 *  MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 *  GNU General Public License for more details.
 *
 *  You should have received a copy of the GNU General Public License
 *  along with with program; see the file COPYING. If not, write to the
 *  Free Software Foundation, Inc., 59 Temple Place, Suite 330, Boston,
 *  MA  02111-1307  USA
 */

#include "manager.hh"
#include "stdio.h"
#include <assert.h>
#include "globalPhenomHM.h"
#include "interpolate.hh"


#ifdef __HIPCC__
/*
GPU error checking
*/
#define gpuErrchk_here(ans) { gpuAssert_here((ans), __FILE__, __LINE__); }
inline void gpuAssert_here(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

#endif

/*
fill the B array on the GPU for response transfer functions.
*/
CUDA_CALLABLE_MEMBER
void fill_B_response(ModeContainer *mode_vals, double *B, int f_length, int num_modes, int mode_i, int i){
    int num_pars = 8;
            if (i == f_length - 1){

                B[(i*num_pars*num_modes) + 0*num_modes + mode_i] = 3.0* (mode_vals[mode_i].phaseRdelay[i] - mode_vals[mode_i].phaseRdelay[i-1]);
                B[(i*num_pars*num_modes) + 1*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL1_re[i] - mode_vals[mode_i].transferL1_re[i-1]);
                B[(i*num_pars*num_modes) + 2*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL1_im[i] - mode_vals[mode_i].transferL1_im[i-1]);
                B[(i*num_pars*num_modes) + 3*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL2_re[i] - mode_vals[mode_i].transferL2_re[i-1]);
                B[(i*num_pars*num_modes) + 4*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL2_im[i] - mode_vals[mode_i].transferL2_im[i-1]);
                B[(i*num_pars*num_modes) + 5*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL3_re[i] - mode_vals[mode_i].transferL3_re[i-1]);
                B[(i*num_pars*num_modes) + 6*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL3_im[i] - mode_vals[mode_i].transferL3_im[i-1]);
                B[(i*num_pars*num_modes) + 7*num_modes + mode_i] = 3.0* (mode_vals[mode_i].time_freq_corr[i] - mode_vals[mode_i].time_freq_corr[i-1]);

            } else if (i == 0){
                B[(i*num_pars*num_modes) + 0*num_modes + mode_i] = 3.0* (mode_vals[mode_i].phaseRdelay[1] - mode_vals[mode_i].phaseRdelay[0]);
                B[(i*num_pars*num_modes) + 1*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL1_re[1] - mode_vals[mode_i].transferL1_re[0]);
                B[(i*num_pars*num_modes) + 2*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL1_im[1] - mode_vals[mode_i].transferL1_im[0]);
                B[(i*num_pars*num_modes) + 3*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL2_re[1] - mode_vals[mode_i].transferL2_re[0]);
                B[(i*num_pars*num_modes) + 4*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL2_im[1] - mode_vals[mode_i].transferL2_im[0]);
                B[(i*num_pars*num_modes) + 5*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL3_re[1] - mode_vals[mode_i].transferL3_re[0]);
                B[(i*num_pars*num_modes) + 6*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL3_im[1] - mode_vals[mode_i].transferL3_im[0]);
                B[(i*num_pars*num_modes) + 7*num_modes + mode_i] = 3.0* (mode_vals[mode_i].time_freq_corr[1] - mode_vals[mode_i].time_freq_corr[0]);
            } else{
                B[(i*num_pars*num_modes) + 0*num_modes + mode_i] = 3.0* (mode_vals[mode_i].phaseRdelay[i+1] - mode_vals[mode_i].phaseRdelay[i-1]);
                B[(i*num_pars*num_modes) + 1*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL1_re[i+1] - mode_vals[mode_i].transferL1_re[i-1]);
                B[(i*num_pars*num_modes) + 2*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL1_im[i+1] - mode_vals[mode_i].transferL1_im[i-1]);
                B[(i*num_pars*num_modes) + 3*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL2_re[i+1] - mode_vals[mode_i].transferL2_re[i-1]);
                B[(i*num_pars*num_modes) + 4*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL2_im[i+1] - mode_vals[mode_i].transferL2_im[i-1]);
                B[(i*num_pars*num_modes) + 5*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL3_re[i+1] - mode_vals[mode_i].transferL3_re[i-1]);
                B[(i*num_pars*num_modes) + 6*num_modes + mode_i] = 3.0* (mode_vals[mode_i].transferL3_im[i+1] - mode_vals[mode_i].transferL3_im[i-1]);
                B[(i*num_pars*num_modes) + 7*num_modes + mode_i] = 3.0* (mode_vals[mode_i].time_freq_corr[i+1] - mode_vals[mode_i].time_freq_corr[i-1]);
            }
}

#ifdef __HIPCC__
CUDA_KERNEL
void fill_B_response_wrap(ModeContainer *mode_vals, double *B, int f_length, int num_modes){
    int num_pars = 8;
    for (int mode_i = blockIdx.y * blockDim.y + threadIdx.y;
         mode_i < num_modes;
         mode_i += blockDim.y * gridDim.y){

       for (int i = blockIdx.x * blockDim.x + threadIdx.x;
            i < f_length;
            i += blockDim.x * gridDim.x){

              fill_B_response(mode_vals, B, f_length, num_modes, mode_i, i);

}
}
}
#else
void cpu_fill_B_response_wrap(ModeContainer *mode_vals, double *B, int f_length, int num_modes){
    int num_pars = 8;
    for (int mode_i = 0;
         mode_i < num_modes;
         mode_i += 1){

       for (int i = 0;
            i < f_length;
            i += 1){

              fill_B_response(mode_vals, B, f_length, num_modes, mode_i, i);

}
}
}
#endif

/*
fill B array on GPU for amp and phase
*/
CUDA_CALLABLE_MEMBER void fill_B_wave(ModeContainer *mode_vals, double *B, int f_length, int num_modes, int mode_i, int i){
    int num_pars = 2;
    if (i == f_length - 1){
        B[i*num_modes + mode_i] = 3.0* (mode_vals[mode_i].amp[i] - mode_vals[mode_i].amp[i-1]);
        B[(num_modes*f_length) + i*num_modes + mode_i] = 3.0* (mode_vals[mode_i].phase[i] - mode_vals[mode_i].phase[i-1]);
    } else if (i == 0){
        B[i*num_modes + mode_i] = 3.0* (mode_vals[mode_i].amp[1] - mode_vals[mode_i].amp[0]);
        B[(num_modes*f_length) + i*num_modes + mode_i] = 3.0* (mode_vals[mode_i].phase[1] - mode_vals[mode_i].phase[0]);
    } else{
        B[i*num_modes + mode_i] = 3.0* (mode_vals[mode_i].amp[i+1] - mode_vals[mode_i].amp[i-1]);
        B[(num_modes*f_length) + i*num_modes + mode_i] = 3.0* (mode_vals[mode_i].phase[i+1] - mode_vals[mode_i].phase[i-1]);
    }
}

#ifdef __HIPCC__
CUDA_KERNEL void fill_B_wave_wrap(ModeContainer *mode_vals, double *B, int f_length, int num_modes){
    int num_pars = 2;
    for (int mode_i = blockIdx.y * blockDim.y + threadIdx.y;
         mode_i < num_modes;
         mode_i += blockDim.y * gridDim.y){

       for (int i = blockIdx.x * blockDim.x + threadIdx.x;
            i < f_length;
            i += blockDim.x * gridDim.x){

              fill_B_wave(mode_vals, B, f_length, num_modes, mode_i, i);

}
}
}
#else
void cpu_fill_B_wave_wrap(ModeContainer *mode_vals, double *B, int f_length, int num_modes){

    int num_pars = 2;
    for (int mode_i = 0;
         mode_i < num_modes;
         mode_i += 1){

       for (int i = 0;
            i < f_length;
            i += 1){
              fill_B_wave(mode_vals, B, f_length, num_modes, mode_i, i);

}
}
}
#endif

/*
find spline constants based on matrix solution for response transfer functions.
*/
CUDA_CALLABLE_MEMBER
void set_spline_constants_response(ModeContainer *mode_vals, double *B, int f_length, int num_modes, int mode_i, int i){
    double D_i, D_ip1, y_i, y_ip1;
    int num_pars = 8;

            D_i = B[(i*num_pars*num_modes) + 0*num_modes + mode_i];
            D_ip1 = B[((i+1)*num_pars*num_modes) + 0*num_modes + mode_i];
            y_i = mode_vals[mode_i].phaseRdelay[i];
            y_ip1 = mode_vals[mode_i].phaseRdelay[i+1];
            mode_vals[mode_i].phaseRdelay_coeff_1[i] = D_i;
            mode_vals[mode_i].phaseRdelay_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
            mode_vals[mode_i].phaseRdelay_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;

            D_i = B[(i*num_pars*num_modes) + 1*num_modes + mode_i];
            D_ip1 = B[((i+1)*num_pars*num_modes) + 1*num_modes + mode_i];
            y_i = mode_vals[mode_i].transferL1_re[i];
            y_ip1 = mode_vals[mode_i].transferL1_re[i+1];
            mode_vals[mode_i].transferL1_re_coeff_1[i] = D_i;
            mode_vals[mode_i].transferL1_re_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
            mode_vals[mode_i].transferL1_re_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;

            D_i = B[(i*num_pars*num_modes) + 2*num_modes + mode_i];
            D_ip1 = B[((i+1)*num_pars*num_modes) + 2*num_modes + mode_i];
            y_i = mode_vals[mode_i].transferL1_im[i];
            y_ip1 = mode_vals[mode_i].transferL1_im[i+1];
            mode_vals[mode_i].transferL1_im_coeff_1[i] = D_i;
            mode_vals[mode_i].transferL1_im_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
            mode_vals[mode_i].transferL1_im_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;

            D_i = B[(i*num_pars*num_modes) + 3*num_modes + mode_i];
            D_ip1 = B[((i+1)*num_pars*num_modes) + 3*num_modes + mode_i];
            y_i = mode_vals[mode_i].transferL2_re[i];
            y_ip1 = mode_vals[mode_i].transferL2_re[i+1];
            mode_vals[mode_i].transferL2_re_coeff_1[i] = D_i;
            mode_vals[mode_i].transferL2_re_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
            mode_vals[mode_i].transferL2_re_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;

            D_i = B[(i*num_pars*num_modes) + 4*num_modes + mode_i];
            D_ip1 = B[((i+1)*num_pars*num_modes) + 4*num_modes + mode_i];
            y_i = mode_vals[mode_i].transferL2_im[i];
            y_ip1 = mode_vals[mode_i].transferL2_im[i+1];
            mode_vals[mode_i].transferL2_im_coeff_1[i] = D_i;
            mode_vals[mode_i].transferL2_im_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
            mode_vals[mode_i].transferL2_im_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;

            D_i = B[(i*num_pars*num_modes) + 5*num_modes + mode_i];
            D_ip1 = B[((i+1)*num_pars*num_modes) + 5*num_modes + mode_i];
            y_i = mode_vals[mode_i].transferL3_re[i];
            y_ip1 = mode_vals[mode_i].transferL3_re[i+1];
            mode_vals[mode_i].transferL3_re_coeff_1[i] = D_i;
            mode_vals[mode_i].transferL3_re_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
            mode_vals[mode_i].transferL3_re_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;

            D_i = B[(i*num_pars*num_modes) + 6*num_modes + mode_i];
            D_ip1 = B[((i+1)*num_pars*num_modes) + 6*num_modes + mode_i];
            y_i = mode_vals[mode_i].transferL3_im[i];
            y_ip1 = mode_vals[mode_i].transferL3_im[i+1];
            mode_vals[mode_i].transferL3_im_coeff_1[i] = D_i;
            mode_vals[mode_i].transferL3_im_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
            mode_vals[mode_i].transferL3_im_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;

            D_i = B[(i*num_pars*num_modes) + 7*num_modes + mode_i];
            D_ip1 = B[((i+1)*num_pars*num_modes) + 7*num_modes + mode_i];
            y_i = mode_vals[mode_i].time_freq_corr[i];
            y_ip1 = mode_vals[mode_i].time_freq_corr[i+1];
            mode_vals[mode_i].time_freq_coeff_1[i] = D_i;
            mode_vals[mode_i].time_freq_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
            mode_vals[mode_i].time_freq_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;

}

#ifdef __HIPCC__
CUDA_KERNEL
void set_spline_constants_response_wrap(ModeContainer *mode_vals, double *B, int f_length, int num_modes){
    double D_i, D_ip1, y_i, y_ip1;
    int num_pars = 8;
    for (int mode_i = blockIdx.y * blockDim.y + threadIdx.y;
         mode_i < num_modes;
         mode_i += blockDim.y * gridDim.y){

       for (int i = blockIdx.x * blockDim.x + threadIdx.x;
            i < f_length-1;
            i += blockDim.x * gridDim.x){

              set_spline_constants_response(mode_vals, B, f_length, num_modes, mode_i, i);

  }
}
}
#else
void cpu_set_spline_constants_response_wrap(ModeContainer *mode_vals, double *B, int f_length, int num_modes){
    double D_i, D_ip1, y_i, y_ip1;
    int num_pars = 8;
    for (int mode_i = 0;
         mode_i < num_modes;
         mode_i += 1){

       for (int i = 0;
            i < f_length-1;
            i += 1){

              set_spline_constants_response(mode_vals, B, f_length, num_modes, mode_i, i);

  }
}
}
#endif

/*
Find spline coefficients after matrix calculation on GPU for amp and phase
*/

CUDA_CALLABLE_MEMBER void set_spline_constants_wave(ModeContainer *mode_vals, double *B, int f_length, int num_modes, int mode_i, int i){

    double D_i, D_ip1, y_i, y_ip1;
    int num_pars = 2;

    D_i = B[i*num_modes + mode_i];
    D_ip1 = B[(i+1)*num_modes + mode_i];
    y_i = mode_vals[mode_i].amp[i];
    y_ip1 = mode_vals[mode_i].amp[i+1];
    mode_vals[mode_i].amp_coeff_1[i] = D_i;
    mode_vals[mode_i].amp_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
    mode_vals[mode_i].amp_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;

    D_i = B[(num_modes*f_length) + i*num_modes + mode_i];
    D_ip1 = B[(num_modes*f_length) + (i+1)*num_modes + mode_i];
    y_i = mode_vals[mode_i].phase[i];
    y_ip1 = mode_vals[mode_i].phase[i+1];
    mode_vals[mode_i].phase_coeff_1[i] = D_i;
    mode_vals[mode_i].phase_coeff_2[i] = 3.0 * (y_ip1 - y_i) - 2.0*D_i - D_ip1;
    mode_vals[mode_i].phase_coeff_3[i] = 2.0 * (y_i - y_ip1) + D_i + D_ip1;
}

#ifdef __HIPCC__
CUDA_KERNEL void set_spline_constants_wave_wrap(ModeContainer *mode_vals, double *B, int f_length, int num_modes){

    double D_i, D_ip1, y_i, y_ip1;
    int num_pars = 2;
     for (int mode_i = blockIdx.y * blockDim.y + threadIdx.y;
          mode_i < num_modes;
          mode_i += blockDim.y * gridDim.y){

        for (int i = blockIdx.x * blockDim.x + threadIdx.x;
             i < f_length-1;
             i += blockDim.x * gridDim.x){

               set_spline_constants_wave(mode_vals, B, f_length, num_modes, mode_i, i);
}
}
}
#else
void cpu_set_spline_constants_wave_wrap(ModeContainer *mode_vals, double *B, int f_length, int num_modes){

    double D_i, D_ip1, y_i, y_ip1;
    int num_pars = 2;
     for (int mode_i = 0;
          mode_i < num_modes;
          mode_i += 1){

        for (int i = 0;
             i < f_length-1;
             i += 1){

               set_spline_constants_wave(mode_vals, B, f_length, num_modes, mode_i, i);
}
}
}
#endif

/*
Interpolate amp, phase, and response transfer functions on GPU.
*/
CUDA_CALLABLE_MEMBER
void interpolate(agcmplx *channel1_out, agcmplx *channel2_out, agcmplx *channel3_out, ModeContainer* old_mode_vals,
    int num_modes, double d_log10f, double *old_freqs, int old_length, double *data_freqs, int data_length,
    double *channel1_ASDinv, double *channel2_ASDinv, double *channel3_ASDinv, int num_walkers,
    double f_min_limit, double f_max_limit, double t_break_start, double t_break_end, double t_obs_end, int walker_i, int i){
    //int mode_i = blockIdx.y;

    double f, x, x2, x3, coeff_0, coeff_1, coeff_2, coeff_3;
    double time_check, amp, phase, phaseRdelay;
    double transferL1_re, transferL1_im, transferL2_re, transferL2_im, transferL3_re, transferL3_im;
    agcmplx ampphasefactor;
    agcmplx I = agcmplx(0.0, 1.0);
    int old_ind_below;
    agcmplx trans_complex;

    //if (mode_i >= num_modes) return;

        channel1_out[walker_i*data_length + i] = agcmplx(0.0, 0.0);
        channel2_out[walker_i*data_length + i] = agcmplx(0.0, 0.0);
        channel3_out[walker_i*data_length + i] = agcmplx(0.0, 0.0);


    /*# if __CUDA_ARCH__>=200
    if (i == 200)
        printf("times: %e %e, %e, %e \n", t0, tRef, t_obs_start, t_break_start);

    #endif*/
    /*for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < data_length;
         i += blockDim.x * gridDim.x)
      {*/
      f = data_freqs[i];
      old_ind_below = floor((log10(f) - log10(old_freqs[walker_i*old_length + 0]))/d_log10f);

      if ((old_ind_below == old_length -1) || (f >= f_max_limit) || (f < f_min_limit) || (old_ind_below >= old_length)){
          return;
      }
      x = (f - old_freqs[walker_i*old_length + old_ind_below])/(old_freqs[walker_i*old_length + old_ind_below+1] - old_freqs[walker_i*old_length + old_ind_below]);
      x2 = x*x;
      x3 = x*x2;

    for (int mode_i=0; mode_i<num_modes; mode_i++){
            // interp time frequency to remove less than 0.0
            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].time_freq_corr[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].time_freq_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].time_freq_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].time_freq_coeff_3[old_ind_below];

            time_check = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);

            if (time_check < t_break_start) {
                continue;
            }

            if ((t_obs_end > 0.0) && (time_check >= t_break_end)){
                continue;
            }

            // interp amplitude
            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].amp[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].amp_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].amp_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].amp_coeff_3[old_ind_below];

            amp = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);

            if (amp < 1e-40){
                continue;
            }
            // interp phase
            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].phase[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].phase_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].phase_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].phase_coeff_3[old_ind_below];

            phase  = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);

            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].phaseRdelay[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].phaseRdelay_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].phaseRdelay_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].phaseRdelay_coeff_3[old_ind_below];

            phaseRdelay  = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);
            ampphasefactor = amp*gcmplx::exp(agcmplx(0.0, phase + phaseRdelay));

            // X or A
            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].transferL1_re[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].transferL1_re_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].transferL1_re_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].transferL1_re_coeff_3[old_ind_below];

            transferL1_re  = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);

            /*# if __CUDA_ARCH__>=200
            if (i == 15000)
                printf("times: %e, %d, %d, %d, %d, %e, %e, %e, %e, %e, %e\n", f, mode_i, walker_i, old_ind_below, old_length, time_check, t_break_start, t0, tRef, amp, transferL1_re);

            #endif //*/

            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].transferL1_im[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].transferL1_im_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].transferL1_im_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].transferL1_im_coeff_3[old_ind_below];

            transferL1_im  = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);

            trans_complex = agcmplx(transferL1_re, transferL1_im)* ampphasefactor * channel1_ASDinv[i]; //TODO may be faster to load as complex number with 0.0 for imaginary part

            channel1_out[walker_i*data_length + i] += trans_complex;
            // Y or E
            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].transferL2_re[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].transferL2_re_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].transferL2_re_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].transferL2_re_coeff_3[old_ind_below];

            transferL2_re  = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);

            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].transferL2_im[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].transferL2_im_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].transferL2_im_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].transferL2_im_coeff_3[old_ind_below];

            transferL2_im  = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);

            trans_complex = agcmplx(transferL2_re, transferL2_im)* ampphasefactor * channel2_ASDinv[i]; //TODO may be faster to load as complex number with 0.0 for imaginary part

            channel2_out[walker_i*data_length + i] += trans_complex;

            // Z or T
            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].transferL3_re[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].transferL3_re_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].transferL3_re_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].transferL3_re_coeff_3[old_ind_below];

            transferL3_re  = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);

            coeff_0 = old_mode_vals[walker_i*num_modes + mode_i].transferL3_im[old_ind_below];
            coeff_1 = old_mode_vals[walker_i*num_modes + mode_i].transferL3_im_coeff_1[old_ind_below];
            coeff_2 = old_mode_vals[walker_i*num_modes + mode_i].transferL3_im_coeff_2[old_ind_below];
            coeff_3 = old_mode_vals[walker_i*num_modes + mode_i].transferL3_im_coeff_3[old_ind_below];

            transferL3_im  = coeff_0 + (coeff_1*x) + (coeff_2*x2) + (coeff_3*x3);

            trans_complex = agcmplx(transferL3_re, transferL3_im)* ampphasefactor * channel3_ASDinv[i]; //TODO may be faster to load as complex number with 0.0 for imaginary part

            channel3_out[walker_i*data_length + i] += trans_complex;

          }
}

#ifdef __HIPCC__
CUDA_KERNEL
void interpolate_wrap(agcmplx *channel1_out, agcmplx *channel2_out, agcmplx *channel3_out, ModeContainer* old_mode_vals,
    int num_modes, double d_log10f, double *old_freqs, int old_length, double *data_freqs, int data_length, double* t0_arr, double* tRef_arr, double *channel1_ASDinv,
    double *channel2_ASDinv, double *channel3_ASDinv, double t_obs_start, double t_obs_end, int num_walkers){
    //int mode_i = blockIdx.y;

    double f_min_limit, f_max_limit, t0, tRef, t_break_start, t_break_end;

    for (int walker_i = blockIdx.z * blockDim.z + threadIdx.z;
         walker_i < num_walkers;
         walker_i += blockDim.z * gridDim.z){

     f_min_limit = old_freqs[walker_i*old_length];
     f_max_limit = old_freqs[walker_i*old_length + old_length-1];
     t0 = t0_arr[walker_i];
     tRef = tRef_arr[walker_i];
     t_break_start = t0*YRSID_SI + tRef - t_obs_start*YRSID_SI; // t0 and t_obs_start in years. tRef in seconds.
     t_break_end = t0*YRSID_SI + tRef - t_obs_end*YRSID_SI;

    for (int i = blockIdx.x * blockDim.x + threadIdx.x;
         i < data_length;
         i += blockDim.x * gridDim.x){

            interpolate(channel1_out, channel2_out, channel3_out, old_mode_vals, num_modes, d_log10f, old_freqs, old_length,
                        data_freqs, data_length, channel1_ASDinv, channel2_ASDinv, channel3_ASDinv, num_walkers,
                        f_min_limit, f_max_limit, t_break_start, t_break_end, t_obs_end, walker_i, i);

}
}
}

#else
void cpu_interpolate_wrap(agcmplx *channel1_out, agcmplx *channel2_out, agcmplx *channel3_out, ModeContainer* old_mode_vals,
    int num_modes, double d_log10f, double *old_freqs, int old_length, double *data_freqs, int data_length, double* t0_arr, double* tRef_arr, double *channel1_ASDinv,
    double *channel2_ASDinv, double *channel3_ASDinv, double t_obs_start, double t_obs_end, int num_walkers){
    //int mode_i = blockIdx.y;

    double f_min_limit, f_max_limit, t0, tRef, t_break_start, t_break_end;

    for (int walker_i = 0;
         walker_i < num_walkers;
         walker_i += 1){

     f_min_limit = old_freqs[walker_i*old_length];
     f_max_limit = old_freqs[walker_i*old_length + old_length-1];
     t0 = t0_arr[walker_i];
     tRef = tRef_arr[walker_i];
     t_break_start = t0*YRSID_SI + tRef - t_obs_start*YRSID_SI; // t0 and t_obs_start in years. tRef in seconds.
     t_break_end = t0*YRSID_SI + tRef - t_obs_end*YRSID_SI;

    for (int i = 0;
         i < data_length;
         i += 1){

            interpolate(channel1_out, channel2_out, channel3_out, old_mode_vals, num_modes, d_log10f, old_freqs, old_length,
                        data_freqs, data_length, channel1_ASDinv, channel2_ASDinv, channel3_ASDinv, num_walkers,
                        f_min_limit, f_max_limit, t_break_start, t_break_end, t_obs_end, walker_i, i);

}
}
}
#endif

/*
Interpolation class initializer
*/

Interpolate::Interpolate(){
    int pass = 0;
}

/*
allocate arrays for interpolation
*/

void Interpolate::alloc_arrays(int m, int n, double *d_B){
    w = new double[m];
    a = new double[m];
    b = new double[m];
    c = new double[m];
    x = new double[m*n];

    a[0] = 0.0;
    b[0] = 2.0;
    c[0] = 1.0;

    a[m-1] = 1.0;
    b[m-1] = 2.0;
    c[m-1] = 0.0;

    for (int i = 1;
         i < m-1;
         i += 1){
     a[i] = 1.0;
     b[i] = 4.0;
     c[i] = 1.0;
 }

 for (int i=1; i<m; i++){
     w[i] = a[i]/b[i-1];
     b[i] = b[i] - w[i]*c[i-1];
 }

    #ifdef __HIPCC__
    gpuErrchk_here(hipMalloc(&d_b, m*sizeof(double)));
    gpuErrchk_here(hipMemcpy(d_b, b, m*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk_here(hipMalloc(&d_c, m*sizeof(double)));
    gpuErrchk_here(hipMemcpy(d_c, c, m*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk_here(hipMalloc(&d_w, m*sizeof(double)));
    gpuErrchk_here(hipMemcpy(d_w, w, m*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk_here(hipMalloc(&d_x, m*n*sizeof(double)));
    #endif
}


CUDA_CALLABLE_MEMBER
void fit_constants_serial(int m, int n, double *w, double *b, double *c, double *d_in, double *x_in, int j){

        # pragma unroll
        for (int i=2; i<m; i++){
            //printf("%d\n", i);
            d_in[i*n + j] = d_in[i*n + j] - w[i]*d_in[(i-1)*n + j];
            //printf("in1: %d, %lf, %lf\n", i, w[i], d_in[i*n + j]);
        }

        x_in[(m-1)*n + j] = d_in[(m-1)*n + j]/b[m-1];
        d_in[(m-1)*n + j] = x_in[(m-1)*n + j];
        # pragma unroll
        for (int i=(m-2); i>=0; i--){
            x_in[i*n + j] = (d_in[i*n + j] - c[i]*x_in[(i+1)*n + j])/b[i];
            d_in[i*n + j] = x_in[i*n + j];
            //printf("in2: %d, %lf, %lf\n", i, d_in[i], x_in[i]);
        }
}

#ifdef __HIPCC__
CUDA_KERNEL
void fit_constants_serial_wrap(int m, int n, double *w, double *b, double *c, double *d_in, double *x_in){

    //double *x, *d;
    for (int j = blockIdx.x * blockDim.x + threadIdx.x;
         j < n;
         j += blockDim.x * gridDim.x){

           fit_constants_serial(m, n, w, b, c, d_in, x_in, j);
    }
}

#else
void cpu_fit_constants_serial_wrap(int m, int n, double *w, double *b, double *c, double *d_in, double *x_in){

    //double *x, *d;
    for (int j = 0;
         j < n;
         j += 1){
           fit_constants_serial(m, n, w, b, c, d_in, x_in, j);
    }
}
#endif



/*
solve matrix solution for tridiagonal matrix for cublic spline.
*/
void Interpolate::prep(double *B, int m_, int n_, int to_gpu_){
    m = m_;
    n = n_;
    to_gpu = to_gpu_;

    #ifdef __HIPCC__
    int NUM_THREADS = 256;
    int num_blocks = std::ceil((n + NUM_THREADS -1)/NUM_THREADS);
    fit_constants_serial_wrap<<<num_blocks, NUM_THREADS>>>(m, n, d_w, d_b, d_c, B, d_x);
    hipDeviceSynchronize();
    gpuErrchk_here(hipGetLastError());
    #else
    cpu_fit_constants_serial_wrap(m, n, w, b, c, B, x);
    #endif
}


/*
Deallocate
*/
Interpolate::~Interpolate(){
  delete[] w;
  delete[] a;
  delete[] b;
  delete[] c;
  delete[] x;

  #ifdef __HIPCC__
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_w);
    hipFree(d_x);

  #endif
}
