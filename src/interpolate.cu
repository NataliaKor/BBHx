#include "hip/hip_runtime.h"
#include "manager.hh"
#include "stdio.h"
#include <assert.h>
#include <hipsparse.h>

#define ERR_NE(X,Y) do { if ((X) != (Y)) { \
                             fprintf(stderr,"Error in %s at %s:%d\n",__func__,__FILE__,__LINE__); \
                             exit(-1);}} while(0)

#define CUDA_CALL(X) ERR_NE((X),hipSuccess)
#define CUSPARSE_CALL(X) ERR_NE((X),HIPSPARSE_STATUS_SUCCESS)
using namespace std;

Interpolate::Interpolate(){
    int pass = 0;
}
void Interpolate::prep(double *B, int m_, int n_, int to_gpu_){
    m = m_;
    n = n_;
    to_gpu = to_gpu_;

    dl = new double[m];
    d = new double[m];
    du = new double[m];

    dl[0] = 0.0;
    d[0] = 2.0;
    du[0] = 1.0;
    d[m-1] = 2.0;
    du[m-1] = 0.0;
    dl[m-1] = 1.0;
    int i;
    for (i=1; i<m-1; i++){
        dl[i] = 1.0;
        du[i] = 1.0;
        d[i] = 4.0;
    }
    if (to_gpu == 1){
        err = hipMalloc(&d_dl, m*sizeof(double));
        assert(err == 0);
        err = hipMalloc(&d_d, m*sizeof(double));
        assert(err == 0);
        err = hipMalloc(&d_du, m*sizeof(double));
        assert(err == 0);
        err = hipMemcpy(d_dl, dl, m*sizeof(double), hipMemcpyHostToDevice);
        assert(err == 0);
        err = hipMemcpy(d_d, d, m*sizeof(double), hipMemcpyHostToDevice);
        assert(err == 0);
        err = hipMemcpy(d_du, du, m*sizeof(double), hipMemcpyHostToDevice);
        assert(err == 0);
    }

    if (to_gpu == 1){
        Interpolate::gpu_fit_constants(B);
    }
    else Interpolate::fit_constants(B);
    //dx_old = x_old[1] - x_old[0];
}

__host__ void Interpolate::gpu_fit_constants(double *B){
    /*double *h_B;
    int f_length= 20;
    int num_modes = 6;
    h_B = new double[2*f_length*num_modes];
    hipMemcpy(h_B, B, 2*f_length*num_modes*sizeof(double), hipMemcpyDeviceToHost);
    for (int i=0; i<2*f_length*num_modes; i++) printf("%e\n", h_B[i]);
    h_B = new double[2*f_length*num_modes];*/
    CUSPARSE_CALL( hipsparseCreate(&handle) );
    hipsparseStatus_t status = cusparseDgtsv(handle, m, n, d_dl, d_d, d_du, B, m);
    if (status !=  HIPSPARSE_STATUS_SUCCESS) assert(0);
}

__host__ void Interpolate::fit_constants(double *B){
    int i;
    double *w = new double[m];
    double *D = new double[m];
    for (i=2; i<m; i++){
        //printf("%d\n", i);
        w[i] = dl[i]/d[i-1];
        d[i] = d[i] - w[i]*du[i-1];
        B[i] = B[i] - w[i]*B[i-1];
        //printf("%lf, %lf, %lf\n", w[i], d[i], b[i]);
    }

    D[m-1] = B[m-1]/d[m-1];
    for (i=(m-2); i>=0; i--){
        D[i] = (B[i] - du[i]*D[i+1])/d[i];
    }

    for (int i=0; i<m; i++) B[i] = D[i];
    delete D;
    delete w;
    /*for (i=0;i<N-1; i++){
        coeff_1[i] = D[i];
        coeff_2[i] = 3.0*(y_old[i+1] - y_old[i]) - 2.0*D[i] - D[i+1];
        coeff_3[i] = 2.0*(y_old[i] - y_old[i+1]) + D[i] + D[i+1];
    }*/
}

__host__ void Interpolate::transferToDevice(){
    hipMemcpy(dev_coeff_1, coeff_1, (N-1)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_coeff_2, coeff_2, (N-1)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_coeff_3, coeff_3, (N-1)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_x_old, x_old, (N)*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(dev_y_old, y_old, (N)*sizeof(double), hipMemcpyHostToDevice);
}

__device__ double Interpolate::call(double x_new){
    int index;
    if ((x_new > dev_x_old[0]) && (x_new < dev_x_old[N-1])){
        index = (int)floor(x_new/dx_old);
    } else if (x_new <= dev_x_old[0]){
        index = 0;
    } else {
        index = N-1;
    }
    double x = x_new - dev_x_old[index];
    double x2 = x*x;
    double x3 = x2*x;
    double y_new = dev_y_old[index] + dev_coeff_1[index]*x + dev_coeff_2[index]*x2 + dev_coeff_3[index]*x3;
    return y_new;
}

__host__ double Interpolate::cpu_call(double x_new){
    int index;
    if ((x_new > x_old[0]) && (x_new < x_old[N-1])){
        index = (int)floor(x_new/dx_old);
    } else if (x_new <= x_old[0]){
        index = 0;
    } else {
        index = N-1;
    }

    double x = x_new - x_old[index];
    double x2 = x*x;
    double x3 = x2*x;
    double y_new = y_old[index] + coeff_1[index]*x + coeff_2[index]*x2 + coeff_3[index]*x3;
    return y_new;
}

__host__ Interpolate::~Interpolate(){
    delete dl;
    delete d;
    delete du;

    if (to_gpu == 1){
        hipError_t err;
        err = hipFree(d_dl);
        assert(err == 0);
        err = hipFree(d_d);
        assert(err == 0);
        err = hipFree(d_du);
        assert(err == 0);
        hipsparseDestroy(handle);
    }

}

__global__ void run_interp(double *x_new, double *y_new, int num, Interpolate interp){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= num) return;

    y_new[i] = interp.call(x_new[i]);
}

__global__ void wave_interpolate(double *f_new, double *amp_new, double *phase_new, int num_modes, int length, Interpolate *interp_all){
    int i = blockIdx.y * blockDim.x + threadIdx.x;
    if (i >= length) return;
    int mode_i = blockIdx.x;
    if (mode_i >= num_modes) return;
    //Interpolate amp_interp = interp_all[mode_i];
    //Interpolate phase_interp = interp_all[num_modes + mode_i];
    amp_new[num_modes*length + i] = interp_all[mode_i].call(f_new[i]);
    phase_new[num_modes*length + i] = interp_all[num_modes + mode_i].call(f_new[i]);
}

/*
int main(){
    hipError_t err;
    int num = 100;
    double *x = new double[num];
    double *y = new double[num];

    for (int i=0; i<num; i++){
        x[i] = (double) i+1.0;
        y[i] = x[i] * x[i] * x[i];
    }

    Interpolate interp;
    interp.prep(x, y, num);
    int new_num = 200;
    double *x_new = new double[new_num];
    double *y_new = new double[new_num];

    double dx = (x[num-1] - x[0])/(new_num + 1);
    for (int i=0; i<new_num; i++){
        x_new[i] = (i+1)*dx;
        y_new[i] = interp.cpu_call(x_new[i]);
        //y_new[i] = 0.0;
        printf("%lf, %lf\n", x_new[i], y_new[i]);
    }

    double *d_x_new, *d_y_new, *y_check;
    err = hipMalloc(&d_x_new, new_num*sizeof(double));
    assert(err == 0);
    err = hipMalloc(&d_y_new, new_num*sizeof(double));
    assert(err == 0);
    y_check = new double[new_num];

    err = hipMemcpy(d_x_new, x_new, new_num*sizeof(double), hipMemcpyHostToDevice);
    assert(err == 0);
    interp.transferToDevice();
    int NUM_THREADS = 256;
    int num_blocks = (new_num + NUM_THREADS -1)/NUM_THREADS;
    run_interp<<<num_blocks, NUM_THREADS>>>(d_x_new, d_y_new, new_num, interp);
    hipDeviceSynchronize();

    err = hipMemcpy(y_check, d_y_new, new_num*sizeof(double), hipMemcpyDeviceToHost);
    assert(err == 0);
    for (int i=0; i<new_num; i++){
        //y_new[i] = 0.0;
        if (y_check[i] != y_new[i]) printf("%lf, %lf\n", y_new[i], y_check[i]);
    }

    //interp.Interpolate;
    err = hipFree(d_x_new);
    assert(err == 0);
    err = hipFree(d_y_new);
    assert(err == 0);
    delete x;
    delete y;
    delete x_new;
    delete y_new;
    delete y_check;

    printf("check\n");
    return(0);
}*/
