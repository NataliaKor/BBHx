#include "hip/hip_runtime.h"
/*
This is the central piece of code. This file implements a class
(interface in gpuadder.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
//#include <reduction.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include "globalPhenomHM.h"
#include <complex>
#include "hip/hip_complex.h"
#include "hipblas.h"


using namespace std;

GPUPhenomHM::GPUPhenomHM (double *freqs_,
    int f_length_,
    unsigned int *l_vals_,
    unsigned int *m_vals_,
    int num_modes_,
    int to_gpu_){

    freqs = freqs_;
    f_length = f_length_;
    l_vals = l_vals_;
    m_vals = m_vals_;
    num_modes = num_modes_;
    to_gpu = to_gpu_;

    f_length = f_length_;

    hipError_t err;

    // DECLARE ALL THE  NECESSARY STRUCTS

    freqs_geom_trans = new double[f_length];

    pHM_trans = new PhenomHMStorage;

    pAmp_trans = new IMRPhenomDAmplitudeCoefficients;

    amp_prefactors_trans = new AmpInsPrefactors;

    pDPreComp_all_trans = new PhenDAmpAndPhasePreComp[num_modes];

    q_all_trans = new HMPhasePreComp[num_modes];

    factorp_trans = new std::complex<double>[num_modes];
    factorc_trans = new std::complex<double>[num_modes];


  if ((to_gpu == 0) || (to_gpu == 2)){
      printf("cpu\n");
      hptilde = new std::complex<double>[num_modes*f_length];
      hctilde = new std::complex<double>[num_modes*f_length];
  }
  if ((to_gpu == 1) || (to_gpu == 2)){

      printf("was here\n");

      size_t freqs_size = f_length*sizeof(double);
      hipMalloc(&d_freqs_geom, freqs_size);

      size_t mode_array_size = num_modes*sizeof(unsigned int);
      hipMalloc(&d_l_vals, mode_array_size);
      hipMalloc(&d_m_vals, mode_array_size);
      hipMemcpy(d_l_vals, l_vals, mode_array_size, hipMemcpyHostToDevice);
      hipMemcpy(d_m_vals, m_vals, mode_array_size, hipMemcpyHostToDevice);

      size_t h_size = num_modes*f_length*sizeof(hipDoubleComplex);
      hipMalloc(&d_hptilde, h_size);
      hipMalloc(&d_hctilde, h_size);


      // DECLARE ALL THE  NECESSARY STRUCTS
      hipMalloc(&d_pHM_trans, sizeof(PhenomHMStorage));

      hipMalloc(&d_pAmp_trans, sizeof(IMRPhenomDAmplitudeCoefficients));

      hipMalloc(&d_amp_prefactors_trans, sizeof(AmpInsPrefactors));

      hipMalloc(&d_pDPreComp_all_trans, num_modes*sizeof(PhenDAmpAndPhasePreComp));

      err = hipMalloc((void**) &d_q_all_trans, num_modes*sizeof(HMPhasePreComp));
      assert(err == 0);

      size_t complex_factor_size = num_modes*sizeof(hipDoubleComplex);
      err = hipMalloc(&d_factorp_trans, complex_factor_size);
      assert(err == 0);
      err = hipMalloc(&d_factorc_trans, complex_factor_size);
      assert(err == 0);

      double cShift[7] = {0.0,
                           PI_2 /* i shift */,
                           0.0,
                           -PI_2 /* -i shift */,
                           PI /* 1 shift */,
                           PI_2 /* -1 shift */,
                           0.0};

      err = hipMalloc(&d_cShift, 7*sizeof(double));
      assert(err == 0);
      err = hipMemcpy(d_cShift, &cShift, 7*sizeof(double), hipMemcpyHostToDevice);
      assert(err == 0);

      // for likelihood
      // --------------
      hipHostMalloc((hipDoubleComplex**) &result, sizeof(hipDoubleComplex));

      stat = hipblasCreate(&handle);
      if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("CUBLAS initialization failed\n");
              exit(0);
          }
      // ----------------

      NUM_THREADS = 256;
      num_blocks = std::ceil((f_length + NUM_THREADS -1)/NUM_THREADS);
      dim3 gridDim(num_modes, num_blocks);
      printf("blocks %d\n", num_blocks);
      this->gridDim = gridDim;
  }




  //double t0_;
  t0 = 0.0;

  //double phi0_;
  phi0 = 0.0;

  //double amp0_;
  amp0 = 0.0;
}


void GPUPhenomHM::gpu_gen_PhenomHM(
    double m1_, //solar masses
    double m2_, //solar masses
    double chi1z_,
    double chi2z_,
    double distance_,
    double inclination_,
    double phiRef_,
    double deltaF_,
    double f_ref_){

    assert((to_gpu == 1) || (to_gpu == 2));

    GPUPhenomHM::cpu_gen_PhenomHM(
        m1_, //solar masses
        m2_, //solar masses
        chi1z_,
        chi2z_,
        distance_,
        inclination_,
        phiRef_,
        deltaF_,
        f_ref_);


    // Initialize inputs

    hipError_t err;

    err = hipMemcpy(d_freqs_geom, freqs_geom_trans, f_length*sizeof(double), hipMemcpyHostToDevice);
    assert(err == 0);

    err = hipMemcpy(d_pHM_trans, pHM_trans, sizeof(PhenomHMStorage), hipMemcpyHostToDevice);
    assert(err == 0);

    err = hipMemcpy(d_pAmp_trans, pAmp_trans, sizeof(IMRPhenomDAmplitudeCoefficients), hipMemcpyHostToDevice);
    assert(err == 0);

    err = hipMemcpy(d_amp_prefactors_trans, amp_prefactors_trans, sizeof(AmpInsPrefactors), hipMemcpyHostToDevice);
    assert(err == 0);

    err = hipMemcpy(d_pDPreComp_all_trans, pDPreComp_all_trans, num_modes*sizeof(PhenDAmpAndPhasePreComp), hipMemcpyHostToDevice);
    assert(err == 0);

    err = hipMemcpy(d_q_all_trans, q_all_trans, num_modes*sizeof(HMPhasePreComp), hipMemcpyHostToDevice);
    assert(err == 0);

    err = hipMemcpy(d_factorp_trans, factorp_trans, num_modes*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    assert(err == 0);
    err = hipMemcpy(d_factorc_trans, factorc_trans, num_modes*sizeof(hipDoubleComplex), hipMemcpyHostToDevice);
    assert(err == 0);


    /* main: evaluate model at given frequencies */

    kernel_calculate_all_modes<<<gridDim, NUM_THREADS>>>(d_hptilde,
          d_hctilde,
          d_l_vals,
          d_m_vals,
          d_pHM_trans,
          d_freqs_geom,
          d_pAmp_trans,
          d_amp_prefactors_trans,
          d_pDPreComp_all_trans,
          d_q_all_trans,
          amp0,
          d_factorp_trans,
          d_factorc_trans,
          num_modes,
          f_length,
          t0,
          phi0,
          d_cShift
      );
     hipDeviceSynchronize();
     err = hipGetLastError();
     assert(err == 0);
}


void GPUPhenomHM::cpu_gen_PhenomHM(
    double m1_, //solar masses
    double m2_, //solar masses
    double chi1z_,
    double chi2z_,
    double distance_,
    double inclination_,
    double phiRef_,
    double deltaF_,
    double f_ref_){

    m1 = m1_; //solar masses
    m2 = m2_; //solar masses
    chi1z = chi1z_;
    chi2z = chi2z_;
    distance = distance_;
    inclination = inclination_;
    phiRef = phiRef_;
    deltaF = deltaF_;
    f_ref = f_ref_;

    m1_SI = m1*MSUN_SI;
    m2_SI = m2*MSUN_SI;

    /* main: evaluate model at given frequencies */
    retcode = 0;
    retcode = IMRPhenomHMCore(
        hptilde,
        hctilde,
        freqs,
        freqs_geom_trans,
        f_length,
        m1_SI,
        m2_SI,
        chi1z,
        chi2z,
        distance,
        inclination,
        phiRef,
        deltaF,
        f_ref,
        l_vals,
        m_vals,
        num_modes,
        to_gpu,
        pHM_trans,
        pAmp_trans,
        amp_prefactors_trans,
        pDPreComp_all_trans,
        q_all_trans,
        factorp_trans,
        factorc_trans,
        &t0,
        &phi0,
        &amp0);
    assert (retcode == 1); //,PD_EFUNC, "IMRPhenomHMCore failed in
}

double GPUPhenomHM::Likelihood (){

    stat = hipblasZdotc(handle, f_length*num_modes,
            d_hptilde, 1,
            d_hptilde, 1,
            result);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
            printf ("CUBLAS initialization failed\n");
            return EXIT_FAILURE;
        }
    return hipCreal(result[0]);
}

void GPUPhenomHM::Get_Waveform (std::complex<double>* hptilde_, std::complex<double>* hctilde_) {

assert ((to_gpu == 0) || (to_gpu == 2));
 memcpy(hptilde_, hptilde, num_modes*f_length*sizeof(std::complex<double>));
 memcpy(hctilde_, hctilde, num_modes*f_length*sizeof(std::complex<double>));

}

void GPUPhenomHM::gpu_Get_Waveform (std::complex<double>* hptilde_, std::complex<double>* hctilde_) {
  assert((to_gpu == 1) || (to_gpu == 2));
    hipError_t err;
     err = hipMemcpy(hptilde_, d_hptilde, num_modes*f_length*sizeof(std::complex<double>), hipMemcpyDeviceToHost);
     assert(err == 0);
     err = hipMemcpy(hctilde_, d_hctilde, num_modes*f_length*sizeof(std::complex<double>), hipMemcpyDeviceToHost);
     assert(err == 0);
}

GPUPhenomHM::~GPUPhenomHM() {
  delete freqs_geom_trans;
  delete pHM_trans;
  delete pAmp_trans;
  delete amp_prefactors_trans;
  delete pDPreComp_all_trans;
  delete q_all_trans;
  delete factorp_trans;
  delete factorc_trans;

  if ((to_gpu ==0) || (to_gpu == 2)){
      delete hptilde;
      delete hctilde;
  }
  if ((to_gpu == 1) || (to_gpu == 2)){
      hipFree(d_freqs_geom);
      hipFree(d_l_vals);
      hipFree(d_m_vals);
      hipFree(d_pHM_trans);
      hipFree(d_pAmp_trans);
      hipFree(d_amp_prefactors_trans);
      hipFree(d_pDPreComp_all_trans);
      hipFree(d_q_all_trans);
      hipFree(d_factorp_trans);
      hipFree(d_factorc_trans);
      hipFree(d_hptilde);
      hipFree(d_hctilde);
      hipFree(d_cShift);
      hipFree(result);
      hipblasDestroy(handle);
  }
}
