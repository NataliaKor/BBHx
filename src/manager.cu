/*
This is the central piece of code. This file implements a class
(interface in gpuadder.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include "globalPhenomHM.h"
#include "tester.hh"
#include "complex.h"


using namespace std;

GPUPhenomHM::GPUPhenomHM (int* array_host_, int length_,
    double *freqs_,
    int f_length_,
    unsigned int *l_vals_,
    unsigned int *m_vals_,
    int num_modes_,
    int to_gpu_){

    freqs = freqs_;
    f_length = f_length_;
    l_vals = l_vals_;
    m_vals = m_vals_;
    num_modes = num_modes_;
    to_gpu = to_gpu_;

    f_length = f_length_;

  if (!to_gpu){
      std::complex<double> *hptilde = new std::complex<double>[num_modes*f_length];
      std::complex<double> *hctilde = new std::complex<double>[num_modes*f_length];
      this->hptilde = hptilde;
      this->hctilde = hctilde;
  }

      // DECLARE ALL THE  NECESSARY STRUCTS

  PhenomHMStorage *pHM_trans = new PhenomHMStorage;
  this->pHM_trans = pHM_trans;

  IMRPhenomDAmplitudeCoefficients *pAmp_trans = new IMRPhenomDAmplitudeCoefficients;
  this->pAmp_trans = pAmp_trans;

  AmpInsPrefactors *amp_prefactors_trans = new AmpInsPrefactors;
  this->amp_prefactors_trans = amp_prefactors_trans;

  PhenDAmpAndPhasePreComp *pDPreComp_all_trans = new PhenDAmpAndPhasePreComp[num_modes];
  this->pDPreComp_all_trans = pDPreComp_all_trans;

  HMPhasePreComp *q_all_trans = new HMPhasePreComp[num_modes];
  this->q_all_trans = q_all_trans;

  std::complex<double> *factorp_trans = new std::complex<double>[num_modes];
  this->factorp_trans = factorp_trans;

  std::complex<double> *factorc_trans = new std::complex<double>[num_modes];
  this->factorc_trans = factorc_trans;

  //double t0_;
  this->t0 = 0.0;

  //double phi0_;
  this->phi0 = 0.0;

  //double amp0_;
  this->amp0 = 0.0;

  int retcode;

  array_host = array_host_;
  length = length_;
  int size = length * sizeof(int);
  hipError_t err = hipMalloc((void**) &array_device, size);
  assert(err == 0);
  err = hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);
  assert(err == 0);

  int sizex = sizeof(StructTest);
  x = (StructTest*) malloc(sizex);
  x->a = 10;

  err = hipMalloc((void**) &d_x, sizex);
  assert(err == 0);
  err = hipMemcpy(d_x, x, sizex, hipMemcpyHostToDevice);
  assert(err == 0);

}

void GPUPhenomHM::increment() {
  kernel_add_one<<<64, 64>>>(array_device, length, d_x);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

void GPUPhenomHM::cpu_gen_PhenomHM(
    double m1_, //solar masses
    double m2_, //solar masses
    double chi1z_,
    double chi2z_,
    double distance_,
    double inclination_,
    double phiRef_,
    double deltaF_,
    double f_ref_){

    m1 = m1_; //solar masses
    m2 = m2_; //solar masses
    chi1z = chi1z_;
    chi2z = chi2z_;
    distance = distance_;
    inclination = inclination_;
    phiRef = phiRef_;
    deltaF = deltaF_;
    f_ref = f_ref_;

    m1_SI = m1*MSUN_SI;
    m2_SI = m2*MSUN_SI;

    /* main: evaluate model at given frequencies */
    retcode = 0;
    retcode = IMRPhenomHMCore(
        hptilde,
        hctilde,
        freqs,
        f_length,
        m1_SI,
        m2_SI,
        chi1z,
        chi2z,
        distance,
        inclination,
        phiRef,
        deltaF,
        f_ref,
        l_vals,
        m_vals,
        num_modes,
        to_gpu,
        pHM_trans,
        pAmp_trans,
        amp_prefactors_trans,
        pDPreComp_all_trans,
        q_all_trans,
        factorp_trans,
        factorc_trans,
        &t0,
        &phi0,
        &amp0);
    assert (retcode == 1); //,PD_EFUNC, "IMRPhenomHMCore failed in IMRPhenomHM.");
    /*int i, j;
    printf("f_length %d\n\n", f_length);
    double check;
    for (i=0; i<num_modes; i++){
        for (j=0; j<f_length; j++){
            check = std::real(hptilde[i*f_length + j]);
            if (j % 100 == 0) printf("%e, %e, %e, %e, %e\n", freqs[j], std::real(hptilde[i*f_length + j]), std::imag(hptilde[i*f_length + j]), std::real(hctilde[i*f_length + j]), std::imag(hctilde[i*f_length + j]));
        }
    }
    //this->hptilde = hptilde;
    printf("\n\n\n\n\n\n\n");
     printf("\nhptilde %e\n\n", hptilde[0].real());*/

}

void GPUPhenomHM::retreive() {
  int size = length * sizeof(int);
  int sizex = sizeof(StructTest);
  hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);
  hipMemcpy(x, d_x, sizex, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  if(err != 0) { cout << err << endl; assert(0); }
  cout << x->a;
}


void GPUPhenomHM::retreive_to (int* array_host_, int length_) {
  assert(length == length_);
  int size = length * sizeof(int);
  hipMemcpy(array_host_, array_device, size, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

void GPUPhenomHM::Get_Waveform (std::complex<double>* hptilde_, std::complex<double>* hctilde_) {
  //hptilde[10] = std::complex<double>(10.0, 9.0);
  //printf("%e\n", hptilde[0].real());
  //printf("%d %d\n", length_, f_length);
if (this->to_gpu == 0){
     memcpy(hptilde_, hptilde, num_modes*f_length*sizeof(std::complex<double>));
     memcpy(hctilde_, hctilde, num_modes*f_length*sizeof(std::complex<double>));
}
  //array_host_[0] = this->hptilde[0];
  //printf("%e\n", array_host_[0].real());
}

GPUPhenomHM::~GPUPhenomHM() {
  hipFree(array_device);
  hipFree(d_x);
  delete pHM_trans;
  delete pAmp_trans;
  delete amp_prefactors_trans;
  delete pDPreComp_all_trans;
  delete q_all_trans;
  delete factorp_trans;
  delete factorc_trans;
  free(x);
  //free(freqs);
  delete hptilde;
  delete hctilde;
}
