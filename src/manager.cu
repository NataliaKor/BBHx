/*
This is the central piece of code. This file implements a class
(interface in gpuadder.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
//#include "tester.hh"
using namespace std;

GPUAdder::GPUPhenomHM (int* array_host_, int length_) {
  array_host = array_host_;
  length = length_;
  double_errthing(array_host, length);
  int size = length * sizeof(int);
  hipError_t err = hipMalloc((void**) &array_device, size);
  assert(err == 0);
  err = hipMemcpy(array_device, array_host, size, hipMemcpyHostToDevice);
  assert(err == 0);

  int sizex = sizeof(StructTest);
  x = (StructTest*) malloc(sizex);
  x->a = 10;

  err = hipMalloc((void**) &d_x, sizex);
  assert(err == 0);
  err = hipMemcpy(d_x, x, sizex, hipMemcpyHostToDevice);
  assert(err == 0);


}

void GPUAdder::increment() {
  kernel_add_one<<<64, 64>>>(array_device, length, d_x);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

void GPUAdder::retreive() {
  int size = length * sizeof(int);
  int sizex = sizeof(StructTest);
  hipMemcpy(array_host, array_device, size, hipMemcpyDeviceToHost);
  hipMemcpy(x, d_x, sizex, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  if(err != 0) { cout << err << endl; assert(0); }
  cout << x->a;
}


void GPUAdder::retreive_to (int* array_host_, int length_) {
  assert(length == length_);
  int size = length * sizeof(int);
  hipMemcpy(array_host_, array_device, size, hipMemcpyDeviceToHost);
  hipError_t err = hipGetLastError();
  assert(err == 0);
}

GPUAdder::~GPUAdder() {
  hipFree(array_device);
  hipFree(d_x);
  free(x);
}
