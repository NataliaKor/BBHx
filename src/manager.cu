#include "hip/hip_runtime.h"
/*
This is the central piece of code. This file implements a class
that takes data in on the cpu side, copies
it to the gpu, and exposes functions that let
you perform actions with the GPU

This class will get translated into python via cython
*/

#include <kernel.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include "globalPhenomHM.h"
#include <complex>
#include "hip/hip_complex.h"
#include "hipblas.h"
#include "interpolate.cu"
#include "fdresponse.h"
#include "createGPUHolders.cu"
#include "kernel_response.cu"


using namespace std;

PhenomHM::PhenomHM (int max_length_init_,
    unsigned int *l_vals_,
    unsigned int *m_vals_,
    int num_modes_,
    double *data_freqs_,
    cmplx *data_stream_, int data_stream_length_, double *X_ASDinv_, double *Y_ASDinv_, double *Z_ASDinv_){

    max_length_init = max_length_init_;
    l_vals = l_vals_;
    m_vals = m_vals_;
    num_modes = num_modes_;
    data_freqs = data_freqs_;
    data_stream = data_stream_;
    data_stream_length = data_stream_length_;
    X_ASDinv = X_ASDinv_;
    Y_ASDinv = Y_ASDinv_;
    Z_ASDinv = Z_ASDinv_;

    to_gpu = 1;

    hipError_t err;

    // DECLARE ALL THE  NECESSARY STRUCTS
    pHM_trans = new PhenomHMStorage;

    pAmp_trans = new IMRPhenomDAmplitudeCoefficients;

    amp_prefactors_trans = new AmpInsPrefactors;

    pDPreComp_all_trans = new PhenDAmpAndPhasePreComp[num_modes];

    q_all_trans = new HMPhasePreComp[num_modes];

  gpuErrchk(hipMalloc(&d_B, 7*data_stream_length_*num_modes*sizeof(double)));

  mode_vals = cpu_create_modes(num_modes, l_vals, m_vals, max_length_init, to_gpu, 1);

  gpuErrchk(hipMalloc(&d_H, 9*num_modes*sizeof(hipDoubleComplex)));

  gpuErrchk(hipMalloc(&d_X, data_stream_length*num_modes*sizeof(hipDoubleComplex)));
  gpuErrchk(hipMalloc(&d_Y, data_stream_length*num_modes*sizeof(hipDoubleComplex)));
  gpuErrchk(hipMalloc(&d_Z, data_stream_length*num_modes*sizeof(hipDoubleComplex)));

  d_mode_vals = gpu_create_modes(num_modes, l_vals, m_vals, max_length_init, to_gpu, 1);

  gpuErrchk(hipMalloc(&d_freqs, max_length_init*sizeof(double)));

  gpuErrchk(hipMalloc(&d_data_freqs, data_stream_length*sizeof(double)));
  gpuErrchk(hipMemcpy(d_data_freqs, data_freqs, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc(&d_data_stream, data_stream_length*sizeof(hipDoubleComplex)));
  gpuErrchk(hipMemcpy(d_data_stream, data_stream, data_stream_length*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc(&d_X_ASDinv, data_stream_length*sizeof(double)));
  gpuErrchk(hipMemcpy(d_X_ASDinv, X_ASDinv, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc(&d_Y_ASDinv, data_stream_length*sizeof(double)));
  gpuErrchk(hipMemcpy(d_Y_ASDinv, Y_ASDinv, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc(&d_Z_ASDinv, data_stream_length*sizeof(double)));
  gpuErrchk(hipMemcpy(d_Z_ASDinv, Z_ASDinv, data_stream_length*sizeof(double), hipMemcpyHostToDevice));

  gpuErrchk(hipMalloc(&d_pHM_trans, sizeof(PhenomHMStorage)));

  gpuErrchk(hipMalloc(&d_pAmp_trans, sizeof(IMRPhenomDAmplitudeCoefficients)));

  gpuErrchk(hipMalloc(&d_amp_prefactors_trans, sizeof(AmpInsPrefactors)));

  gpuErrchk(hipMalloc(&d_pDPreComp_all_trans, num_modes*sizeof(PhenDAmpAndPhasePreComp)));

  gpuErrchk(hipMalloc((void**) &d_q_all_trans, num_modes*sizeof(HMPhasePreComp)));


  double cShift[7] = {0.0,
                       PI_2 /* i shift */,
                       0.0,
                       -PI_2 /* -i shift */,
                       PI /* 1 shift */,
                       PI_2 /* -1 shift */,
                       0.0};

  gpuErrchk(hipMalloc(&d_cShift, 7*sizeof(double)));

  gpuErrchk(hipMemcpy(d_cShift, &cShift, 7*sizeof(double), hipMemcpyHostToDevice));


  // for likelihood
  // --------------
  stat = hipblasCreate(&handle);
  if (stat != HIPBLAS_STATUS_SUCCESS) {
          printf ("CUBLAS initialization failed\n");
          exit(0);
      }
      // ----------------

  //double t0_;
  t0 = 0.0;

  //double phi0_;
  phi0 = 0.0;

  //double amp0_;
  amp0 = 0.0;

  H = new cmplx[9*num_modes];
}


void PhenomHM::gen_amp_phase(double *freqs_, int current_length_,
    double m1_, //solar masses
    double m2_, //solar masses
    double chi1z_,
    double chi2z_,
    double distance_,
    double phiRef_,
    double f_ref_){

    assert(to_gpu == 1);
    assert(current_length_ <= max_length_init);

    PhenomHM::gen_amp_phase_prep(freqs_, current_length_,
        m1_, //solar masses
        m2_, //solar masses
        chi1z_,
        chi2z_,
        distance_,
        phiRef_,
        f_ref_);

    freqs = freqs_;
    current_length = current_length_;
    m1 = m1_; //solar masses
    m2 = m2_; //solar masses
    chi1z = chi1z_;
    chi2z = chi2z_;
    distance = distance_;
    phiRef = phiRef_;
    f_ref = f_ref_;

    gpuErrchk(hipMemcpy(d_freqs, freqs, current_length*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_pHM_trans, pHM_trans, sizeof(PhenomHMStorage), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_pAmp_trans, pAmp_trans, sizeof(IMRPhenomDAmplitudeCoefficients), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_amp_prefactors_trans, amp_prefactors_trans, sizeof(AmpInsPrefactors), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_pDPreComp_all_trans, pDPreComp_all_trans, num_modes*sizeof(PhenDAmpAndPhasePreComp), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_q_all_trans, q_all_trans, num_modes*sizeof(HMPhasePreComp), hipMemcpyHostToDevice));

    double M_tot_sec = (m1+m2)*MTSUN_SI;
    /* main: evaluate model at given frequencies */
    NUM_THREADS = 256;
    num_blocks = std::ceil((current_length + NUM_THREADS -1)/NUM_THREADS);
    dim3 gridDim(num_blocks, num_modes);
    //printf("blocks %d\n", num_blocks);
    kernel_calculate_all_modes<<<gridDim, NUM_THREADS>>>(d_mode_vals,
          d_pHM_trans,
          d_freqs,
          M_tot_sec,
          d_pAmp_trans,
          d_amp_prefactors_trans,
          d_pDPreComp_all_trans,
          d_q_all_trans,
          amp0,
          num_modes,
          t0,
          phi0,
          d_cShift
      );
     hipDeviceSynchronize();
     gpuErrchk(hipGetLastError());

     // ensure calls are run in correct order
     current_status = 1;
}

void PhenomHM::gen_amp_phase_prep(double *freqs, int current_length,
    double m1, //solar masses
    double m2, //solar masses
    double chi1z,
    double chi2z,
    double distance,
    double phiRef,
    double f_ref){

    // for phenomHM internal calls
    deltaF = -1.0;

    for (int i=0; i<num_modes; i++){
        mode_vals[i].length = current_length;
    }

    m1_SI = m1*MSUN_SI;
    m2_SI = m2*MSUN_SI;

    /* main: evaluate model at given frequencies */
    retcode = 0;
    retcode = IMRPhenomHMCore(
        mode_vals,
        freqs,
        current_length,
        m1_SI,
        m2_SI,
        chi1z,
        chi2z,
        distance,
        phiRef,
        deltaF,
        f_ref,
        num_modes,
        to_gpu,
        pHM_trans,
        pAmp_trans,
        amp_prefactors_trans,
        pDPreComp_all_trans,
        q_all_trans,
        &t0,
        &phi0,
        &amp0);
    assert (retcode == 1); //,PD_EFUNC, "IMRPhenomHMCore failed in
}


void PhenomHM::setup_interp_wave(){

    assert(current_status >= 1);
    dim3 waveInterpDim(num_blocks, num_modes);

    fill_B_wave<<<waveInterpDim, NUM_THREADS>>>(d_mode_vals, d_B, current_length, num_modes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    interp.prep(d_B, current_length, 2*num_modes, 1);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    set_spline_constants_wave<<<waveInterpDim, NUM_THREADS>>>(d_mode_vals, d_B, current_length, num_modes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    if (current_status == 1) current_status = 2;
}

void PhenomHM::LISAresponseFD(double inc_, double lam_, double beta_, double psi_, double t0_epoch_, double tRef_, double merger_freq_, int TDItag_){
    inc = inc_;
    lam = lam_;
    beta = beta_;
    psi = psi_;
    t0_epoch = t0_epoch_;
    tRef = tRef_;
    TDItag = TDItag_;
    merger_freq = merger_freq_;

    assert(current_status >= 2);

    prep_H_info(H, l_vals, m_vals, num_modes, inc, lam, beta, psi, phiRef);
    gpuErrchk(hipMemcpy(d_H, H, 9*num_modes*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    double d_log10f = log10(freqs[1]) - log10(freqs[0]);

    int num_blocks = std::ceil((current_length + NUM_THREADS - 1)/NUM_THREADS);
    dim3 gridDim(num_blocks, num_modes);

    kernel_JustLISAFDresponseTDI_wrap<<<gridDim, NUM_THREADS>>>(d_mode_vals, d_H, d_freqs, d_freqs, d_log10f, d_l_vals, d_m_vals, num_modes, current_length, inc, lam, beta, psi, phiRef, t0_epoch, tRef, merger_freq, TDItag, 0);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    if (current_status == 2) current_status = 3;
}

void PhenomHM::setup_interp_response(){

    assert(current_status >= 3);

    dim3 responseInterpDim(num_blocks, num_modes);

    fill_B_response<<<responseInterpDim, NUM_THREADS>>>(d_mode_vals, d_B, current_length, num_modes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    interp.prep(d_B, current_length, 7*num_modes, 1);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    set_spline_constants_response<<<responseInterpDim, NUM_THREADS>>>(d_mode_vals, d_B, current_length, num_modes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    if (current_status == 3) current_status = 4;
}

void PhenomHM::perform_interp(double f_min, double df, int length_new){
    assert(current_status >= 4);
    assert(length_new == data_stream_length);
    int num_block_interp = std::ceil((length_new + NUM_THREADS - 1)/NUM_THREADS);
    dim3 mainInterpDim(num_block_interp, num_modes);
    double d_log10f = log10(freqs[1]) - log10(freqs[0]);

    interpolate<<<mainInterpDim, NUM_THREADS>>>(d_X, d_Y, d_Z, d_mode_vals, num_modes, f_min, df, d_log10f, d_freqs, current_length, length_new, t0, tRef, d_X_ASDinv, d_Y_ASDinv, d_Z_ASDinv);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    if (current_status == 4) current_status = 5;
}

void PhenomHM::Likelihood (double *like_out_){

     assert(current_status == 5);
     double d_h = 0.0;
     double h_h = 0.0;
     char * status;
     double res;
     hipDoubleComplex result;
     for (int mode_i=0; mode_i<num_modes; mode_i++){
         stat = hipblasZdotc(handle, data_stream_length,
                 &d_X[mode_i*data_stream_length], 1,
                 d_data_stream, 1,
                 &result);
         status = _cudaGetErrorEnum(stat);
          hipDeviceSynchronize();

          if (stat != HIPBLAS_STATUS_SUCCESS) {
                  exit(0);
              }
         d_h += hipCreal(result);

         stat = hipblasZdotc(handle, data_stream_length,
                 &d_Y[mode_i*data_stream_length], 1,
                 d_data_stream, 1,
                 &result);
         status = _cudaGetErrorEnum(stat);
          hipDeviceSynchronize();

          if (stat != HIPBLAS_STATUS_SUCCESS) {
                  exit(0);
              }
         d_h += hipCreal(result);

         stat = hipblasZdotc(handle, data_stream_length,
                 &d_Z[mode_i*data_stream_length], 1,
                 d_data_stream, 1,
                 &result);
         status = _cudaGetErrorEnum(stat);
          hipDeviceSynchronize();

          if (stat != HIPBLAS_STATUS_SUCCESS) {
                  exit(0);
              }
         d_h += hipCreal(result);
     }

     // d_X d_X for h_h
     stat = hipblasDznrm2(handle, num_modes*data_stream_length,
             d_X, 1, &res);
     status = _cudaGetErrorEnum(stat);
      hipDeviceSynchronize();

      if (stat != HIPBLAS_STATUS_SUCCESS) {
              exit(0);
          }
        h_h += res*res; //TODO: MAKE SURE THIS IS RIGHT

      // d_Y d_Y for h_h
      stat = hipblasDznrm2(handle, num_modes*data_stream_length,
              d_Y, 1, &res);
      status = _cudaGetErrorEnum(stat);
       hipDeviceSynchronize();

       if (stat != HIPBLAS_STATUS_SUCCESS) {
               exit(0);
           }
         h_h += res*res;

       // d_Z d_Z for h_h
       stat = hipblasDznrm2(handle, num_modes*data_stream_length,
               d_Z, 1, &res);
       status = _cudaGetErrorEnum(stat);
        hipDeviceSynchronize();

        if (stat != HIPBLAS_STATUS_SUCCESS) {
                exit(0);
            }
     h_h += res*res;

     like_out_[0] = d_h;
     like_out_[1] = h_h;
}

void PhenomHM::GetTDI (cmplx* X_, cmplx* Y_, cmplx* Z_) {

  assert(current_status > 4);
  gpuErrchk(hipMemcpy(X_, d_X, data_stream_length*num_modes*sizeof(cmplx), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(Y_, d_Y, data_stream_length*num_modes*sizeof(cmplx), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(Z_, d_Z, data_stream_length*num_modes*sizeof(cmplx), hipMemcpyDeviceToHost));
}

__global__ void read_out_amp_phase(ModeContainer *mode_vals, double *amp, double *phase, int num_modes, int length){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int mode_i = blockIdx.y;
    if (i >= length) return;
    if (mode_i >= num_modes) return;
    amp[mode_i*length + i] = mode_vals[mode_i].amp[i];
    phase[mode_i*length + i] = mode_vals[mode_i].phase[i];
}

void PhenomHM::GetAmpPhase(double* amp_, double* phase_) {
  assert(current_status > 1);
  double *amp, *phase;
  gpuErrchk(hipMalloc(&amp, num_modes*current_length*sizeof(double)));
  gpuErrchk(hipMalloc(&phase, num_modes*current_length*sizeof(double)));

  dim3 readOutDim(num_blocks, num_modes);
  read_out_amp_phase<<<readOutDim, NUM_THREADS>>>(d_mode_vals, amp, phase, num_modes, current_length);
  hipDeviceSynchronize();
  gpuErrchk(hipGetLastError());

  gpuErrchk(hipMemcpy(amp_, amp, num_modes*current_length*sizeof(double), hipMemcpyDeviceToHost));
  gpuErrchk(hipMemcpy(phase_, phase, num_modes*current_length*sizeof(double), hipMemcpyDeviceToHost));

  gpuErrchk( hipFree(amp));
  gpuErrchk(hipFree(phase));
}


PhenomHM::~PhenomHM() {
  delete pHM_trans;
  delete pAmp_trans;
  delete amp_prefactors_trans;
  delete[] pDPreComp_all_trans;
  delete[] q_all_trans;
  cpu_destroy_modes(mode_vals);
  delete[] H;

  gpuErrchk(hipFree(d_freqs));
  gpuErrchk(hipFree(d_data_freqs));
  gpuErrchk(hipFree(d_data_stream));
  gpu_destroy_modes(d_mode_vals);
  gpuErrchk(hipFree(d_pHM_trans));
  gpuErrchk(hipFree(d_pAmp_trans));
  gpuErrchk(hipFree(d_amp_prefactors_trans));
  gpuErrchk(hipFree(d_pDPreComp_all_trans));
  gpuErrchk(hipFree(d_q_all_trans));
  gpuErrchk(hipFree(d_cShift));
  gpuErrchk(hipFree(d_X));
  gpuErrchk(hipFree(d_Y));
  gpuErrchk(hipFree(d_Z));
  gpuErrchk(hipFree(d_X_ASDinv));
  gpuErrchk(hipFree(d_Y_ASDinv));
  gpuErrchk(hipFree(d_Z_ASDinv));
  hipblasDestroy(handle);
  gpuErrchk(hipFree(d_B));
}
