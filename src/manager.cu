#include "hip/hip_runtime.h"
/*
This is the central piece of code. This file implements a class
(interface in gpuadder.hh) that takes data in on the cpu side, copies
it to the gpu, and exposes functions (increment and retreive) that let
you perform actions with the GPU

This class will get translated into python via swig
*/

#include <kernel.cu>
//#include <reduction.cu>
#include <manager.hh>
#include <assert.h>
#include <iostream>
#include "globalPhenomHM.h"
#include <complex>
#include "hip/hip_complex.h"
#include "hipblas.h"
#include "interpolate.cu"
#include "fdresponse.h"
#include "createGPUHolders.cu"
#include "kernel_response.cu"


using namespace std;

GPUPhenomHM::GPUPhenomHM (int max_length_,
    unsigned int *l_vals_,
    unsigned int *m_vals_,
    int num_modes_,
    int to_gpu_,
    int to_interp_,
    std::complex<double> *data_stream_, int data_stream_length_){

    max_length = max_length_;
    l_vals = l_vals_;
    m_vals = m_vals_;
    num_modes = num_modes_;
    to_gpu = to_gpu_;
    to_interp = to_interp_;
    data_stream = data_stream_;
    data_stream_length = data_stream_length_;

    hipError_t err;

    // DECLARE ALL THE  NECESSARY STRUCTS
    pHM_trans = new PhenomHMStorage;

    pAmp_trans = new IMRPhenomDAmplitudeCoefficients;

    amp_prefactors_trans = new AmpInsPrefactors;

    pDPreComp_all_trans = new PhenDAmpAndPhasePreComp[num_modes];

    q_all_trans = new HMPhasePreComp[num_modes];

    hI = new std::complex<double>[data_stream_length*num_modes];
    hII = new std::complex<double>[data_stream_length*num_modes];

    X = new std::complex<double>[data_stream_length*num_modes];
    Y = new std::complex<double>[data_stream_length*num_modes];
    Z = new std::complex<double>[data_stream_length*num_modes];


  mode_vals = cpu_create_modes(num_modes, l_vals, m_vals, max_length, to_gpu, to_interp);

  if (to_gpu == 1){

      gpuErrchk(hipMalloc(&d_H, 9*num_modes*sizeof(hipDoubleComplex)));

      hipDoubleComplex * ones = new hipDoubleComplex[num_modes];
      for (int i=0; i<(num_modes); i++) ones[i] = make_hipDoubleComplex(1.0, 0.0);
      gpuErrchk(hipMalloc(&d_ones, num_modes*sizeof(hipDoubleComplex)));
      gpuErrchk(hipMemcpy(d_ones, ones, num_modes*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
      delete ones;

      gpuErrchk(hipMalloc(&d_hI, data_stream_length*num_modes*sizeof(hipDoubleComplex)));
      gpuErrchk(hipMalloc(&d_hII, data_stream_length*num_modes*sizeof(hipDoubleComplex)));

      gpuErrchk(hipMalloc(&d_hI_out, data_stream_length*sizeof(hipDoubleComplex)));
      gpuErrchk(hipMalloc(&d_hII_out, data_stream_length*sizeof(hipDoubleComplex)));

      gpuErrchk(hipMalloc(&d_X, data_stream_length*num_modes*sizeof(hipDoubleComplex)));
      gpuErrchk(hipMalloc(&d_Y, data_stream_length*num_modes*sizeof(hipDoubleComplex)));
      gpuErrchk(hipMalloc(&d_Z, data_stream_length*num_modes*sizeof(hipDoubleComplex)));

      d_mode_vals = gpu_create_modes(num_modes, l_vals, m_vals, max_length, to_gpu, to_interp);

      gpuErrchk(hipMalloc(&d_freqs, max_length*sizeof(double)));

      gpuErrchk(hipMalloc(&d_data_stream, data_stream_length*sizeof(hipDoubleComplex)));
      gpuErrchk(hipMemcpy(d_data_stream, data_stream, data_stream_length*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));

      //gpuErrchk(hipMalloc(&d_mode_vals, num_modes*sizeof(d_mode_vals)));
      //gpuErrchk(hipMemcpy(d_mode_vals, mode_vals, num_modes*sizeof(d_mode_vals), hipMemcpyHostToDevice));

      // DECLARE ALL THE  NECESSARY STRUCTS
      gpuErrchk(hipMalloc(&d_pHM_trans, sizeof(PhenomHMStorage)));

      gpuErrchk(hipMalloc(&d_pAmp_trans, sizeof(IMRPhenomDAmplitudeCoefficients)));

      gpuErrchk(hipMalloc(&d_amp_prefactors_trans, sizeof(AmpInsPrefactors)));

      gpuErrchk(hipMalloc(&d_pDPreComp_all_trans, num_modes*sizeof(PhenDAmpAndPhasePreComp)));

      gpuErrchk(hipMalloc((void**) &d_q_all_trans, num_modes*sizeof(HMPhasePreComp)));


      double cShift[7] = {0.0,
                           PI_2 /* i shift */,
                           0.0,
                           -PI_2 /* -i shift */,
                           PI /* 1 shift */,
                           PI_2 /* -1 shift */,
                           0.0};

      gpuErrchk(hipMalloc(&d_cShift, 7*sizeof(double)));

      gpuErrchk(hipMemcpy(d_cShift, &cShift, 7*sizeof(double), hipMemcpyHostToDevice));


      // for likelihood
      // --------------
      gpuErrchk(hipHostMalloc((hipDoubleComplex**) &result, sizeof(hipDoubleComplex)));

      stat = hipblasCreate(&handle);
      if (stat != HIPBLAS_STATUS_SUCCESS) {
              printf ("CUBLAS initialization failed\n");
              exit(0);
          }
      // ----------------
  }
  //double t0_;
  t0 = 0.0;

  //double phi0_;
  phi0 = 0.0;

  //double amp0_;
  amp0 = 0.0;
}


void GPUPhenomHM::add_interp(int max_interp_length_){
    max_interp_length = max_interp_length_;

    assert(to_interp == 1);
    if (to_gpu == 0){
        out_mode_vals = cpu_create_modes(num_modes, m_vals, l_vals, max_interp_length, to_gpu, 0);
        B = new double[7*max_interp_length*num_modes];
    }
    if (to_gpu){

        h_indices = new int[max_interp_length];
        hipMalloc(&d_indices, max_interp_length*sizeof(int));
        //d_out_mode_vals = gpu_create_modes(num_modes, m_vals, l_vals, max_interp_length, to_gpu, 0);
        //h_B = new double[2*f_length*num_modes];
        //h_B1 = new double[2*f_length*num_modes];*/
        gpuErrchk(hipMalloc(&d_B, 7*max_interp_length_*num_modes*sizeof(double)));
    }
}



void GPUPhenomHM::gpu_gen_PhenomHM(double *freqs_, int f_length_,
    double m1_, //solar masses
    double m2_, //solar masses
    double chi1z_,
    double chi2z_,
    double distance_,
    double inclination_,
    double phiRef_,
    double deltaF_,
    double f_ref_){

    assert((to_gpu == 1) || (to_gpu == 2));

    GPUPhenomHM::cpu_gen_PhenomHM(freqs_, f_length_,
        m1_, //solar masses
        m2_, //solar masses
        chi1z_,
        chi2z_,
        distance_,
        inclination_,
        phiRef_,
        deltaF_,
        f_ref_);

    // Initialize inputs
    //gpuErrchk(hipMemcpy(d_mode_vals, mode_vals, num_modes*sizeof(ModeContainer), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_freqs, freqs, f_length*sizeof(double), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_pHM_trans, pHM_trans, sizeof(PhenomHMStorage), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_pAmp_trans, pAmp_trans, sizeof(IMRPhenomDAmplitudeCoefficients), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_amp_prefactors_trans, amp_prefactors_trans, sizeof(AmpInsPrefactors), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_pDPreComp_all_trans, pDPreComp_all_trans, num_modes*sizeof(PhenDAmpAndPhasePreComp), hipMemcpyHostToDevice));

    gpuErrchk(hipMemcpy(d_q_all_trans, q_all_trans, num_modes*sizeof(HMPhasePreComp), hipMemcpyHostToDevice));

    double M_tot_sec = (m1+m2)*MTSUN_SI;
    /* main: evaluate model at given frequencies */
    NUM_THREADS = 256;
    num_blocks = std::ceil((f_length + NUM_THREADS -1)/NUM_THREADS);
    dim3 gridDim(num_modes, num_blocks);
    //printf("blocks %d\n", num_blocks);
    kernel_calculate_all_modes<<<gridDim, NUM_THREADS>>>(d_mode_vals,
          d_pHM_trans,
          d_freqs,
          M_tot_sec,
          d_pAmp_trans,
          d_amp_prefactors_trans,
          d_pDPreComp_all_trans,
          d_q_all_trans,
          amp0,
          num_modes,
          t0,
          phi0,
          d_cShift
      );
     hipDeviceSynchronize();
     gpuErrchk(hipGetLastError());

}


void GPUPhenomHM::cpu_gen_PhenomHM(double *freqs_, int f_length_,
    double m1_, //solar masses
    double m2_, //solar masses
    double chi1z_,
    double chi2z_,
    double distance_,
    double inclination_,
    double phiRef_,
    double deltaF_,
    double f_ref_){

    freqs = freqs_;
    f_length = f_length_;
    m1 = m1_; //solar masses
    m2 = m2_; //solar masses
    chi1z = chi1z_;
    chi2z = chi2z_;
    distance = distance_;
    inclination = inclination_;
    phiRef = phiRef_;
    deltaF = deltaF_;
    f_ref = f_ref_;

    for (int i=0; i<num_modes; i++){
        mode_vals[i].length = f_length;
    }

    m1_SI = m1*MSUN_SI;
    m2_SI = m2*MSUN_SI;

    /* main: evaluate model at given frequencies */
    retcode = 0;
    retcode = IMRPhenomHMCore(
        mode_vals,
        freqs,
        f_length,
        m1_SI,
        m2_SI,
        chi1z,
        chi2z,
        distance,
        inclination,
        phiRef,
        deltaF,
        f_ref,
        num_modes,
        to_gpu,
        pHM_trans,
        pAmp_trans,
        amp_prefactors_trans,
        pDPreComp_all_trans,
        q_all_trans,
        &t0,
        &phi0,
        &amp0);
    assert (retcode == 1); //,PD_EFUNC, "IMRPhenomHMCore failed in

}


__global__ void read_out_kernel2(ModeContainer *mode_vals, double *coef0, double *coef1, double *coef2, double *coef3, int mode_i, int length){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= length) return;
    coef0[i] = mode_vals[mode_i].amp[i];
    coef1[i] = mode_vals[mode_i].amp_coeff_1[i];
    coef2[i] = mode_vals[mode_i].amp_coeff_2[i];
    coef3[i] = mode_vals[mode_i].amp_coeff_3[i];
    //phase[i] = mode_vals[mode_i].phase[i];
}

__global__ void debug(ModeContainer *mode_vals, int num_modes, int length){
    int i = blockIdx.y * blockDim.x + threadIdx.x;
    int mode_i = blockIdx.x;
    if (mode_i >= num_modes) return;
    if (i >= length) return;
    double amp = mode_vals[mode_i].amp[i];
    double phase = mode_vals[mode_i].phase[i];
    //phase[i] = mode_vals[mode_i].phase[i];
}

void GPUPhenomHM::gpu_setup_interp_wave(){

    dim3 check_dim(num_modes, num_blocks);
    int check_num_threads = 256;
    /*debug<<<check_dim, NUM_THREADS>>>(d_mode_vals, num_modes, f_length);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());*/

    fill_B_wave<<<check_dim, NUM_THREADS>>>(d_mode_vals, d_B, f_length, num_modes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    interp.prep(d_B, f_length, 2*num_modes, 1);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    set_spline_constants_wave<<<check_dim, NUM_THREADS>>>(d_mode_vals, d_B, f_length, num_modes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
}

void GPUPhenomHM::cpu_setup_interp_wave(){
    host_fill_B_wave(mode_vals, B, f_length, num_modes);
    interp.prep(B, f_length, 2*num_modes, 0);
    host_set_spline_constants_wave(mode_vals, B, f_length, num_modes);
}

//void GPUPhenomHM::cpu_setup_interp_response(double f_min, double df, int length_new)


void GPUPhenomHM::gpu_LISAresponseFD(double inc_, double lam_, double beta_, double psi_, double tc_, double tShift_, int TDItag_){
    inc = inc_;
    lam = lam_;
    beta = beta_;
    psi = psi_;
    tc = tc_;
    tShift = tShift_;
    TDItag = TDItag_;

    H = prep_H_info(l_vals, m_vals, num_modes, inc, lam, beta, psi, phi0);
    gpuErrchk(hipMemcpy(d_H, H, 9*num_modes*sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    double d_log10f = log10(freqs[1]) - log10(freqs[0]);

    int num_blocks = std::ceil((f_length + NUM_THREADS - 1)/NUM_THREADS);
    dim3 gridDim(num_modes, num_blocks);

    kernel_JustLISAFDresponseTDI_wrap<<<gridDim, NUM_THREADS>>>(d_mode_vals, d_H, d_freqs, d_freqs, d_log10f, d_l_vals, d_m_vals, num_modes, f_length, inc, lam, beta, psi, phi0, tc, tShift, TDItag, 0);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
}

void GPUPhenomHM::cpu_LISAresponseFD(double inc_, double lam_, double beta_, double psi_, double tc_, double tShift_, int TDItag_){
    inc = inc_;
    lam = lam_;
    beta = beta_;
    psi = psi_;
    tc = tc_;
    tShift = tShift_;
    TDItag = TDItag_;

    H = prep_H_info(l_vals, m_vals, num_modes, inc, lam, beta, psi, phi0);
    double d_log10f = log10(freqs[1]) - log10(freqs[0]);
    JustLISAFDresponseTDI_wrap(mode_vals, H, freqs, freqs, d_log10f, l_vals, m_vals, num_modes, f_length, inc, lam, beta, psi, phi0, tc, tShift, TDItag, 0);
}

void GPUPhenomHM::gpu_setup_interp_response(){

    dim3 check_dim(num_modes, num_blocks);
    int check_num_threads = 256;
    /*debug<<<check_dim, NUM_THREADS>>>(d_mode_vals, num_modes, f_length);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());*/

    fill_B_response<<<check_dim, NUM_THREADS>>>(d_mode_vals, d_B, f_length, num_modes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    interp.prep(d_B, f_length, 7*num_modes, 1);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());

    set_spline_constants_response<<<check_dim, NUM_THREADS>>>(d_mode_vals, d_B, f_length, num_modes);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
}

void GPUPhenomHM::cpu_setup_interp_response(){
    host_fill_B_response(mode_vals, B, f_length, num_modes);
    interp.prep(B, f_length, 7*num_modes, 0);
    host_set_spline_constants_response(mode_vals, B, f_length, num_modes);
}

void GPUPhenomHM::gpu_perform_interp(double f_min, double df, int length_new){
    int num_block_interp = std::ceil((length_new + NUM_THREADS - 1)/NUM_THREADS);
    dim3 interp_dim(num_modes, num_block_interp);
    double d_log10f = log10(freqs[1]) - log10(freqs[0]);
    //printf("NUM MODES %d\n", num_modes);
    interpolate<<<interp_dim, NUM_THREADS>>>(d_X, d_Y, d_Z, d_mode_vals, num_modes, f_min, df, d_log10f, d_freqs, length_new, tc, tShift);
    hipDeviceSynchronize();
    gpuErrchk(hipGetLastError());
    //TODO need to make this more adaptable (especially for smaller amounts)
}

void GPUPhenomHM::cpu_perform_interp(double f_min, double df, int length_new){
    double d_log10f = log10(freqs[1]) - log10(freqs[0]);
    host_interpolate(X, Y, Z, mode_vals, num_modes, f_min, df, d_log10f, freqs, length_new, tc, tShift);
}

/*
__global__ void debug2(hipDoubleComplex *hI, hipDoubleComplex *hI_out, hipDoubleComplex *ones, int length, int num_modes){
    int i = blockIdx.y * blockDim.x + threadIdx.x;
    int mode_i = blockIdx.x;
    if (mode_i >= num_modes) return;
    if (i >= length) return;
    int j = 0;
    //phase[i] = mode_vals[mode_i].phase[i];
}*/

int GpuVec(hipDoubleComplex* d_A, hipDoubleComplex* d_x, hipDoubleComplex* d_y, const int row,const int col){
hipError_t cudastat;
hipblasStatus_t stat;
int size=row*col;
hipblasHandle_t handle;
/*hipDoubleComplex* d_A;  //device matrix
hipDoubleComplex* d_x;  //device vector
hipDoubleComplex* d_y;  //device result
cudastat=hipMalloc((void**)&d_A,size*sizeof(hipDoubleComplex));
cudastat=hipMalloc((void**)&d_x,col*sizeof(hipDoubleComplex));
cudastat=hipMalloc((void**)&d_y,row*sizeof(hipDoubleComplex));// when I copy y to d_y ,can I cout d_y?

hipMemcpy(d_A,A,sizeof(hipDoubleComplex)*size,hipMemcpyHostToDevice);  //copy A to device d_A
hipMemcpy(d_x,x,sizeof(hipDoubleComplex)*col,hipMemcpyHostToDevice);*/   //copy x to device d_x

hipDoubleComplex alf=make_hipDoubleComplex(1.0,0.0);
hipDoubleComplex beta=make_hipDoubleComplex(0.0,0.0);
    stat=hipblasCreate(&handle);
/*int NUM_THREADS = 256;
int num_blockshere = (int)(row + NUM_THREADS -1)/NUM_THREADS;
dim3 likeDim(col, num_blockshere);
debug2<<<likeDim, NUM_THREADS>>>(d_A, d_y, d_x, row, col);
hipDeviceSynchronize();
gpuErrchk(hipGetLastError());*/
stat=hipblasZgemv(handle,HIPBLAS_OP_T,col,row,&alf,d_A,col,d_x,1,&beta,d_y,1);//swap col and row
/*hipMemcpy(y,d_y,sizeof(hipDoubleComplex)*row,hipMemcpyDeviceToHost); // copy device result to host
hipFree(d_A);
hipFree(d_x);
hipFree(d_y);*/
hipblasDestroy(handle);
return 0;
}


double GPUPhenomHM::Likelihood (int like_length){

     hipDoubleComplex res_out = make_hipDoubleComplex(0.0, 0.0);
     char * status;
     for (int mode_i=0; mode_i<num_modes; mode_i++){
         stat = hipblasZdotc(handle, like_length,
                 //d_hI_out, 1,
                 &d_hI[mode_i*like_length], 1,
                 d_data_stream, 1,
                 result);
         status = _cudaGetErrorEnum(stat);
          hipDeviceSynchronize();
          //printf ("%s\n", status);
          if (stat != HIPBLAS_STATUS_SUCCESS) {
                  exit(0);
              }
         res_out = hipCadd(res_out, result[0]);
     }

    //gpuErrchk(hipGetLastError());


    //return hipCreal(result[0]);
    return hipCreal(res_out);
    //return 0.0;
}

void GPUPhenomHM::Get_Waveform (std::complex<double>* X_, std::complex<double>* Y_, std::complex<double>* Z_) {
  assert(to_gpu == 0);
  memcpy(X_, X, max_interp_length*num_modes*sizeof(std::complex<double>));
  memcpy(Y_, Y, max_interp_length*num_modes*sizeof(std::complex<double>));
  memcpy(Z_, Z, max_interp_length*num_modes*sizeof(std::complex<double>));
}

__global__ void read_out_kernel(ModeContainer *mode_vals, double *amp, double *phase, int mode_i, int length){
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= length) return;
    amp[i] = mode_vals[mode_i].amp[i];
    phase[i] = mode_vals[mode_i].phase[i];
}

void GPUPhenomHM::gpu_Get_Waveform (std::complex<double>* hI_) {
  assert(to_gpu == 1);
  gpuErrchk(hipMemcpy(hI_, d_hI, max_interp_length*num_modes*sizeof(std::complex<double>), hipMemcpyDeviceToHost));
}

GPUPhenomHM::~GPUPhenomHM() {
  delete pHM_trans;
  delete pAmp_trans;
  delete amp_prefactors_trans;
  delete pDPreComp_all_trans;
  delete q_all_trans;
  cpu_destroy_modes(mode_vals);
  delete hI;
  delete hII;
  delete X;
  delete Y;
  delete Z;

  if (to_gpu == 1){
      hipFree(d_ones);
      hipFree(d_hI);
      hipFree(d_hII);
      hipFree(d_hI_out);
      hipFree(d_hII_out);
      hipFree(d_freqs);
      hipFree(d_data_stream);
      gpu_destroy_modes(d_mode_vals);
      hipFree(d_pHM_trans);
      hipFree(d_pAmp_trans);
      hipFree(d_amp_prefactors_trans);
      hipFree(d_pDPreComp_all_trans);
      hipFree(d_q_all_trans);
      hipFree(d_cShift);
      hipFree(result);
      hipFree(d_X);
      hipFree(d_Y);
      hipFree(d_Z);
      hipblasDestroy(handle);
  }
  if (to_interp == 1){
      if (to_gpu == 0){
          delete B;
          cpu_destroy_modes(out_mode_vals);
      }
      delete h_indices;
      hipFree(d_indices);
      hipFree(d_B);
      gpu_destroy_modes(d_out_mode_vals);
      //delete interp;
  }
}
