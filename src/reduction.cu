
#include <hip/hip_runtime.h>
#include "hip/hip_complex.h"
static const int blockSize = 256;

__global__ void sumCommSingleBlock(const hipDoubleComplex *a, double *out, int arraySize) {
    int idx = threadIdx.x;
    //static const int blockSize = blockDim.x;
    double sum_re = 0.0;
    double sum_im = 0.0;
    for (int i = idx; i < arraySize; i += blockSize){
        sum_re += hipCreal(a[i]);
        sum_im += hipCimag(a[i]);
    }
    __shared__ double r[blockSize];
    __shared__ double im[blockSize];
    r[idx] = sum_re;
    im[idx] = sum_im;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (idx<size){
            r[idx] += r[idx+size];
            im[idx] += im[idx+size];
        }
        __syncthreads();
    }
    if (idx == 0){
        out[0] = r[0];
        out[1] = im[0];
    }
}


__global__ void sumCommMultiBlock(const hipDoubleComplex *gArr, int arraySize, hipDoubleComplex *gOut, int first_run) {
    int thIdx = threadIdx.x;
    int gthIdx = thIdx + blockIdx.x*blockSize;
    const int gridSize = blockSize*gridDim.x;
    double sum1 = 0;
    double sum2 = 0;
    hipDoubleComplex trans;
    for (int i = gthIdx; i < arraySize; i += gridSize){
        if (first_run == 1) trans = hipCmul(hipConj(gArr[i]), gArr[i]);
        else trans = gArr[i];
        sum1 += hipCreal(trans);
        sum2 += hipCimag(trans);
    }
    __shared__ double shArr1[blockSize];
    __shared__ double shArr2[blockSize];
    shArr1[thIdx] = sum1;
    shArr2[thIdx] = sum2;
    __syncthreads();
    for (int size = blockSize/2; size>0; size/=2) { //uniform
        if (thIdx<size){
            shArr1[thIdx] += shArr1[thIdx+size];
            shArr2[thIdx] += shArr2[thIdx+size];
        }

        __syncthreads();
    }
    if (thIdx == 0)
        gOut[blockIdx.x] = make_hipDoubleComplex(shArr1[0], shArr2[0]);
}
